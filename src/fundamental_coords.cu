#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <complex.h>
#include <math.h>
#include "constants.h"
#include "cudacheck.h"
#include "woden_precision_defs.h"

__device__ void calc_uvw(double *d_X_diff, double *d_Y_diff,
                         double *d_Z_diff,
                         double sdec0, double cdec0,
                         double sha0, double cha0,
                         int iBaseline, int num_baselines,
                         user_precision_t * u, user_precision_t * v,
                         user_precision_t * w) {

  int mod_baseline = iBaseline - num_baselines*floorf((float)iBaseline / (float)num_baselines);

  * u = (sha0*d_X_diff[mod_baseline]) + (cha0*d_Y_diff[mod_baseline]);
  * v = -(sdec0*cha0*d_X_diff[mod_baseline]) + (sdec0*sha0*d_Y_diff[mod_baseline]) + (cdec0*d_Z_diff[mod_baseline]);
  * w = (cdec0*cha0*d_X_diff[mod_baseline]) - (cdec0*sha0*d_Y_diff[mod_baseline]) + (sdec0*d_Z_diff[mod_baseline]);

}

__global__ void kern_calc_uvw(double *d_X_diff, double *d_Y_diff,
           double *d_Z_diff, user_precision_t *d_u_metres,
           user_precision_t *d_v_metres, user_precision_t *d_w_metres,
           user_precision_t *d_u, user_precision_t *d_v, user_precision_t *d_w, user_precision_t *d_wavelengths,
           double sdec0, double cdec0,
           double *d_cha0s, double *d_sha0s,
           int num_visis, int num_baselines){
  // Start by computing which baseline we're going to do
  const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);

  if (iBaseline < num_visis){
    user_precision_t u, v, w;

    double d_sha0 = d_sha0s[iBaseline];
    double d_cha0 = d_cha0s[iBaseline];

    calc_uvw(d_X_diff, d_Y_diff, d_Z_diff,
               sdec0, cdec0, d_sha0, d_cha0,
               iBaseline, num_baselines,
               &u, &v, &w);

    d_u_metres[iBaseline] = u;
    d_v_metres[iBaseline] = v;
    d_w_metres[iBaseline] = w;

    user_precision_t d_wavelength = d_wavelengths[iBaseline];

    d_u[iBaseline] = u / d_wavelength;
    d_v[iBaseline] = v / d_wavelength;
    d_w[iBaseline] = w / d_wavelength;
  }
}

/*TODO: this might be faster to just loop over the inside the kernel? */
__global__ void kern_calc_uvw_shapelet(double *d_X_diff,
      double *d_Y_diff, double *d_Z_diff,
      user_precision_t *d_u_shapes, user_precision_t *d_v_shapes,
      user_precision_t *d_w_shapes, user_precision_t *d_wavelengths,
      double *d_lsts, double *d_ras, double *d_decs,
      const int num_baselines, const int num_visis,
      const int num_shapes) {
  // Start by computing which baseline we're going to do
  const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);
  const int iComponent = threadIdx.y + (blockDim.y*blockIdx.y);

  if(iBaseline < num_visis && iComponent < num_shapes) {

    user_precision_t u_shape, v_shape, w_shape;
    //TODO do the sin/cos outside of the GPU kernel?
    double d_sdec0 = sin(d_decs[iComponent]);
    double d_cdec0 = cos(d_decs[iComponent]);
    double d_sha0 = sin(d_lsts[iBaseline] - d_ras[iComponent]);
    double d_cha0 = cos(d_lsts[iBaseline] - d_ras[iComponent]);
    user_precision_t d_wavelength = d_wavelengths[iBaseline];

    calc_uvw(d_X_diff, d_Y_diff, d_Z_diff,
               d_sdec0, d_cdec0, d_sha0, d_cha0,
               iBaseline, num_baselines,
               &u_shape, &v_shape, &w_shape);

    d_u_shapes[num_visis*iComponent + iBaseline] = u_shape / d_wavelength;
    d_v_shapes[num_visis*iComponent + iBaseline] = v_shape / d_wavelength;
    d_w_shapes[num_visis*iComponent + iBaseline] = w_shape / d_wavelength;
  }
}

__device__ void calc_lmn(double ra0, double sdec0,
                         double cdec0,
                         double ra, double dec,
                         double * l, double * m, double * n){
  double cdec;
  double sdec;
  double cdra;
  double sdra;

  cdec = cos(dec);
  sdec = sin(dec);
  cdra = cos((ra - ra0));
  sdra = sin((ra - ra0));

  * l = cdec*sdra;
  * m = sdec*cdec0 - cdec*sdec0*cdra;
  * n = sdec*sdec0 + cdec*cdec0*cdra;

  //Note we could calculate n this way, which gives exactly zero at the horizon,
  //but anything below the horizon should have a negative n, and this makes n
  //positive everywhere
  // double temp_n = sqrt(1.0 -temp_l*temp_l - temp_m*temp_m );
}

__global__ void kern_calc_lmn(double ra0, double sdec0,
                              double cdec0,
                              double *d_ras, double *d_decs,
                              double *d_l, double *d_m, double *d_n,
                              int num_components){

  const int iComponent = threadIdx.x + (blockDim.x*blockIdx.x);

  if (iComponent < num_components){
    double l, m, n;

    calc_lmn(ra0, sdec0, cdec0,
             d_ras[iComponent], d_decs[iComponent],
             &l, &m, &n);

    d_l[iComponent] = l;
    d_m[iComponent] = m;
    d_n[iComponent] = n;

  }
}

/*******************************************************************************
                 Functions below to be used in unit tests
*******************************************************************************/

extern "C" void test_kern_calc_lmn(double ra0, double dec0,
                                   double *ras, double *decs, int num_coords,
                                   double * ls, double * ms, double * ns) {

  double *d_ls = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_ls, num_coords*sizeof(double) ) );

  double *d_ms = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_ms, num_coords*sizeof(double) ) );

  double *d_ns = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_ns, num_coords*sizeof(double) ) );


  double *d_ras = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_ras, num_coords*sizeof(double) ) );
  cudaErrorCheckCall( hipMemcpy(d_ras, ras,
                           num_coords*sizeof(double), hipMemcpyHostToDevice ) );

  double *d_decs = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_decs, num_coords*sizeof(double) ) );
  cudaErrorCheckCall( hipMemcpy(d_decs, decs,
                           num_coords*sizeof(double), hipMemcpyHostToDevice ) );

  dim3 grid, threads;

  threads.x = 128;
  grid.x = (int)ceil( (float)num_coords / (float)threads.x );

  cudaErrorCheckKernel("kern_calc_lmn",
          kern_calc_lmn, grid, threads,
          ra0, sin(dec0), cos(dec0),
          d_ras, d_decs, d_ls, d_ms, d_ns,
          num_coords);

  cudaErrorCheckCall( hipMemcpy(ls, d_ls,
                             num_coords*sizeof(double),hipMemcpyDeviceToHost) );
  cudaErrorCheckCall( hipMemcpy(ms, d_ms,
                             num_coords*sizeof(double),hipMemcpyDeviceToHost) );
  cudaErrorCheckCall( hipMemcpy(ns, d_ns,
                             num_coords*sizeof(double),hipMemcpyDeviceToHost) );

  cudaErrorCheckCall( hipFree(d_ls) );
  cudaErrorCheckCall( hipFree(d_ms) );
  cudaErrorCheckCall( hipFree(d_ns) );

  cudaErrorCheckCall( hipFree(d_ras) );
  cudaErrorCheckCall( hipFree(d_decs) );

}

extern "C" void test_kern_calc_uvw(double *X_diff,
   double *Y_diff, double *Z_diff,
   user_precision_t *u_metres, user_precision_t *v_metres, user_precision_t *w_metres,
   user_precision_t *us, user_precision_t *vs, user_precision_t *ws,
   user_precision_t *wavelengths,
   double dec0, double *cha0s, double *sha0s,
   int num_visis, int num_baselines) {

  double *d_X_diff = NULL;
  double *d_Y_diff = NULL;
  double *d_Z_diff = NULL;

  cudaErrorCheckCall( hipMalloc( (void**)&d_X_diff,
                                     num_baselines*sizeof(double) ) );
  cudaErrorCheckCall( hipMemcpy( d_X_diff, X_diff,
             num_baselines*sizeof(double), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_Y_diff,
                                     num_baselines*sizeof(double) ) );
  cudaErrorCheckCall( hipMemcpy( d_Y_diff, Y_diff,
             num_baselines*sizeof(double), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_Z_diff,
                                     num_baselines*sizeof(double) ) );
  cudaErrorCheckCall( hipMemcpy( d_Z_diff, Z_diff,
             num_baselines*sizeof(double), hipMemcpyHostToDevice ) );

  double *d_sha0s = NULL;
  double *d_cha0s = NULL;
  user_precision_t *d_wavelengths = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_sha0s,
                                         num_visis*sizeof(double) ) );
  cudaErrorCheckCall( hipMemcpy( d_sha0s, sha0s,
                 num_visis*sizeof(double), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_cha0s,
                                         num_visis*sizeof(double) ) );
  cudaErrorCheckCall( hipMemcpy( d_cha0s, cha0s,
                 num_visis*sizeof(double), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_wavelengths,
                                         num_visis*sizeof(user_precision_t) ) );
  cudaErrorCheckCall( hipMemcpy( d_wavelengths, wavelengths,
                 num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

  user_precision_t *d_u_metres = NULL;
  user_precision_t *d_v_metres = NULL;
  user_precision_t *d_w_metres = NULL;
  user_precision_t *d_us = NULL;
  user_precision_t *d_vs = NULL;
  user_precision_t *d_ws = NULL;

  cudaErrorCheckCall( hipMalloc( (void**)&d_u_metres, num_visis*sizeof(user_precision_t) ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_v_metres, num_visis*sizeof(user_precision_t) ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_w_metres, num_visis*sizeof(user_precision_t) ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_us, num_visis*sizeof(user_precision_t) ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_vs, num_visis*sizeof(user_precision_t) ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_ws, num_visis*sizeof(user_precision_t) ) );

  dim3 grid, threads;

  threads.x = 128;
  grid.x = (int)ceil( (float)num_visis / (float)threads.x );

  cudaErrorCheckKernel("kern_calc_uvw",
          kern_calc_uvw, grid, threads,
          d_X_diff, d_Y_diff, d_Z_diff,
          d_u_metres, d_v_metres, d_w_metres,
          d_us, d_vs, d_ws, d_wavelengths,
          sin(dec0), cos(dec0),
          d_cha0s, d_sha0s,
          num_visis, num_baselines);

  cudaErrorCheckCall( hipMemcpy(us, d_us,
                   num_visis*sizeof(user_precision_t),hipMemcpyDeviceToHost) );
  cudaErrorCheckCall( hipMemcpy(vs, d_vs,
                   num_visis*sizeof(user_precision_t),hipMemcpyDeviceToHost) );
  cudaErrorCheckCall( hipMemcpy(ws, d_ws,
                   num_visis*sizeof(user_precision_t),hipMemcpyDeviceToHost) );

  cudaErrorCheckCall( hipMemcpy(u_metres, d_u_metres,
                   num_visis*sizeof(user_precision_t),hipMemcpyDeviceToHost) );
  cudaErrorCheckCall( hipMemcpy(v_metres, d_v_metres,
                   num_visis*sizeof(user_precision_t),hipMemcpyDeviceToHost) );
  cudaErrorCheckCall( hipMemcpy(w_metres, d_w_metres,
                   num_visis*sizeof(user_precision_t),hipMemcpyDeviceToHost) );

  cudaErrorCheckCall( hipFree(d_us) );
  cudaErrorCheckCall( hipFree(d_vs) );
  cudaErrorCheckCall( hipFree(d_ws) );

  cudaErrorCheckCall( hipFree(d_u_metres) );
  cudaErrorCheckCall( hipFree(d_v_metres) );
  cudaErrorCheckCall( hipFree(d_w_metres) );

  cudaErrorCheckCall( hipFree(d_sha0s) );
  cudaErrorCheckCall( hipFree(d_cha0s) );

  cudaErrorCheckCall( hipFree(d_X_diff) );
  cudaErrorCheckCall( hipFree(d_Y_diff) );
  cudaErrorCheckCall( hipFree(d_Z_diff) );

}

extern "C" void test_kern_calc_uvw_shapelet(double *X_diff,
                     double *Y_diff, double *Z_diff,
                     user_precision_t *u_shapes, user_precision_t *v_shapes,
                     user_precision_t *w_shapes, user_precision_t *wavelengths,
                     double *lsts, double *ras, double *decs,
                     int num_baselines, int num_visis, int num_shapes) {

  double *d_X_diff = NULL;
  double *d_Y_diff = NULL;
  double *d_Z_diff = NULL;

  cudaErrorCheckCall( hipMalloc( (void**)&d_X_diff,
                                     num_baselines*sizeof(double) ) );
  cudaErrorCheckCall( hipMemcpy( d_X_diff, X_diff,
             num_baselines*sizeof(double), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_Y_diff,
                                     num_baselines*sizeof(double) ) );
  cudaErrorCheckCall( hipMemcpy( d_Y_diff, Y_diff,
             num_baselines*sizeof(double), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_Z_diff,
                                     num_baselines*sizeof(double) ) );
  cudaErrorCheckCall( hipMemcpy( d_Z_diff, Z_diff,
             num_baselines*sizeof(double), hipMemcpyHostToDevice ) );

  double *d_lsts = NULL;
  double *d_ras = NULL;
  double *d_decs = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_lsts, num_visis*sizeof(double) ) );
  cudaErrorCheckCall( hipMemcpy( d_lsts, lsts,
                      num_visis*sizeof(double), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_ras, num_visis*sizeof(double) ) );
  cudaErrorCheckCall( hipMemcpy( d_ras, ras,
                      num_visis*sizeof(double), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_decs, num_visis*sizeof(double) ) );
  cudaErrorCheckCall( hipMemcpy( d_decs, decs,
                      num_visis*sizeof(double), hipMemcpyHostToDevice ) );

  user_precision_t *d_wavelengths = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_wavelengths, num_visis*sizeof(user_precision_t) ) );
  cudaErrorCheckCall( hipMemcpy( d_wavelengths, wavelengths,
                      num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

  user_precision_t *d_u_shapes = NULL;
  user_precision_t *d_v_shapes = NULL;
  user_precision_t *d_w_shapes = NULL;

  cudaErrorCheckCall( hipMalloc( (void**)&d_u_shapes,
                              num_shapes*num_visis*sizeof(user_precision_t) ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_v_shapes,
                              num_shapes*num_visis*sizeof(user_precision_t) ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_w_shapes,
                              num_shapes*num_visis*sizeof(user_precision_t) ) );

  dim3 grid, threads;

  threads.x = 64;
  grid.x = (int)ceil( (float)num_visis / (float)threads.x );

  threads.y = 2;
  grid.y = (int)ceil( (float)num_shapes / (float)threads.y );

  cudaErrorCheckKernel("kern_calc_uvw_shapelet",
          kern_calc_uvw_shapelet, grid, threads,
          d_X_diff, d_Y_diff, d_Z_diff,
          d_u_shapes, d_v_shapes, d_w_shapes, d_wavelengths,
          d_lsts, d_ras, d_decs,
          num_baselines, num_visis, num_shapes);

  cudaErrorCheckCall( hipMemcpy(u_shapes, d_u_shapes,
         num_shapes*num_visis*sizeof(user_precision_t),hipMemcpyDeviceToHost) );
  cudaErrorCheckCall( hipMemcpy(v_shapes, d_v_shapes,
         num_shapes*num_visis*sizeof(user_precision_t),hipMemcpyDeviceToHost) );
  cudaErrorCheckCall( hipMemcpy(w_shapes, d_w_shapes,
         num_shapes*num_visis*sizeof(user_precision_t),hipMemcpyDeviceToHost) );

  cudaErrorCheckCall( hipFree(d_u_shapes) );
  cudaErrorCheckCall( hipFree(d_v_shapes) );
  cudaErrorCheckCall( hipFree(d_w_shapes) );

  cudaErrorCheckCall( hipFree(d_lsts) );
  cudaErrorCheckCall( hipFree(d_ras) );
  cudaErrorCheckCall( hipFree(d_decs) );

  cudaErrorCheckCall( hipFree(d_X_diff) );
  cudaErrorCheckCall( hipFree(d_Y_diff) );
  cudaErrorCheckCall( hipFree(d_Z_diff) );

}
