#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <complex.h>
#include <math.h>
#include "constants.h"

__device__ void calc_uvw(float *d_X_diff, float *d_Y_diff, float *d_Z_diff,
           float d_sdec0, float d_cdec0, float d_sha0, float d_cha0,
           int iBaseline, int num_baselines,
           float * u, float * v, float * w) {

  int mod_baseline = iBaseline - num_baselines*floorf((float)iBaseline / (float)num_baselines);

  * u = (d_sha0*d_X_diff[mod_baseline]) + (d_cha0*d_Y_diff[mod_baseline]);
  * v = (d_sdec0*d_sha0*d_Y_diff[mod_baseline]) + (d_cdec0*d_Z_diff[mod_baseline]) - (d_sdec0*d_cha0*d_X_diff[mod_baseline]);
  * w = (d_cdec0*d_cha0*d_X_diff[mod_baseline]) - (d_cdec0*d_sha0*d_Y_diff[mod_baseline]) + (d_sdec0*d_Z_diff[mod_baseline]);

}

__global__ void kern_calc_uvw(float *d_X_diff, float *d_Y_diff, float *d_Z_diff,
           float *d_u_metres, float *d_v_metres, float *d_w_metres,
           float *d_u, float *d_v, float *d_w, float *d_wavelengths,
           float sdec0, float cdec0,
           float *d_cha0s, float *d_sha0s,
           int num_visis, int num_baselines){
  // Start by computing which baseline we're going to do
  const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);

  if (iBaseline < num_visis){
    float u, v, w;

    float d_sha0 = d_sha0s[iBaseline];
    float d_cha0 = d_cha0s[iBaseline];

    calc_uvw(d_X_diff, d_Y_diff, d_Z_diff,
               sdec0, cdec0, d_sha0, d_cha0,
               iBaseline, num_baselines,
               &u, &v, &w);

    d_u_metres[iBaseline] = u;
    d_v_metres[iBaseline] = v;
    d_w_metres[iBaseline] = w;

    float d_wavelength = d_wavelengths[iBaseline];

    d_u[iBaseline] = u / d_wavelength;
    d_v[iBaseline] = v / d_wavelength;
    d_w[iBaseline] = w / d_wavelength;
  }
}

__global__ void kern_calc_uvw_shapelet(float *d_X_diff, float *d_Y_diff, float *d_Z_diff,
      float *d_u_s_metres, float *d_v_s_metres, float *d_w_s_metres,
      float *d_lsts, float *d_ras, float *d_decs,
      const int num_baselines, const int num_visis,
      const int num_shapes) {
  // Start by computing which baseline we're going to do
  const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);
  const int iComponent = threadIdx.y + (blockDim.y*blockIdx.y);

  if(iBaseline < num_visis && iComponent < num_shapes) {

    float u_s, v_s, w_s;
    //TODO do the sin/cos outside of the GPU kernel?
    float d_sdec0 = sinf(d_decs[iComponent]);
    float d_cdec0 = cosf(d_decs[iComponent]);
    float d_sha0 = sinf(d_lsts[iBaseline] - d_ras[iComponent]);
    float d_cha0 = cosf(d_lsts[iBaseline] - d_ras[iComponent]);

    calc_uvw(d_X_diff, d_Y_diff, d_Z_diff,
               d_sdec0, d_cdec0, d_sha0, d_cha0,
               iBaseline, num_baselines,
               &u_s, &v_s, &w_s);

    d_u_s_metres[num_visis*iComponent + iBaseline] = u_s;
    d_v_s_metres[num_visis*iComponent + iBaseline] = v_s;
    d_w_s_metres[num_visis*iComponent + iBaseline] = w_s;
  }
}

__device__ void calc_lmn(float ra0, float sdec0, float cdec0,
                         float d_ra, float d_dec,
                         float * l, float * m, float * n){
  // float d_sdec0 = d_angles_array[0];
  // float d_cdec0 = d_angles_array[1];
  // float d_ra0 = d_angles_array[2];

  float cdec;
  float sdec;
  float cdra;
  float sdra;

  cdec = cosf(d_dec);
  sdec = sinf(d_dec);
  cdra = cosf((d_ra - ra0));
  sdra = sinf((d_ra - ra0));

  * l = cdec*sdra;
  * m = sdec*cdec0 - cdec*sdec0*cdra;
  * n = sdec*sdec0 + cdec*cdec0*cdra;

}

__global__ void kern_calc_lmn(float ra0, float sdec0, float cdec0,
                              float *d_ras, float *d_decs,
                              float *d_l, float *d_m, float *d_n,
                              int num_components){

  const int iComponent = threadIdx.x + (blockDim.x*blockIdx.x);

  if (iComponent < num_components){
    float l, m, n;

    calc_lmn(ra0, sdec0, cdec0,
             d_ras[iComponent], d_decs[iComponent],
             &l, &m, &n);

    d_l[iComponent] = l;
    d_m[iComponent] = m;
    d_n[iComponent] = n;

  }
}
