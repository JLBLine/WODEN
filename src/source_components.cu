#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <complex.h>
#include <math.h>
#include "cudacomplex.h"
#include "fundamental_coords.h"
#include "constants.h"
#include "shapelet_basis.h"
#include "read_and_write.h"
#include "source_components.h"

__device__ void extrap_flux(float *d_wavelengths, float *d_freqs,
           float *d_fluxes, int iComponent, int iBaseline,
           float * extrap_flux){

  float d_wavelength = d_wavelengths[iBaseline];
  float cat_wavelength = VELC / d_freqs[iComponent];
  * extrap_flux = d_fluxes[iComponent] * powf(cat_wavelength / d_wavelength,DEFAULT_SI);
}

__device__ void extrap_stokes(float *d_wavelengths, float *d_ref_freqs,
           float *d_ref_stokesI, float *d_ref_stokesQ,
           float *d_ref_stokesU, float *d_ref_stokesV,
           float *d_SIs, int iComponent, int iBaseline,
           float * flux_I, float * flux_Q, float * flux_U, float * flux_V){

  float d_freq = VELC / d_wavelengths[iBaseline];
  float d_ref_freq = d_ref_freqs[iComponent];

  float flux_ratio = powf(d_freq / d_ref_freq, d_SIs[iComponent]);

  * flux_I = d_ref_stokesI[iComponent] * flux_ratio;
  * flux_Q = d_ref_stokesQ[iComponent] * flux_ratio;
  * flux_U = d_ref_stokesU[iComponent] * flux_ratio;
  * flux_V = d_ref_stokesV[iComponent] * flux_ratio;

  if (iBaseline == 0) {
    // printf("Extrap fluxes %.5f %.5f %.5f %.5f\n", d_freq, d_ref_freq, d_SIs[iComponent], flux_ratio);
    // printf("Extrap fluxes %d %.5f %.5f %.5f %.5f\n",iComponent,* flux_I,* flux_Q,* flux_U,* flux_V );
  }
}

__global__ void kern_extrap_stokes(int num_visis, int num_components,
           float *d_wavelengths, float *d_ref_freqs, float *d_SIs,
           float *d_ref_stokesI, float *d_ref_stokesQ,
           float *d_ref_stokesU, float *d_ref_stokesV,
           float *d_flux_I, float *d_flux_Q,
           float *d_flux_U, float *d_flux_V ) {

  // Start by computing which baseline we're going to do
  const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);
  const int iComponent = threadIdx.y + (blockDim.y*blockIdx.y);

  if(iBaseline < num_visis && iComponent < num_components) {
    // printf("Here 2? %d %d\n",iBaseline,iComponent );

    float flux_I, flux_Q, flux_U, flux_V;

    extrap_stokes(d_wavelengths, d_ref_freqs,
                 d_ref_stokesI, d_ref_stokesQ,
                 d_ref_stokesU, d_ref_stokesV,
                 d_SIs, iComponent, iBaseline,
                 &flux_I, &flux_Q, &flux_U, &flux_V);

      d_flux_I[iComponent] = flux_I;
      d_flux_Q[iComponent] = flux_Q;
      d_flux_U[iComponent] = flux_U;
      d_flux_V[iComponent] = flux_V;

  }
}

extern "C" void test_extrap_flux(catsource_t *catsource,
           int num_visis, int num_components,
           float *wavelengths, float *flux_I, float *flux_Q,
           float *flux_U, float *flux_V ){

  float *d_wavelengths = NULL;
  float *d_ref_freqs = NULL;
  float *d_SIs = NULL;
  float *d_ref_stokesI = NULL;
  float *d_ref_stokesQ = NULL;
  float *d_ref_stokesU = NULL;
  float *d_ref_stokesV = NULL;
  float *d_flux_I = NULL;
  float *d_flux_Q = NULL;
  float *d_flux_U = NULL;
  float *d_flux_V = NULL;

  hipMalloc( (void**)&d_wavelengths, num_visis*sizeof(float));
  hipMalloc( (void**)&d_ref_freqs, num_components*sizeof(float));
  hipMalloc( (void**)&d_SIs, num_components*sizeof(float));
  hipMalloc( (void**)&d_ref_stokesI, num_components*sizeof(float));
  hipMalloc( (void**)&d_ref_stokesQ, num_components*sizeof(float));
  hipMalloc( (void**)&d_ref_stokesU, num_components*sizeof(float));
  hipMalloc( (void**)&d_ref_stokesV, num_components*sizeof(float));
  hipMalloc( (void**)&d_flux_I, num_components*sizeof(float));
  hipMalloc( (void**)&d_flux_Q, num_components*sizeof(float));
  hipMalloc( (void**)&d_flux_U, num_components*sizeof(float));
  hipMalloc( (void**)&d_flux_V, num_components*sizeof(float));

  // printf("CUDA error 1: %s\n", hipGetErrorString( hipGetLastError() ) );
  hipMemcpy(d_wavelengths, wavelengths, num_visis*sizeof(float), hipMemcpyHostToDevice );

  // printf("CUDA error 2: %s\n", hipGetErrorString( hipGetLastError() ) );
  hipMemcpy(d_ref_stokesI, catsource->point_ref_stokesI, num_components*sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy(d_ref_stokesQ, catsource->point_ref_stokesQ, num_components*sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy(d_ref_stokesU, catsource->point_ref_stokesU, num_components*sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy(d_ref_stokesV, catsource->point_ref_stokesV, num_components*sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy(d_ref_freqs, catsource->point_ref_freqs, num_components*sizeof(float), hipMemcpyHostToDevice );
  hipMemcpy(d_SIs, catsource->point_SIs, num_components*sizeof(float), hipMemcpyHostToDevice );

  // float *zero_array = NULL;
  // // hipHostMalloc( (void**)&zero_array, num_time_steps*num_components*MAX_POLS*sizeof(float _Complex) );
  // zero_array = (float *)malloc( num_components*sizeof(float) );
  //
  // for (int i = 0; i < num_components; i++) {
  //   zero_array[i] = 0.0;
  // }
  //
  // hipMemcpy(d_ref_stokesQ, zero_array, num_components*sizeof(float), hipMemcpyHostToDevice );
  // hipMemcpy(d_ref_stokesU, zero_array, num_components*sizeof(float), hipMemcpyHostToDevice );
  // hipMemcpy(d_ref_stokesV, zero_array, num_components*sizeof(float), hipMemcpyHostToDevice );
  //
  // free(zero_array);

  // printf("Here 4?\n");

  dim3 grid, threads;

  threads.x = 64;
  threads.y = 2;
  grid.x = (int)ceil( (float)num_visis / (float)threads.x );
  grid.y = (int)ceil( (float)num_components / (float)threads.y );

  kern_extrap_stokes<<< grid, threads >>>(num_visis, num_components,
                     d_wavelengths, d_ref_freqs, d_SIs,
                     d_ref_stokesI, d_ref_stokesQ,
                     d_ref_stokesU, d_ref_stokesV,
                     d_flux_I, d_flux_Q,
                     d_flux_U, d_flux_V);

  hipMemcpy(flux_I, d_flux_I, num_components*sizeof(float),hipMemcpyDeviceToHost);
  hipMemcpy(flux_Q, d_flux_Q, num_components*sizeof(float),hipMemcpyDeviceToHost);
  hipMemcpy(flux_U, d_flux_U, num_components*sizeof(float),hipMemcpyDeviceToHost);
  hipMemcpy(flux_V, d_flux_V, num_components*sizeof(float),hipMemcpyDeviceToHost);

  hipFree( d_wavelengths );
  hipFree( d_ref_freqs );
  hipFree( d_SIs );
  hipFree( d_ref_stokesI );
  hipFree( d_ref_stokesQ );
  hipFree( d_ref_stokesU );
  hipFree( d_ref_stokesV );
  hipFree( d_flux_I );
  hipFree( d_flux_Q );
  hipFree( d_flux_U );
  hipFree( d_flux_V );


}

__device__ hipFloatComplex calc_measurement_equation(float *d_us,
           float *d_vs, float *d_ws, float *d_ls, float *d_ms, float *d_ns,
           const int iBaseline, const int iComponent){

  float u, v, w;
  float l, m, n;

  u = d_us[iBaseline];
  v = d_vs[iBaseline];
  w = d_ws[iBaseline];

  l = d_ls[iComponent];
  m = d_ms[iComponent];
  n = d_ns[iComponent];

  hipFloatComplex visi;

  //Not sure why, but get exact match with oskar sims and correct location
  //on sky through wsclean without negative infront on 2pi
  float temp = 2*M_PI*( u*l + v*m + w*(n-1) );
  sincosf(temp, &(visi.y), &(visi.x));

  return visi;
}

__device__ void apply_beam_gains(hipFloatComplex g1xx, hipFloatComplex g1xy,
          hipFloatComplex g1yx, hipFloatComplex g1yy,
          hipFloatComplex g2xx, hipFloatComplex g2xy,
          hipFloatComplex g2yx, hipFloatComplex g2yy,
          float flux_I, float flux_Q,
          float flux_U, float flux_V,
          hipFloatComplex visi,
          hipFloatComplex * visi_XX, hipFloatComplex * visi_XY,
          hipFloatComplex * visi_YX, hipFloatComplex * visi_YY,
          int beamtype ) {

  //Conjugate the second beam gains
  hipFloatComplex g2xx_conj = make_hipFloatComplex(g2xx.x,-g2xx.y);
  hipFloatComplex g2xy_conj = make_hipFloatComplex(g2xy.x,-g2xy.y);
  hipFloatComplex g2yx_conj = make_hipFloatComplex(g2yx.x,-g2yx.y);
  hipFloatComplex g2yy_conj = make_hipFloatComplex(g2yy.x,-g2yy.y);

  //Create the Stokes visibilities
  hipFloatComplex visi_I = hipCmulf(make_hipComplex(flux_I,0.0),visi );
  hipFloatComplex visi_Q = hipCmulf(make_hipComplex(flux_Q,0.0),visi );
  hipFloatComplex visi_U = hipCmulf(make_hipComplex(flux_U,0.0),visi );
  hipFloatComplex visi_V = hipCmulf(make_hipComplex(flux_V,0.0),visi );

  hipFloatComplex this_XX;
  hipFloatComplex this_XY;
  hipFloatComplex this_YX;
  hipFloatComplex this_YY;

  //Convert the Stokes into instrumental visibilities
  this_XX = hipCmulf(hipCmulf(g1xx,g2xx_conj) + hipCmulf(g1xy,g2xy_conj),visi_I);
  this_XX += hipCmulf(hipCmulf(g1xx,g2xx_conj) - hipCmulf(g1xy,g2xy_conj),visi_Q);
  this_XX += hipCmulf(hipCmulf(g1xx,g2xy_conj) + hipCmulf(g1xy,g2xx_conj),visi_U);
  this_XX += hipCmulf(hipCmulf(make_hipFloatComplex(0.0,1.0),visi_V), hipCmulf(g1xx,g2xy_conj) - hipCmulf(g1xy,g2xx_conj) );

  this_XY = hipCmulf(hipCmulf(g1xx,g2yx_conj) + hipCmulf(g1xy,g2yy_conj),visi_I);
  this_XY += hipCmulf(hipCmulf(g1xx,g2yx_conj) - hipCmulf(g1xy,g2yy_conj),visi_Q);
  this_XY += hipCmulf(hipCmulf(g1xx,g2yy_conj) + hipCmulf(g1xy,g2yx_conj),visi_U);
  this_XY += hipCmulf(hipCmulf(make_hipFloatComplex(0.0,1.0),visi_V), hipCmulf(g1xx,g2yy_conj) - hipCmulf(g1xy,g2yx_conj) );

  this_YX = hipCmulf(hipCmulf(g1yx,g2xx_conj) + hipCmulf(g1yy,g2xy_conj),visi_I);
  this_YX += hipCmulf(hipCmulf(g1yx,g2xx_conj) - hipCmulf(g1yy,g2xy_conj),visi_Q);
  this_YX += hipCmulf(hipCmulf(g1yx,g2xy_conj) + hipCmulf(g1yy,g2xx_conj),visi_U);
  this_YX += hipCmulf(hipCmulf(make_hipFloatComplex(0.0,1.0),visi_V), hipCmulf(g1yx,g2xy_conj) - hipCmulf(g1yy,g2xx_conj) );

  this_YY = hipCmulf(hipCmulf(g1yx,g2yx_conj) + hipCmulf(g1yy,g2yy_conj),visi_I);
  this_YY += hipCmulf(hipCmulf(g1yx,g2yx_conj) - hipCmulf(g1yy,g2yy_conj),visi_Q);
  this_YY += hipCmulf(hipCmulf(g1yx,g2yy_conj) + hipCmulf(g1yy,g2yx_conj),visi_U);
  this_YY += hipCmulf(hipCmulf(make_hipFloatComplex(0.0,1.0),visi_V), hipCmulf(g1yx,g2yy_conj) - hipCmulf(g1yy,g2yx_conj) );

  // if (beamtype == FEE_BEAM) {
  //   * visi_XX = this_YY;
  //   * visi_XY = this_YX;
  //   * visi_YX = this_XY;
  //   * visi_YY = this_XX;
  // }
  //
  // else {
  * visi_XX = this_XX;
  * visi_XY = this_XY;
  * visi_YX = this_YX;
  * visi_YY = this_YY;
  // }


}

__device__ void get_beam_gains(int iBaseline, int iComponent, int num_freqs,
           int num_baselines, int num_components, int num_times, int beamtype,
           float *d_gauss_beam_reals, float *d_gauss_beam_imags,
           hipFloatComplex *d_analy_beam_X, hipFloatComplex *d_analy_beam_Y,
           hipFloatComplex *d_FEE_beam_gain_matrices,
           hipFloatComplex * g1xx, hipFloatComplex * g1xy,
           hipFloatComplex * g1yx, hipFloatComplex * g1yy,
           hipFloatComplex * g2xx, hipFloatComplex * g2xy,
           hipFloatComplex * g2yx, hipFloatComplex * g2yy){

  int beam_ind = 0;
  int time_ind = 0;
  int freq_ind = 0;

  if (beamtype == GAUSS_BEAM) {
    //Do some epic indexing to work out which beam value
    time_ind = (int)floorf( (float)iBaseline / ((float)num_baselines * (float)num_freqs));
    freq_ind = (int)floorf( ((float)iBaseline - ((float)time_ind*(float)num_baselines * (float)num_freqs)) / (float)num_baselines);
    beam_ind = num_freqs*time_ind*num_components + (num_components*freq_ind) + iComponent;

    hipFloatComplex gauss_beam_complex = make_hipFloatComplex(d_gauss_beam_reals[beam_ind],d_gauss_beam_imags[beam_ind]);

    * g1xx = gauss_beam_complex;
    * g2xx = gauss_beam_complex;
    * g1yy = gauss_beam_complex;
    * g2yy = gauss_beam_complex;
    * g1xy = make_hipComplex(0.0, 0.0);
    * g2xy = make_hipComplex(0.0, 0.0);
    * g1yx = make_hipComplex(0.0, 0.0);
    * g2yx = make_hipComplex(0.0, 0.0);

    // printf("%d %d %d %d %d %d %f %f\n",iBaseline,num_baselines,num_freqs,time_ind,freq_ind,beam_ind,beam_real,beam_imag);
  }

  else if (beamtype == FEE_BEAM) {
    // printf("Like, here? %d\n", iBaseline);
    //

    time_ind = (int)floorf( (float)iBaseline / ((float)num_baselines * (float)num_freqs));
    beam_ind = iComponent*num_times + time_ind;

    * g1xx = d_FEE_beam_gain_matrices[beam_ind*MAX_POLS + 0];
    * g1xy = d_FEE_beam_gain_matrices[beam_ind*MAX_POLS + 1];
    * g1yx = d_FEE_beam_gain_matrices[beam_ind*MAX_POLS + 2];
    * g1yy = d_FEE_beam_gain_matrices[beam_ind*MAX_POLS + 3];

    * g2xx = d_FEE_beam_gain_matrices[beam_ind*MAX_POLS + 0];
    * g2xy = d_FEE_beam_gain_matrices[beam_ind*MAX_POLS + 1];
    * g2yx = d_FEE_beam_gain_matrices[beam_ind*MAX_POLS + 2];
    * g2yy = d_FEE_beam_gain_matrices[beam_ind*MAX_POLS + 3];

    // hipFloatComplex thing1 = * g1xx;
    // hipFloatComplex thing2 = * g1xy;
    // hipFloatComplex thing3 = * g1yx;
    // hipFloatComplex thing4 = * g1yy;
    //
    // if (iBaseline == 0) {
    //   if (thing1.x > 2.0 || thing2.x > 2.0 || thing3.x > 2.0 || thing4.x > 2.0) {
    //     printf("%d %.5f %.5f %.5f %.5f\n",iComponent,thing1.x,thing2.x,thing3.x,thing4.x );
    //   }
    // }

  }

  else if (beamtype == ANALY_DIPOLE) {

    time_ind = (int)floorf( (float)iBaseline / ((float)num_baselines * (float)num_freqs));
    freq_ind = (int)floorf( ((float)iBaseline - ((float)time_ind*(float)num_baselines * (float)num_freqs)) / (float)num_baselines);
    beam_ind = num_freqs*time_ind*num_components + (num_components*freq_ind) + iComponent;

    * g1xx = d_analy_beam_X[beam_ind];
    * g2xx = d_analy_beam_X[beam_ind];
    * g1yy = d_analy_beam_Y[beam_ind];
    * g2yy = d_analy_beam_Y[beam_ind];

    * g1xy = make_hipComplex(0.0, 0.0);
    * g2xy = make_hipComplex(0.0, 0.0);
    * g1yx = make_hipComplex(0.0, 0.0);
    * g2yx = make_hipComplex(0.0, 0.0);

    hipFloatComplex thing1 = * g1xx;
    hipFloatComplex thing2 = * g1xy;
    hipFloatComplex thing3 = * g1yx;
    hipFloatComplex thing4 = * g1yy;

    // if (iBaseline == 0) {
    //   if (thing1.x > 2.0 || thing2.x > 2.0 || thing3.x > 2.0 || thing4.x > 2.0 || thing1.y > 2.0 || thing2.y > 2.0 || thing3.y > 2.0 || thing4.y > 2.0) {
    //         printf("%d %.5f %.5f %.5f %.5f\n",iComponent,thing1.x,thing2.x,thing3.x,thing4.x );
    //   }
    // }

  }

  else {
    * g1xx = make_hipComplex(1.0, 0.0);
    * g2xx = make_hipComplex(1.0, 0.0);
    * g1yy = make_hipComplex(1.0, 0.0);
    * g2yy = make_hipComplex(1.0, 0.0);
    * g1xy = make_hipComplex(0.0, 0.0);
    * g2xy = make_hipComplex(0.0, 0.0);
    * g1yx = make_hipComplex(0.0, 0.0);
    * g2yx = make_hipComplex(0.0, 0.0);
  }

} //end __device__ get_beam_gains

__device__ void update_sum_visis(int iBaseline, int iComponent, int num_freqs,
           int num_baselines, int num_components, int num_times, int beamtype,
           float *d_gauss_beam_reals, float *d_gauss_beam_imags,
           hipFloatComplex *d_FEE_beam_gain_matrices,
           hipFloatComplex *d_analy_beam_X, hipFloatComplex *d_analy_beam_Y,
           hipFloatComplex visi,
           float flux_I, float flux_Q, float flux_U, float flux_V,
           float *d_sum_visi_XX_real, float *d_sum_visi_XX_imag,
           float *d_sum_visi_XY_real, float *d_sum_visi_XY_imag,
           float *d_sum_visi_YX_real, float *d_sum_visi_YX_imag,
           float *d_sum_visi_YY_real, float *d_sum_visi_YY_imag){

    hipFloatComplex g1xx;
    hipFloatComplex g1xy;
    hipFloatComplex g1yx;
    hipFloatComplex g1yy;
    hipFloatComplex g2xx;
    hipFloatComplex g2xy;
    hipFloatComplex g2yx;
    hipFloatComplex g2yy;

    get_beam_gains(iBaseline, iComponent, num_freqs,
               num_baselines, num_components, num_times, beamtype,
               d_gauss_beam_reals, d_gauss_beam_imags,
               d_analy_beam_X, d_analy_beam_Y,
               d_FEE_beam_gain_matrices,
               &g1xx, &g1xy, &g1yx, &g1yy, &g2xx, &g2xy, &g2yx, &g2yy);

    hipFloatComplex visi_XX;
    hipFloatComplex visi_XY;
    hipFloatComplex visi_YX;
    hipFloatComplex visi_YY;

    apply_beam_gains(g1xx, g1xy, g1yx, g1yy, g2xx, g2xy, g2yx, g2yy,
                    flux_I, flux_Q, flux_U, flux_V,
                    visi, &visi_XX, &visi_XY, &visi_YX, &visi_YY, beamtype );

    atomicAdd(&d_sum_visi_XX_real[iBaseline],visi_XX.x);
    atomicAdd(&d_sum_visi_XX_imag[iBaseline],visi_XX.y);

    atomicAdd(&d_sum_visi_XY_real[iBaseline],visi_XY.x);
    atomicAdd(&d_sum_visi_XY_imag[iBaseline],visi_XY.y);

    atomicAdd(&d_sum_visi_YX_real[iBaseline],visi_YX.x);
    atomicAdd(&d_sum_visi_YX_imag[iBaseline],visi_YX.y);

    atomicAdd(&d_sum_visi_YY_real[iBaseline],visi_YY.x);
    atomicAdd(&d_sum_visi_YY_imag[iBaseline],visi_YY.y);

}

__global__ void kern_calc_visi_point(float *d_point_ras, float *d_point_decs,
           float *d_point_freqs, float *d_point_stokesI, float *d_point_stokesQ,
           float *d_point_stokesU, float *d_point_stokesV, float *d_point_SIs,
           float *d_us, float *d_vs, float *d_ws,
           float *d_sum_visi_XX_real, float *d_sum_visi_XX_imag,
           float *d_sum_visi_XY_real, float *d_sum_visi_XY_imag,
           float *d_sum_visi_YX_real, float *d_sum_visi_YX_imag,
           float *d_sum_visi_YY_real, float *d_sum_visi_YY_imag,
           float *d_angles_array, float *d_wavelengths,
           float *d_ls, float *d_ms, float *d_ns,
           int num_points, int num_baselines, int num_freqs, int num_visis,
           int num_times,
           float *d_gauss_beam_reals, float *d_gauss_beam_imags, int beamtype,
           hipFloatComplex *d_FEE_beam_gain_matrices,
           hipFloatComplex *d_analy_beam_X, hipFloatComplex *d_analy_beam_Y) {

  // Start by computing which baseline we're going to do
  const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);
  const int iComponent = threadIdx.y + (blockDim.y*blockIdx.y);

  if(iBaseline < num_visis && iComponent < num_points) {
    // float point_flux_I;
    // extrap_flux(d_wavelengths, d_point_freqs, d_point_fluxes,
    //               iComponent, iBaseline, &point_flux_I);
    //
    // float point_flux_Q = 0.0;
    // float point_flux_U = 0.0;
    // float point_flux_V = 0.0;

    float point_flux_I;
    float point_flux_Q;
    float point_flux_U;
    float point_flux_V;

    extrap_stokes(d_wavelengths, d_point_freqs,
                 d_point_stokesI, d_point_stokesQ,
                 d_point_stokesU, d_point_stokesV,
                 d_point_SIs, iComponent, iBaseline,
                 &point_flux_I, &point_flux_Q, &point_flux_U, &point_flux_V);

    // if (iBaseline == 0) {
    //   printf("FLUXES %.5f %.5f %.5f %.5f\n", point_flux_I, point_flux_Q, point_flux_U, point_flux_V);
    // }



    hipFloatComplex visi;
    visi = calc_measurement_equation(d_us, d_vs, d_ws,
                           d_ls, d_ms, d_ns,
                           iBaseline, iComponent);

    update_sum_visis(iBaseline, iComponent, num_freqs,
           num_baselines, num_points, num_times, beamtype,
           d_gauss_beam_reals, d_gauss_beam_imags,
           d_FEE_beam_gain_matrices,
           d_analy_beam_X, d_analy_beam_Y,
           visi,
           point_flux_I, point_flux_Q, point_flux_U, point_flux_V,
           d_sum_visi_XX_real, d_sum_visi_XX_imag,
           d_sum_visi_XY_real, d_sum_visi_XY_imag,
           d_sum_visi_YX_real, d_sum_visi_YX_imag,
           d_sum_visi_YY_real, d_sum_visi_YY_imag);

  }
}

__global__ void kern_calc_visi_gaussian(float *d_gauss_ras, float *d_gauss_decs,
           float *d_gauss_freqs, float *d_gauss_stokesI, float *d_gauss_stokesQ,
           float *d_gauss_stokesU, float *d_gauss_stokesV, float *d_gauss_SIs,
           float *d_us, float *d_vs, float *d_ws,
           float *d_sum_visi_XX_real, float *d_sum_visi_XX_imag,
           float *d_sum_visi_XY_real, float *d_sum_visi_XY_imag,
           float *d_sum_visi_YX_real, float *d_sum_visi_YX_imag,
           float *d_sum_visi_YY_real, float *d_sum_visi_YY_imag,
           float *d_angles_array, float *d_wavelengths,
           float *d_ls, float *d_ms, float *d_ns,
           float *d_gauss_pas, float *d_gauss_majors, float *d_gauss_minors,
           int num_gauss, int num_baselines, int num_freqs, int num_visis,
           int num_times,
           float *d_gauss_beam_reals, float *d_gauss_beam_imags, int beamtype,
           hipFloatComplex *d_FEE_beam_gain_matrices) {

  // Start by computing which baseline we're going to do
  const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);
  const int iComponent = threadIdx.y + (blockDim.y*blockIdx.y);

  if(iBaseline < num_visis && iComponent < num_gauss) {

    float gauss_flux_I;
    float gauss_flux_Q;
    float gauss_flux_U;
    float gauss_flux_V;

    extrap_stokes(d_wavelengths, d_gauss_freqs,
                 d_gauss_stokesI, d_gauss_stokesQ,
                 d_gauss_stokesU, d_gauss_stokesV,
                 d_gauss_SIs, iComponent, iBaseline,
                 &gauss_flux_I, &gauss_flux_Q, &gauss_flux_U, &gauss_flux_V);

    hipFloatComplex visi;
    visi = calc_measurement_equation(d_us, d_vs, d_ws,
                           d_ls, d_ms, d_ns,
                           iBaseline, iComponent);

    hipFloatComplex V_envelop = make_hipFloatComplex( 1.0, 0.0 );

    float pa = d_gauss_pas[iComponent];
    float u = d_us[iBaseline];
    float v = d_vs[iBaseline];
    float sinpa = sin(pa);
    float cospa = cos(pa);

    float x =  cospa*v + sinpa*u; // major axis
    float y = -sinpa*v + cospa*u; // minor axis
    float invsig_x = d_gauss_majors[iComponent];
    float invsig_y = d_gauss_minors[iComponent];

    V_envelop = make_hipFloatComplex( exp( -0.5 * ( x*x*invsig_x*invsig_x*M_PI_2_2_LN_2 + y*y*invsig_y*invsig_y*M_PI_2_2_LN_2 ) ), 0.0 );

    visi = hipCmulf(visi, V_envelop);

    hipFloatComplex *d_analy_beam_X = NULL;
    hipFloatComplex *d_analy_beam_Y = NULL;

    update_sum_visis(iBaseline, iComponent, num_freqs,
           num_baselines, num_gauss, num_times, beamtype,
           d_gauss_beam_reals, d_gauss_beam_imags,
           d_FEE_beam_gain_matrices,
           d_analy_beam_X, d_analy_beam_Y,
           visi,
           gauss_flux_I, gauss_flux_Q, gauss_flux_U, gauss_flux_V,
           d_sum_visi_XX_real, d_sum_visi_XX_imag,
           d_sum_visi_XY_real, d_sum_visi_XY_imag,
           d_sum_visi_YX_real, d_sum_visi_YX_imag,
           d_sum_visi_YY_real, d_sum_visi_YY_imag);

  }
}

__global__ void kern_calc_visi_shapelets(float *d_shape_ras,
      float *d_shape_decs, float *d_shape_fluxes, float *d_shape_freqs,
      float *d_us, float *d_vs, float *d_ws,
      float *d_wavelengths,
      float *d_u_s_metres, float *d_v_s_metres, float *d_w_s_metres,
      float *d_sum_visi_XX_real, float *d_sum_visi_XX_imag,
      float *d_sum_visi_XY_real, float *d_sum_visi_XY_imag,
      float *d_sum_visi_YX_real, float *d_sum_visi_YX_imag,
      float *d_sum_visi_YY_real, float *d_sum_visi_YY_imag,
      float *d_angles_array, float *d_shape_pas, float *d_shape_majors,
      float *d_shape_minors,
      float *d_shape_n1s, float *d_shape_n2s, float *d_shape_coeffs,
      float *d_shape_param_indexes,
      float *d_shape_ls, float *d_shape_ms, float *d_shape_ns,
      float *d_sbf,
      int num_shapes, int num_baselines, int num_freqs, int num_visis,
      const int num_coeffs, int num_times,
      float *d_gauss_beam_reals, float *d_gauss_beam_imags, int beamtype,
      hipFloatComplex *d_FEE_beam_gain_matrices) {

  // Start by computing which baseline we're going to do
  const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);
  const int iCoeff = threadIdx.y + (blockDim.y*blockIdx.y);


  if (iBaseline < num_visis && iCoeff < num_coeffs) {
    // printf("Made it here all g %d %d %d %d\n", iBaseline,num_visis,iCoeff,num_coeffs);
    int iComponent = d_shape_param_indexes[iCoeff];

    float shape_flux_I;
    //Use param index below and not iCoeff as there
    extrap_flux(d_wavelengths, d_shape_freqs,
               d_shape_fluxes, iComponent, iBaseline,
               &shape_flux_I);

    float shape_flux_Q = 0.0;
    float shape_flux_U = 0.0;
    float shape_flux_V = 0.0;

    hipFloatComplex visi;
    visi = calc_measurement_equation(d_us, d_vs, d_ws,
                          d_shape_ls, d_shape_ms, d_shape_ns,
                          iBaseline, iComponent);

    float pa = d_shape_pas[iComponent];
    float sinpa = sin(pa);
    float cospa = cos(pa);

    float d_wavelength = d_wavelengths[iBaseline];

    float u_s = d_u_s_metres[iComponent*num_visis + iBaseline] / d_wavelength;
    float v_s = d_v_s_metres[iComponent*num_visis + iBaseline] / d_wavelength;
    //
    float x = (cospa*v_s + sinpa*u_s); // major axis
    float y = (-sinpa*v_s + cospa*u_s); // minor axis

    //Scales the FWHM to std to match basis functions, and account for the
    //basis functions being stored with beta = 1.0
    //Basis functions have been stored in such a way that x is in the same
    //direction as on sky, but y is opposite, so include negative here
    float const_x = (d_shape_majors[iComponent]*SQRT_M_PI_2_2_LN_2)/sbf_dx;
    float const_y = -(d_shape_minors[iComponent]*SQRT_M_PI_2_2_LN_2)/sbf_dx;

    // I^(n1+n2) = Ipow_lookup[(n1+n2) % 4]
    hipFloatComplex Ipow_lookup[] = { make_hipFloatComplex(  1.0,  0.0 ),
                                     make_hipFloatComplex(  0.0,  1.0 ),
                                     make_hipFloatComplex( -1.0,  0.0 ),
                                     make_hipFloatComplex(  0.0, -1.0 ) };

    float xlow, xhigh, ylow, yhigh, u_value, v_value, f_hat, *sbf_n;

    // find the indices in the basis functions for u*beta_u and v*beta_v

    float xpos = x*const_x + sbf_c;
    float ypos = y*const_y + sbf_c;

    int xindex = (int)floor(xpos);
    int yindex = (int)floor(ypos);
    //
    int n1 = (int)d_shape_n1s[iCoeff];
    int n2 = (int)d_shape_n2s[iCoeff];

    // if ( n1 < 0 || n2 < 0 || n1 >= sbf_N || n2 >= sbf_N ) continue;

    f_hat = d_shape_coeffs[iCoeff];
    //
    sbf_n = &d_sbf[n1*sbf_L];
    xlow  = sbf_n[xindex];
    xhigh = sbf_n[xindex+1];
    u_value = xlow + (xhigh-xlow)*(xpos-xindex);

    sbf_n = &d_sbf[n2*sbf_L];
    ylow  = sbf_n[yindex];
    yhigh = sbf_n[yindex+1];
    v_value = ylow + (yhigh-ylow)*(ypos-yindex);

    // accumulate the intensity model for baseline pair (u,v)
    hipFloatComplex V_envelop = make_hipFloatComplex( 0.0, 0.0 );
    V_envelop = V_envelop + Ipow_lookup[(n1+n2) % 4] * f_hat * u_value*v_value;

    visi = hipCmulf(visi, V_envelop);

    hipFloatComplex *d_analy_beam_X = NULL;
    hipFloatComplex *d_analy_beam_Y = NULL;

    update_sum_visis(iBaseline, iComponent, num_freqs,
           num_baselines, num_shapes, num_times, beamtype,
           d_gauss_beam_reals, d_gauss_beam_imags,
           d_FEE_beam_gain_matrices,
           d_analy_beam_X, d_analy_beam_Y,
           visi,
           shape_flux_I, shape_flux_Q, shape_flux_U, shape_flux_V,
           d_sum_visi_XX_real, d_sum_visi_XX_imag,
           d_sum_visi_XY_real, d_sum_visi_XY_imag,
           d_sum_visi_YX_real, d_sum_visi_YX_imag,
           d_sum_visi_YY_real, d_sum_visi_YY_imag);

  }

}
