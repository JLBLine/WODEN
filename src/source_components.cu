#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <complex.h>
#include <math.h>
#include "cudacomplex.h"
#include "fundamental_coords.h"
#include "constants.h"
#include "shapelet_basis.h"
#include "source_components.h"
#include "cudacheck.h"
#include "woden_struct_defs.h"
#include "primary_beam_cuda.h"
#include "woden_precision_defs.h"

#include "chunk_sky_model.h"

__device__ void extrap_stokes(user_precision_t *d_allsteps_wavelengths,
           double *d_power_ref_freqs,
           user_precision_t *d_power_ref_stokesI, user_precision_t *d_power_ref_stokesQ,
           user_precision_t *d_power_ref_stokesU, user_precision_t *d_power_ref_stokesV,
           user_precision_t *d_power_SIs, int iComponent, int iBaseline,
           user_precision_t * flux_I, user_precision_t * flux_Q,
           user_precision_t * flux_U, user_precision_t * flux_V){

  double d_freq = VELC / d_allsteps_wavelengths[iBaseline];
  double d_ref_freq = d_power_ref_freqs[iComponent];

  user_precision_t flux_ratio = pow(d_freq / d_ref_freq, d_power_SIs[iComponent]);

  * flux_I = d_power_ref_stokesI[iComponent] * flux_ratio;
  * flux_Q = d_power_ref_stokesQ[iComponent] * flux_ratio;
  * flux_U = d_power_ref_stokesU[iComponent] * flux_ratio;
  * flux_V = d_power_ref_stokesV[iComponent] * flux_ratio;

}

__device__  cuUserComplex calc_measurement_equation(user_precision_t *d_us,
           user_precision_t *d_vs, user_precision_t *d_ws,
           double *d_ls, double *d_ms, double *d_ns,
           const int iBaseline, const int iComponent){

  cuUserComplex visi;

  double u, v, w;
  double l, m, n;

  u = (double)d_us[iBaseline];
  v = (double)d_vs[iBaseline];
  w = (double)d_ws[iBaseline];

  l = d_ls[iComponent];
  m = d_ms[iComponent];
  n = d_ns[iComponent];

  //Not sure why, but get match with OSKAR/RTS sims, and correct location
  //on sky through WSClean, without negative infront on 2pi
  double temp = 2*M_PI*( u*l + v*m + w*(n-1) );

  visi.y = (user_precision_t)sin(temp);
  visi.x = (user_precision_t)cos(temp);

  return visi;
}

__device__ void apply_beam_gains(cuUserComplex g1x, cuUserComplex D1x,
          cuUserComplex D1y, cuUserComplex g1y,
          cuUserComplex g2x, cuUserComplex D2x,
          cuUserComplex D2y, cuUserComplex g2y,
          user_precision_t flux_I, user_precision_t flux_Q,
          user_precision_t flux_U, user_precision_t flux_V,
          cuUserComplex visi_component,
          cuUserComplex * visi_XX, cuUserComplex * visi_XY,
          cuUserComplex * visi_YX, cuUserComplex * visi_YY) {

  //Conjugate the second beam gains
  cuUserComplex g2x_conj = make_cuUserComplex(g2x.x,-g2x.y);
  cuUserComplex D2x_conj = make_cuUserComplex(D2x.x,-D2x.y);
  cuUserComplex D2y_conj = make_cuUserComplex(D2y.x,-D2y.y);
  cuUserComplex g2y_conj = make_cuUserComplex(g2y.x,-g2y.y);

  //Create the Stokes visibilities
  cuUserComplex visi_I = make_cuUserComplex(flux_I, 0.0)*visi_component;
  cuUserComplex visi_Q = make_cuUserComplex(flux_Q, 0.0)*visi_component;
  cuUserComplex visi_U = make_cuUserComplex(flux_U, 0.0)*visi_component;
  cuUserComplex visi_V = make_cuUserComplex(flux_V, 0.0)*visi_component;

  cuUserComplex this_XX;
  cuUserComplex this_XY;
  cuUserComplex this_YX;
  cuUserComplex this_YY;

  // this_XX = (g1x*g2x_conj + D1x*D2x_conj);
  // this_XY = (g1x*D2y_conj + D1x*g2y_conj);
  // this_YX = (D1y*g2x_conj + g1y*D2x_conj);
  // this_YY = (D1y*D2y_conj + g1y*g2y_conj);
  //
  // printf("XX %.16f %.16f\n",this_XX.x, this_XX.y );
  // printf("XY %.16f %.16f\n",this_XY.x, this_XY.y );
  // printf("YX %.16f %.16f\n",this_YX.x, this_YX.y );
  // printf("YY %.16f %.16f\n",this_YY.x, this_YY.y );

  this_XX = (g1x*g2x_conj + D1x*D2x_conj)*visi_I;
  this_XX += (g1x*g2x_conj - D1x*D2x_conj)*visi_Q;
  this_XX += (g1x*D2x_conj + D1x*g2x_conj)*visi_U;
  this_XX += (make_cuUserComplex(0.0,1.0)*visi_V)*(g1x*D2x_conj - D1x*g2x_conj);

  this_XY = (g1x*D2y_conj + D1x*g2y_conj)*visi_I;
  this_XY += (g1x*D2y_conj - D1x*g2y_conj)*visi_Q;
  this_XY += (g1x*g2y_conj + D1x*D2y_conj)*visi_U;
  this_XY += (make_cuUserComplex(0.0,1.0)*visi_V)* (g1x*g2y_conj - D1x*D2y_conj);

  this_YX = (D1y*g2x_conj + g1y*D2x_conj)*visi_I;
  this_YX += (D1y*g2x_conj - g1y*D2x_conj)*visi_Q;
  this_YX += (D1y*D2x_conj + g1y*g2x_conj)*visi_U;
  this_YX += (make_cuUserComplex(0.0,1.0)*visi_V)* (D1y*D2x_conj - g1y*g2x_conj);

  this_YY = (D1y*D2y_conj + g1y*g2y_conj)*visi_I;
  this_YY += (D1y*D2y_conj - g1y*g2y_conj)*visi_Q;
  this_YY += (D1y*g2y_conj + g1y*D2y_conj)*visi_U;
  this_YY += (make_cuUserComplex(0.0,1.0)*visi_V)* (D1y*g2y_conj - g1y*D2y_conj);

  * visi_XX = this_XX;
  * visi_XY = this_XY;
  * visi_YX = this_YX;
  * visi_YY = this_YY;

}

__device__ void get_beam_gains(int iBaseline, int iComponent, int num_freqs,
           int num_baselines, int num_components, int num_times, int beamtype,
           cuUserComplex *d_primay_beam_J00, cuUserComplex *d_primay_beam_J01,
           cuUserComplex *d_primay_beam_J10, cuUserComplex *d_primay_beam_J11,
           cuUserComplex * g1x, cuUserComplex * D1x,
           cuUserComplex * D1y, cuUserComplex * g1y,
           cuUserComplex * g2x, cuUserComplex * D2x,
           cuUserComplex * D2y, cuUserComplex * g2y){

  int beam_ind = 0;
  int time_ind = 0;
  int freq_ind = 0;

  time_ind = (int)floorf( (user_precision_t)iBaseline / ((user_precision_t)num_baselines * (user_precision_t)num_freqs));
  freq_ind = (int)floorf( ((user_precision_t)iBaseline - ((user_precision_t)time_ind*(user_precision_t)num_baselines * (user_precision_t)num_freqs)) / (user_precision_t)num_baselines);
  beam_ind = num_freqs*time_ind*num_components + (num_components*freq_ind) + iComponent;

    //Set gains to one if no beam
  if (beamtype == NO_BEAM) {
    * g1x = make_cuUserComplex(1.0, 0.0);
    * g2x = make_cuUserComplex(1.0, 0.0);
    * g1y = make_cuUserComplex(1.0, 0.0);
    * g2y = make_cuUserComplex(1.0, 0.0);
  }

  //Get gains if using a beam
  else {
    * g1x = d_primay_beam_J00[beam_ind];
    * g2x = d_primay_beam_J00[beam_ind];
    * g1y = d_primay_beam_J11[beam_ind];
    * g2y = d_primay_beam_J11[beam_ind];

  }

  //Only MWA models have leakge terms at the moment
  if (beamtype == FEE_BEAM || beamtype == FEE_BEAM_INTERP || beamtype == MWA_ANALY) {
    * D1x = d_primay_beam_J01[beam_ind];
    * D2x = d_primay_beam_J01[beam_ind];
    * D1y = d_primay_beam_J10[beam_ind];
    * D2y = d_primay_beam_J10[beam_ind];
  }
  // Set leakage to zero if no leakage
  else {
    * D1x = make_cuUserComplex(0.0, 0.0);
    * D2x = make_cuUserComplex(0.0, 0.0);
    * D1y = make_cuUserComplex(0.0, 0.0);
    * D2y = make_cuUserComplex(0.0, 0.0);
  }
} //end __device__ get_beam_gains

__device__ void update_sum_visis(int iBaseline, int iComponent, int num_freqs,
    int num_baselines, int num_components, int num_times, int beamtype,
    cuUserComplex *d_primay_beam_J00, cuUserComplex *d_primay_beam_J01,
    cuUserComplex *d_primay_beam_J10, cuUserComplex *d_primay_beam_J11,
    cuUserComplex visi_component,
    user_precision_t flux_I, user_precision_t flux_Q,
    user_precision_t flux_U, user_precision_t flux_V,
    user_precision_t *d_sum_visi_XX_real, user_precision_t *d_sum_visi_XX_imag,
    user_precision_t *d_sum_visi_XY_real, user_precision_t *d_sum_visi_XY_imag,
    user_precision_t *d_sum_visi_YX_real, user_precision_t *d_sum_visi_YX_imag,
    user_precision_t *d_sum_visi_YY_real, user_precision_t *d_sum_visi_YY_imag){

    cuUserComplex g1x;
    cuUserComplex D1x;
    cuUserComplex D1y;
    cuUserComplex g1y;
    cuUserComplex g2x;
    cuUserComplex D2x;
    cuUserComplex D2y;
    cuUserComplex g2y;

    get_beam_gains(iBaseline, iComponent, num_freqs,
               num_baselines, num_components, num_times, beamtype,
               d_primay_beam_J00, d_primay_beam_J01,
               d_primay_beam_J10, d_primay_beam_J11,
               &g1x, &D1x, &D1y, &g1y, &g2x, &D2x, &D2y, &g2y);

    cuUserComplex visi_XX;
    cuUserComplex visi_XY;
    cuUserComplex visi_YX;
    cuUserComplex visi_YY;

    apply_beam_gains(g1x, D1x, D1y, g1y, g2x, D2x, D2y, g2y,
                    flux_I, flux_Q, flux_U, flux_V,
                    visi_component, &visi_XX, &visi_XY, &visi_YX, &visi_YY);

    d_sum_visi_XX_real[iBaseline] += visi_XX.x;
    d_sum_visi_XX_imag[iBaseline] += visi_XX.y;

    d_sum_visi_XY_real[iBaseline] += visi_XY.x;
    d_sum_visi_XY_imag[iBaseline] += visi_XY.y;

    d_sum_visi_YX_real[iBaseline] += visi_YX.x;
    d_sum_visi_YX_imag[iBaseline] += visi_YX.y;

    d_sum_visi_YY_real[iBaseline] += visi_YY.x;
    d_sum_visi_YY_imag[iBaseline] += visi_YY.y;



}

//Allocate space for the extrapolated Stokes parameters
void malloc_extrapolated_flux_arrays(components_t *d_components, int num_comps,
                                     int num_freqs){
  // printf("DEFO MALLOCing FLUX ARRAYS\n");
  d_components->extrap_stokesI = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_components->extrap_stokesI,
                                   num_comps*num_freqs*sizeof(double) ));
  d_components->extrap_stokesQ = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_components->extrap_stokesQ,
                                   num_comps*num_freqs*sizeof(double) ));
  d_components->extrap_stokesU = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_components->extrap_stokesU,
                                   num_comps*num_freqs*sizeof(double) ));
  d_components->extrap_stokesV = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_components->extrap_stokesV,
                                   num_comps*num_freqs*sizeof(double) ));
}

__device__ void extrap_stokes_power_law(components_t d_components,
           double *d_extrap_freqs, int iFluxComp, int iFreq,
           user_precision_t * flux_I, user_precision_t * flux_Q,
           user_precision_t * flux_U, user_precision_t * flux_V){

  double d_freq = d_extrap_freqs[iFreq];
  double d_ref_freq = d_components.power_ref_freqs[iFluxComp];

  user_precision_t flux_ratio = pow(d_freq / d_ref_freq, d_components.power_SIs[iFluxComp]);

  * flux_I = d_components.power_ref_stokesI[iFluxComp] * flux_ratio;
  * flux_Q = d_components.power_ref_stokesQ[iFluxComp] * flux_ratio;
  * flux_U = d_components.power_ref_stokesU[iFluxComp] * flux_ratio;
  * flux_V = d_components.power_ref_stokesV[iFluxComp] * flux_ratio;

}

__global__ void kern_extrap_power_laws(int num_extrap_freqs, double *d_extrap_freqs,
                                       int num_comps, components_t d_components) {

  // Start by computing which baseline we're going to do
  const int iFluxComp = threadIdx.x + (blockDim.x*blockIdx.x);
  const int iFreq = threadIdx.y + (blockDim.y*blockIdx.y);
  // if(iBaseline < num_visis && iComponent < num_points) {
  if(iFluxComp < num_comps && iFreq < num_extrap_freqs) {

    user_precision_t flux_I;
    user_precision_t flux_Q;
    user_precision_t flux_U;
    user_precision_t flux_V;

    extrap_stokes_power_law(d_components, d_extrap_freqs,
                 iFluxComp, iFreq,
                 &flux_I, &flux_Q, &flux_U, &flux_V);

    int iComponent = d_components.power_comp_inds[iFluxComp];
    int extrap_ind = num_extrap_freqs*iComponent + iFreq;


    d_components.extrap_stokesI[extrap_ind] = flux_I;
    d_components.extrap_stokesQ[extrap_ind] = flux_Q;
    d_components.extrap_stokesU[extrap_ind] = flux_U;
    d_components.extrap_stokesV[extrap_ind] = flux_V;

    // printf("kern_POWER %d %.1f %.1f %.1f %.1f\n",extrap_ind,
    //                               d_components.extrap_stokesI[extrap_ind],
    //                               d_components.extrap_stokesQ[extrap_ind],
    //                               d_components.extrap_stokesU[extrap_ind],
    //                               d_components.extrap_stokesV[extrap_ind] );

  }
}

__device__ void extrap_stokes_curved_power_law(components_t d_components,
           double *d_extrap_freqs, int iFluxComp, int iFreq,
           user_precision_t * flux_I, user_precision_t * flux_Q,
           user_precision_t * flux_U, user_precision_t * flux_V){

  double d_freq = d_extrap_freqs[iFreq];
  double d_ref_freq = d_components.curve_ref_freqs[iFluxComp];

  user_precision_t si_ratio = pow(d_freq / d_ref_freq, d_components.curve_SIs[iFluxComp]);

  double logfreq = log(d_freq);
  double logfreq_ref = log(d_ref_freq);

  double q = (double)d_components.curve_qs[iFluxComp];

  double exp_extrap = exp(q*logfreq*logfreq);
  double exp_ref = exp(q*logfreq_ref*logfreq_ref);

  user_precision_t flux_ratio = si_ratio * (exp_extrap / exp_ref);

  * flux_I = d_components.curve_ref_stokesI[iFluxComp] * flux_ratio;
  * flux_Q = d_components.curve_ref_stokesQ[iFluxComp] * flux_ratio;
  * flux_U = d_components.curve_ref_stokesU[iFluxComp] * flux_ratio;
  * flux_V = d_components.curve_ref_stokesV[iFluxComp] * flux_ratio;

}

__global__ void kern_extrap_curved_power_laws(int num_extrap_freqs, double *d_extrap_freqs,
                                              int num_comps, components_t d_components) {

  // Start by computing which baseline we're going to do
  const int iFluxComp = threadIdx.x + (blockDim.x*blockIdx.x);
  const int iFreq = threadIdx.y + (blockDim.y*blockIdx.y);

  if(iFluxComp < num_comps && iFreq < num_extrap_freqs) {

    user_precision_t flux_I;
    user_precision_t flux_Q;
    user_precision_t flux_U;
    user_precision_t flux_V;

    extrap_stokes_curved_power_law(d_components, d_extrap_freqs,
                 iFluxComp, iFreq,
                 &flux_I, &flux_Q, &flux_U, &flux_V);

    int iComponent = d_components.curve_comp_inds[iFluxComp];
    int extrap_ind = num_extrap_freqs*iComponent + iFreq;

    d_components.extrap_stokesI[extrap_ind] = flux_I;
    d_components.extrap_stokesQ[extrap_ind] = flux_Q;
    d_components.extrap_stokesU[extrap_ind] = flux_U;
    d_components.extrap_stokesV[extrap_ind] = flux_V;

    // printf("kern_CURVE %d %.1f %.1f %.1f %.1f\n",extrap_ind,
    //                           d_components.extrap_stokesI[extrap_ind],
    //                           d_components.extrap_stokesQ[extrap_ind],
    //                           d_components.extrap_stokesU[extrap_ind],
    //                           d_components.extrap_stokesV[extrap_ind] );

  }
}

__device__ user_precision_t calc_gradient_extrap_list(user_precision_t *list_fluxes,
          double *list_freqs, double desired_freq, int low_ind_1, int low_ind_2) {

  user_precision_t gradient = (list_fluxes[low_ind_2] - list_fluxes[low_ind_1]) / (list_freqs[low_ind_2] - list_freqs[low_ind_1]);
  user_precision_t extrap_flux = list_fluxes[low_ind_1] + gradient*(desired_freq - list_freqs[low_ind_1]);

  if (list_fluxes[low_ind_2] != 0 && list_fluxes[low_ind_1] != 0) {
    // printf("------------------------------------------------------\n");
    // printf("low freq, flux %.3e %.3f\n", list_freqs[low_ind_1], list_fluxes[low_ind_1]);
    // printf("high freq, flux %.3e %.3f\n", list_freqs[low_ind_2], list_fluxes[low_ind_2]);

    // printf("gradient, extrap_flux %.3e %.4f\n", gradient, extrap_flux);
    // printf("bottom bit %.3f %.3e\n",list_fluxes[low_ind_1],
    //                         desired_freq - list_freqs[low_ind_1]);

    // printf("%.3e %.3e %.3e\n",list_freqs[low_ind_1], desired_freq, list_freqs[low_ind_2] );
  }



  return extrap_flux;
}

__device__ void extrap_stokes_list_flux(components_t d_components,
           double *d_extrap_freqs, int iFluxComp, int iFreq,
           user_precision_t * flux_I, user_precision_t * flux_Q,
           user_precision_t * flux_U, user_precision_t * flux_V){

  int num_list_values = d_components.num_list_values[iFluxComp];
  int list_start_ind = d_components.list_start_indexes[iFluxComp];

  double d_extrap_freq = d_extrap_freqs[iFreq];

  int low_ind_1 = -1;
  int low_ind_2 = -1;

  double low_val_1 = 1e16;
  double low_val_2 = 1e16;

  double ref_freq;
  double abs_diff_freq;

  if (num_list_values == 1) {
    * flux_I = d_components.list_stokesI[list_start_ind];
    * flux_Q = d_components.list_stokesQ[list_start_ind];
    * flux_U = d_components.list_stokesU[list_start_ind];
    * flux_V = d_components.list_stokesV[list_start_ind];
    return;
  }

  //First loop finds the absolute closest frequency
  for (int i = 0; i < num_list_values; i++) {
    ref_freq = d_components.list_freqs[list_start_ind + i];
    abs_diff_freq = abs(ref_freq - d_extrap_freq);

    if (abs_diff_freq < low_val_1) {
      low_val_1 = abs_diff_freq;
      low_ind_1 = i;
    }
  }

  //Depending on the closest frequency, we either want to search above or
  //below the target frequency to find points either side of the target freq

  //We happen to need the reference frequency; just return the refs
  if (d_components.list_freqs[list_start_ind + low_ind_1] == d_extrap_freq) {
    // if (iFluxComp == 5 && iFreq == 13){
      // printf("We are heeeeere iFreq %d\n", iFreq);
    // }
    * flux_I = d_components.list_stokesI[list_start_ind + low_ind_1];
    * flux_Q = d_components.list_stokesQ[list_start_ind + low_ind_1];
    * flux_U = d_components.list_stokesU[list_start_ind + low_ind_1];
    * flux_V = d_components.list_stokesV[list_start_ind + low_ind_1];
  }
  else {
    //We need to search lower than this index
    if (d_components.list_freqs[list_start_ind + low_ind_1] > d_extrap_freq){
      //We are extrapolating to a frequency that is lower than all list entries
      //so just stick low_ind_2 to one above low_ind_1
      if (low_ind_1 == 0) {
        low_ind_2 = 1;
      }
      //Otherwise, need to actually look for closest freq
      else {
        for (int i = 0; i < low_ind_1; i++) {
          ref_freq = d_components.list_freqs[list_start_ind + i];
          abs_diff_freq = abs(ref_freq - d_extrap_freq);

          if (abs_diff_freq < low_val_2) {
            low_val_2 = abs_diff_freq;
            low_ind_2 = i;
          }
        }
      }
    }
    //We need to search higher than this index
    else {
      //We are extrapolating to a frequency that is higher than all list entries
      //so just stick low_ind_2 to one below low_ind_1
      if (low_ind_1 == num_list_values - 1) {
        low_ind_2 = low_ind_1 - 1;
      }
      //Otherwise, need to actually look for closest freq
      else {
        for (int i = low_ind_1 + 1; i < num_list_values; i++) {

          ref_freq = d_components.list_freqs[list_start_ind + i];
          abs_diff_freq = abs(ref_freq - d_extrap_freq);

          if (abs_diff_freq < low_val_2) {
            low_val_2 = abs_diff_freq;
            low_ind_2 = i;
          }
        }
      }
    }
    // printf("low_ind_1, low_ind_2 %d %d \n",low_ind_1, low_ind_2);

    // if (low_ind_1 == low_ind_2){
    //     low_ind_2 = num_list_values - 1;
    //     low_ind_1 = num_list_values - 2;
    // }

    * flux_I = calc_gradient_extrap_list(d_components.list_stokesI,
              d_components.list_freqs, d_extrap_freq,
              list_start_ind + low_ind_1, list_start_ind + low_ind_2);
    * flux_Q = calc_gradient_extrap_list(d_components.list_stokesQ,
              d_components.list_freqs, d_extrap_freq,
              list_start_ind + low_ind_1, list_start_ind + low_ind_2);
    * flux_U = calc_gradient_extrap_list(d_components.list_stokesU,
              d_components.list_freqs, d_extrap_freq,
              list_start_ind + low_ind_1, list_start_ind + low_ind_2);
    * flux_V = calc_gradient_extrap_list(d_components.list_stokesV,
              d_components.list_freqs, d_extrap_freq,
              list_start_ind + low_ind_1, list_start_ind + low_ind_2);

    if (low_ind_2 == -1){
      printf("wrong range %.3e %.3e iFreq %d %.3e low %d %.3e\n", d_components.list_freqs[list_start_ind],
      d_components.list_freqs[list_start_ind + num_list_values-1],
      iFreq, d_extrap_freq,
      low_ind_1, d_components.list_freqs[list_start_ind + low_ind_1]);
      printf("The flooxes %.3e %.3e %.3e %.3e\n",* flux_I, * flux_Q, * flux_U, * flux_V );
    }
  }
}


__global__ void kern_extrap_list_fluxes(int num_extrap_freqs, double *d_extrap_freqs,
                                        int num_comps, components_t d_components) {

  // Start by computing which baseline we're going to do
  const int iFluxComp = threadIdx.x + (blockDim.x*blockIdx.x);
  const int iFreq = threadIdx.y + (blockDim.y*blockIdx.y);
  // if(iBaseline < num_visis && iComponent < num_points) {
  if(iFluxComp < num_comps && iFreq < num_extrap_freqs) {

    user_precision_t flux_I;
    user_precision_t flux_Q;
    user_precision_t flux_U;
    user_precision_t flux_V;

    extrap_stokes_list_flux(d_components, d_extrap_freqs,
                 iFluxComp, iFreq,
                 &flux_I, &flux_Q, &flux_U, &flux_V);

    int iComponent = d_components.list_comp_inds[iFluxComp];
    int extrap_ind = num_extrap_freqs*iComponent + iFreq;

    d_components.extrap_stokesI[extrap_ind] = flux_I;
    d_components.extrap_stokesQ[extrap_ind] = flux_Q;
    d_components.extrap_stokesU[extrap_ind] = flux_U;
    d_components.extrap_stokesV[extrap_ind] = flux_V;

    // printf("kern_LIST %d %.1f %.1f %.1f %.1f\n",extrap_ind,
    //                           d_components.extrap_stokesI[extrap_ind],
    //                           d_components.extrap_stokesQ[extrap_ind],
    //                           d_components.extrap_stokesU[extrap_ind],
    //                           d_components.extrap_stokesV[extrap_ind] );

  }
}


extern "C" void extrapolate_Stokes(source_t *d_chunked_source,
                                   double *d_extrap_freqs, int num_extrap_freqs,
                                   e_component_type comptype){

  components_t d_components;
  // int n_comps = 0;
  int n_powers = 0;
  int n_curves = 0;
  int n_lists = 0;

  //Choose the right components to extrapolate for
  if (comptype == POINT) {
    d_components = d_chunked_source->point_components;
    // n_comps = d_chunked_source->n_points;
    n_powers = d_chunked_source->n_point_powers;
    n_curves = d_chunked_source->n_point_curves;
    n_lists = d_chunked_source->n_point_lists;
  }
  else if (comptype == GAUSSIAN) {
    d_components = d_chunked_source->gauss_components;
    // n_comps = d_chunked_source->n_gauss;
    n_powers = d_chunked_source->n_gauss_powers;
    n_curves = d_chunked_source->n_gauss_curves;
    n_lists = d_chunked_source->n_gauss_lists;
  } else if (comptype == SHAPELET) {
    d_components = d_chunked_source->shape_components;
    // n_comps = d_chunked_source->n_shapes;
    n_powers = d_chunked_source->n_shape_powers;
    n_curves = d_chunked_source->n_shape_curves;
    n_lists = d_chunked_source->n_shape_lists;
  }

  //For some reason, can't do this inside this function - something about
  //memory context or something? If done inside, when trying to access the
  //memory outside this function, get illegal memory issues. Memory
  //management sucks.
  // malloc_extrapolated_flux_arrays(&d_components, n_comps, num_extrap_freqs);

  dim3 grid, threads;

  threads.x = 16;
  threads.y = 16;

  //First up, do the POWER_LAW types
  grid.y = (int)ceilf( (float)num_extrap_freqs / (float)threads.y );

  if (n_powers > 0) {
    grid.x = (int)ceilf( (float)n_powers / (float)threads.x );
    cudaErrorCheckKernel("kern_extrap_power_laws",
                          kern_extrap_power_laws, grid, threads,
                          num_extrap_freqs, d_extrap_freqs,
                          n_powers, d_components);
  }
  //Next up, do the CURVED_POWER_LAW types
  if (n_curves > 0) {
    grid.x = (int)ceilf( (float)n_curves / (float)threads.x );

    cudaErrorCheckKernel("kern_extrap_curved_power_laws",
                          kern_extrap_curved_power_laws, grid, threads,
                          num_extrap_freqs, d_extrap_freqs,
                          n_curves, d_components);
  }

  //Finally, do any list flux peeps
  //Next up, do the CURVED_POWER_LAW types
  if (n_lists > 0) {
    grid.x = (int)ceilf( (float)n_lists / (float)threads.x );

    cudaErrorCheckKernel("kern_extrap_list_fluxes",
                          kern_extrap_list_fluxes, grid, threads,
                          num_extrap_freqs, d_extrap_freqs,
                          n_lists, d_components);
  }
}

extern "C" void source_component_common(woden_settings_t *woden_settings,
           beam_settings_t *beam_settings, double *d_freqs,
           source_t *chunked_source, source_t *d_chunked_source,
           d_beam_gains_t *d_component_beam_gains,
           e_component_type comptype){

  int num_components = 0;
  components_t *components = NULL;
  components_t *d_components = NULL;

  if (comptype == POINT) {
    num_components = d_chunked_source->n_points;
    components = &chunked_source->point_components;
    d_components = &d_chunked_source->point_components;
  } else if (comptype == GAUSSIAN) {
    num_components = d_chunked_source->n_gauss;
    components = &chunked_source->gauss_components;
    d_components = &d_chunked_source->gauss_components;
  } else if (comptype == SHAPELET) {
    num_components = d_chunked_source->n_shapes;
    components = &chunked_source->shape_components;
    d_components = &d_chunked_source->shape_components;
  }

  //Will need this later
  malloc_extrapolated_flux_arrays(d_components, num_components,
                                  woden_settings->num_freqs);

  extrapolate_Stokes(d_chunked_source, d_freqs, woden_settings->num_freqs,
                     comptype);

  //Only the MWA beams currently yields cross pol values, so only malloc what
  //we need here
  //TODO in the future, this might need to be a loop over all primary beams,
  //if we have different beams for different tiles
  if (beam_settings->beamtype == FEE_BEAM || beam_settings->beamtype == MWA_ANALY || beam_settings->beamtype == FEE_BEAM_INTERP) {
    cudaErrorCheckCall( hipMalloc( (void**)&d_component_beam_gains->d_Dxs,
                    d_components->num_primarybeam_values*sizeof(cuUserComplex) ));
    cudaErrorCheckCall( hipMalloc( (void**)&d_component_beam_gains->d_Dys,
                    d_components->num_primarybeam_values*sizeof(cuUserComplex) ));
  }
  cudaErrorCheckCall( hipMalloc( (void**)&d_component_beam_gains->d_gxs,
                    d_components->num_primarybeam_values*sizeof(cuUserComplex) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_component_beam_gains->d_gys,
                    d_components->num_primarybeam_values*sizeof(cuUserComplex) ));
  //
  cudaErrorCheckCall( hipMalloc( (void**)&d_components->ls,
                                               num_components*sizeof(double) ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_components->ms,
                                               num_components*sizeof(double) ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_components->ns,
                                               num_components*sizeof(double) ) );


  dim3 grid, threads;

  threads.x = 128;
  threads.y = 1;
  threads.z = 1;
  grid.x = (int)ceil( (float)num_components / (float)threads.x );
  grid.y = 1;
  grid.z = 1;

  cudaErrorCheckKernel("kern_calc_lmn",
                        kern_calc_lmn, grid, threads,
                        woden_settings->ra0,
                        woden_settings->sdec0, woden_settings->cdec0,
                        d_components->ras, d_components->decs,
                        d_components->ls, d_components->ms, d_components->ns, num_components);

  //If using a gaussian primary beam, calculate beam values for all freqs,
  //lsts and point component locations
  if (beam_settings->beamtype == GAUSS_BEAM) {

    //TODO currently hardcoded to have beam position angle = 0.
    //Should this change with az/za?
    user_precision_t cos_theta = 1.0;
    user_precision_t sin_theta = 0.0;
    user_precision_t sin_2theta = 0.0;
    user_precision_t fwhm_lm = sin(beam_settings->beam_FWHM_rad);

    printf("\tDoing Gaussian Beam\n");

    calculate_gaussian_beam(num_components,
         woden_settings->num_time_steps, woden_settings->num_freqs,
         beam_settings->gauss_ha, beam_settings->gauss_sdec,
         beam_settings->gauss_cdec,
         fwhm_lm, cos_theta, sin_theta, sin_2theta,
         beam_settings->beam_ref_freq, d_freqs,
         components->beam_has,
         components->beam_decs,
         d_component_beam_gains->d_gxs, d_component_beam_gains->d_gys);

  }// end if beam == GAUSS

  else if (beam_settings->beamtype == FEE_BEAM || beam_settings->beamtype == FEE_BEAM_INTERP) {

    double *double_azs = NULL;
    double *double_zas = NULL;

    #ifdef DOUBLE_PRECISION
      double_azs = components->azs;
      double_zas = components->zas;
    #else
      int num_azza = woden_settings->num_time_steps*num_components;

      double_azs = (double*)malloc(num_azza*sizeof(double));
      double_zas = (double*)malloc(num_azza*sizeof(double));

      for (int i = 0; i < num_azza; i++) {
        double_azs[i] = (double)components->azs[i];
        double_zas[i] = (double)components->zas[i];
      }
    #endif

    if (beam_settings->beamtype == FEE_BEAM_INTERP) {
      printf("\tDoing the hyperbeam (interpolated)\n");
    } else {
      printf("\tDoing the hyperbeam\n");
    }


    uint8_t parallactic = 1;
    // int num_freqs = 3;
    run_hyperbeam_cuda(num_components,
           woden_settings->num_time_steps, woden_settings->num_freqs,
           parallactic,
           beam_settings->cuda_fee_beam,
           double_azs, double_zas,
           d_component_beam_gains->d_gxs, d_component_beam_gains->d_Dxs,
           d_component_beam_gains->d_Dys, d_component_beam_gains->d_gys);

    #ifdef DOUBLE_PRECISION
      ;
    #else
      free(double_azs);
      free(double_zas);  
    #endif




  }

  else if (beam_settings->beamtype == ANALY_DIPOLE) {
    printf("\tDoing analytic_dipole (EDA2 beam)\n");

    calculate_analytic_dipole_beam(num_components,
         woden_settings->num_time_steps, woden_settings->num_freqs,
         components->azs, components->zas, d_freqs,
         d_component_beam_gains->d_gxs, d_component_beam_gains->d_gys);
  }

  else if (beam_settings->beamtype == MWA_ANALY) {

    //Always normalise to zenith
    int norm = 1;

    printf("\tDoing analytic MWA beam\n");

    calculate_RTS_MWA_analytic_beam(num_components,
         woden_settings->num_time_steps, woden_settings->num_freqs,
         components->azs, components->zas,
         woden_settings->FEE_ideal_delays, woden_settings->latitude,
         norm, components->beam_has, components->beam_decs,
         d_freqs, d_component_beam_gains->d_gxs, d_component_beam_gains->d_Dxs,
         d_component_beam_gains->d_Dys, d_component_beam_gains->d_gys);
  }

} //END source_component_common


__global__ void kern_calc_visi_point_or_gauss(components_t d_components,
           d_beam_gains_t d_component_beam_gains,
           user_precision_t *d_us, user_precision_t *d_vs, user_precision_t *d_ws,
           user_precision_t *d_sum_visi_XX_real, user_precision_t *d_sum_visi_XX_imag,
           user_precision_t *d_sum_visi_XY_real, user_precision_t *d_sum_visi_XY_imag,
           user_precision_t *d_sum_visi_YX_real, user_precision_t *d_sum_visi_YX_imag,
           user_precision_t *d_sum_visi_YY_real, user_precision_t *d_sum_visi_YY_imag,
           int num_components, int num_baselines, int num_freqs, int num_visis,
           int num_times, e_beamtype beamtype, e_component_type comptype) {

  // Start by computing which baseline we're going to do
  const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);
  if(iBaseline < num_visis) {

    user_precision_t flux_I;
    user_precision_t flux_Q;
    user_precision_t flux_U;
    user_precision_t flux_V;

    cuUserComplex visi_comp;
    cuUserComplex V_envelop;

    user_precision_t pa, sinpa, cospa, u, v, x, y, invsig_x, invsig_y;

    //Find out what time and freq index this baseline corresponds to
    int time_ind = (int)floorf( (float)iBaseline / ((float)num_baselines * (float)num_freqs));
    int freq_ind = (int)floorf( ((float)iBaseline - ((float)time_ind*(float)num_baselines * (float)num_freqs)) / (float)num_baselines);

    for (int iComponent = 0; iComponent < num_components; iComponent++) {


      int extrap_ind = num_freqs*iComponent + freq_ind;

      flux_I = d_components.extrap_stokesI[extrap_ind];
      flux_Q = d_components.extrap_stokesQ[extrap_ind];
      flux_U = d_components.extrap_stokesU[extrap_ind];
      flux_V = d_components.extrap_stokesV[extrap_ind];

      visi_comp = calc_measurement_equation(d_us, d_vs, d_ws,
                             d_components.ls, d_components.ms, d_components.ns,
                             iBaseline, iComponent);

      if (comptype == GAUSSIAN) {

        V_envelop = make_cuUserComplex( 1.0, 0.0 );

        pa = d_components.pas[iComponent];
        sinpa = sin(pa);
        cospa = cos(pa);
        u = d_us[iBaseline];
        v = d_vs[iBaseline];

        x =  cospa*v + sinpa*u; // major axis
        y = -sinpa*v + cospa*u; // minor axis
        invsig_x = d_components.majors[iComponent];
        invsig_y = d_components.minors[iComponent];

        V_envelop = make_cuUserComplex( exp( -0.5 * ( x*x*invsig_x*invsig_x*M_PI_2_2_LN_2 + y*y*invsig_y*invsig_y*M_PI_2_2_LN_2 ) ), 0.0 );

        visi_comp = visi_comp*V_envelop;
      }

      update_sum_visis(iBaseline, iComponent, num_freqs,
             num_baselines, num_components, num_times, beamtype,
             d_component_beam_gains.d_gxs, d_component_beam_gains.d_Dxs,
             d_component_beam_gains.d_Dys, d_component_beam_gains.d_gys,
             visi_comp, flux_I, flux_Q, flux_U, flux_V,
             d_sum_visi_XX_real, d_sum_visi_XX_imag,
             d_sum_visi_XY_real, d_sum_visi_XY_imag,
             d_sum_visi_YX_real, d_sum_visi_YX_imag,
             d_sum_visi_YY_real, d_sum_visi_YY_imag);
    }
  }
}

__global__ void kern_calc_visi_shapelets(components_t d_components,
      d_beam_gains_t d_component_beam_gains,
      user_precision_t *d_us, user_precision_t *d_vs, user_precision_t *d_ws,
      user_precision_t *d_allsteps_wavelengths,
      user_precision_t *d_u_shapes, user_precision_t *d_v_shapes,
      user_precision_t *d_w_shapes,
      user_precision_t *d_sum_visi_XX_real, user_precision_t *d_sum_visi_XX_imag,
      user_precision_t *d_sum_visi_XY_real, user_precision_t *d_sum_visi_XY_imag,
      user_precision_t *d_sum_visi_YX_real, user_precision_t *d_sum_visi_YX_imag,
      user_precision_t *d_sum_visi_YY_real, user_precision_t *d_sum_visi_YY_imag,
      user_precision_t *d_sbf,
      int num_shapes, int num_baselines, int num_freqs, int num_visis,
      const int num_coeffs, int num_times, e_beamtype beamtype) {

  // Start by computing which baseline we're going to do
  const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);

  if (iBaseline < num_visis) {

    user_precision_t shape_flux_I;
    user_precision_t shape_flux_Q;
    user_precision_t shape_flux_U;
    user_precision_t shape_flux_V;
    cuUserComplex visi_shape;

    //Find out what time and freq index this baseline corresponds to
    int time_ind = (int)floorf( (float)iBaseline / ((float)num_baselines * (float)num_freqs));

    int freq_ind = (int)floorf( ((float)iBaseline - ((float)time_ind*(float)num_baselines * (float)num_freqs)) / (float)num_baselines);

    for (int iCoeff = 0; iCoeff < num_coeffs; iCoeff++) {

      //We have multiple coefficients per SHAPELET component - reference
      //them via this array. We chunk over coeffs so might have any
      //number of components here
      int iComponent = d_components.param_indexes[iCoeff];
      int extrap_ind = num_freqs*iComponent + freq_ind;

      shape_flux_I = d_components.extrap_stokesI[extrap_ind];
      shape_flux_Q = d_components.extrap_stokesQ[extrap_ind];
      shape_flux_U = d_components.extrap_stokesU[extrap_ind];
      shape_flux_V = d_components.extrap_stokesV[extrap_ind];

      visi_shape = calc_measurement_equation(d_us, d_vs, d_ws,
                            d_components.ls, d_components.ms, d_components.ns,
                            iBaseline, iComponent);

      user_precision_t pa = d_components.pas[iComponent];
      user_precision_t sinpa = sin(pa);
      user_precision_t cospa = cos(pa);

      user_precision_t u_shape = d_u_shapes[iComponent*num_visis + iBaseline];
      user_precision_t v_shape = d_v_shapes[iComponent*num_visis + iBaseline];

      user_precision_t x = (cospa*v_shape + sinpa*u_shape); // major axis
      user_precision_t y = (-sinpa*v_shape + cospa*u_shape); // minor axis

      //Scales the FWHM to std to match basis functions, and account for the
      //basis functions being stored with beta = 1.0
      //Basis functions have been stored in such a way that x is in the same
      //direction as on sky, but y is opposite, so include negative here
      user_precision_t const_x = (d_components.majors[iComponent]*SQRT_M_PI_2_2_LN_2)/sbf_dx;
      user_precision_t const_y = -(d_components.minors[iComponent]*SQRT_M_PI_2_2_LN_2)/sbf_dx;

      // I^(n1+n2) = Ipow_lookup[(n1+n2) % 4]
      cuUserComplex Ipow_lookup[] = { make_cuUserComplex(  1.0,  0.0 ),
                                       make_cuUserComplex(  0.0,  1.0 ),
                                       make_cuUserComplex( -1.0,  0.0 ),
                                       make_cuUserComplex(  0.0, -1.0 ) };

      user_precision_t xlow, xhigh, ylow, yhigh, u_value, v_value, f_hat, *sbf_n;

      // find the indices in the basis functions for u*beta_u and v*beta_v

      user_precision_t xpos = x*const_x + sbf_c;
      user_precision_t ypos = y*const_y + sbf_c;

      int xindex = (int)floor(xpos);
      int yindex = (int)floor(ypos);
      //
      int n1 = (int)d_components.n1s[iCoeff];
      int n2 = (int)d_components.n2s[iCoeff];

      f_hat = d_components.shape_coeffs[iCoeff];

      sbf_n = &d_sbf[n1*sbf_L];
      xlow  = sbf_n[xindex];
      xhigh = sbf_n[xindex+1];
      u_value = xlow + (xhigh-xlow)*(xpos-xindex);

      sbf_n = &d_sbf[n2*sbf_L];
      ylow  = sbf_n[yindex];
      yhigh = sbf_n[yindex+1];
      v_value = ylow + (yhigh-ylow)*(ypos-yindex);

      // accumulate the intensity model for baseline pair (u,v)
      cuUserComplex V_envelop = make_cuUserComplex( 0.0, 0.0 );
      V_envelop = V_envelop + Ipow_lookup[(n1+n2) % 4] * f_hat * u_value*v_value;

      visi_shape = visi_shape*V_envelop;

      update_sum_visis(iBaseline, iComponent, num_freqs,
             num_baselines, num_shapes, num_times, beamtype,
             d_component_beam_gains.d_gxs, d_component_beam_gains.d_Dxs,
             d_component_beam_gains.d_Dys, d_component_beam_gains.d_gys,
             visi_shape,
             shape_flux_I, shape_flux_Q, shape_flux_U, shape_flux_V,
             d_sum_visi_XX_real, d_sum_visi_XX_imag,
             d_sum_visi_XY_real, d_sum_visi_XY_imag,
             d_sum_visi_YX_real, d_sum_visi_YX_imag,
             d_sum_visi_YY_real, d_sum_visi_YY_imag);

      // if (iBaseline == 0) {
      //   printf("iComp, pa, major, minor, n1, n2, coeff %d %.5f %.5f %.5f %d %d %.5f\n",
      //           iComponent, pa, d_components.majors[iComponent], d_components.minors[iComponent],
      //           (int)d_components.n1s[iCoeff],(int)d_components.n2s[iCoeff], f_hat );
      //   printf("V_envelop %.5f %.5f\n",V_envelop.x, V_envelop.y );
      //   printf("u_value, v_value %.5f %.5f\n",u_value, v_value );
      // }
    }
  }
}

//Copy the sky model info from a set of components from the CPU to the GPU
void copy_components_to_GPU(source_t *chunked_source, source_t *d_chunked_source,
                            e_component_type comptype) {

  components_t *components;
  components_t *d_components;
  int num_comps = 0, num_shape_coeffs = 0;
  int num_powers = 0, num_curves = 0, num_lists = 0;

  if (comptype == POINT) {
    components = &chunked_source->point_components;
    d_components = &d_chunked_source->point_components;

    num_comps = chunked_source->n_points;
    num_shape_coeffs = 0;
    num_powers = chunked_source->n_point_powers;
    num_curves = chunked_source->n_point_curves;
    num_lists = chunked_source->n_point_lists;

  }
  else if (comptype == GAUSSIAN) {
    components = &chunked_source->gauss_components;
    d_components = &d_chunked_source->gauss_components;

    num_comps = chunked_source->n_gauss;
    num_shape_coeffs = 0;
    num_powers = chunked_source->n_gauss_powers;
    num_curves = chunked_source->n_gauss_curves;
    num_lists = chunked_source->n_gauss_lists;

  }
  else if (comptype == SHAPELET) {
    components = &chunked_source->shape_components;
    d_components = &d_chunked_source->shape_components;

    num_comps = chunked_source->n_shapes;
    num_shape_coeffs = chunked_source->n_shape_coeffs;
    num_powers = chunked_source->n_shape_powers;
    num_curves = chunked_source->n_shape_curves;
    num_lists = chunked_source->n_shape_lists;

  }

  // printf("INSIDE MALLOC %d %d %d %d %d\n", num_comps, num_shape_coeffs, num_powers,
  //                           num_curves, num_lists );

  //Common attributes between all flux types and components types
  cudaErrorCheckCall( hipMalloc( (void**)&d_components->ras,
                      num_comps*sizeof(double) ) );
  cudaErrorCheckCall( hipMemcpy( d_components->ras, components->ras,
                      num_comps*sizeof(double), hipMemcpyHostToDevice ) );

  cudaErrorCheckCall( hipMalloc( (void**)&d_components->decs,
                      num_comps*sizeof(double) ) );
  cudaErrorCheckCall( hipMemcpy( d_components->decs, components->decs,
                      num_comps*sizeof(double), hipMemcpyHostToDevice ) );

  d_components->num_primarybeam_values = components->num_primarybeam_values;

  //GAUSSIAN and SHAPELET only attributes
  if (comptype == GAUSSIAN || comptype == SHAPELET ) {
    cudaErrorCheckCall( hipMalloc( (void**)&d_components->pas,
                        num_comps*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->pas, components->pas,
                        num_comps*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->majors,
                        num_comps*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->majors, components->majors,
                        num_comps*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->minors,
                        num_comps*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->minors, components->minors,
                        num_comps*sizeof(user_precision_t), hipMemcpyHostToDevice ) );
  }

  //SHAPELET only attributes
  if (comptype == SHAPELET) {
    cudaErrorCheckCall( hipMalloc( (void**)&d_components->shape_coeffs,
                        num_shape_coeffs*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->shape_coeffs, components->shape_coeffs,
                        num_shape_coeffs*sizeof(user_precision_t),
                        hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->n1s,
                        num_shape_coeffs*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->n1s, components->n1s,
                        num_shape_coeffs*sizeof(user_precision_t),
                        hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->n2s,
                        num_shape_coeffs*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->n2s, components->n2s,
                        num_shape_coeffs*sizeof(user_precision_t),
                        hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->param_indexes,
                        num_shape_coeffs*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->param_indexes, components->param_indexes,
                        num_shape_coeffs*sizeof(user_precision_t),
                        hipMemcpyHostToDevice ) );
  }

  //POWER_LAW flux things
  if (num_powers > 0) {
    // printf("MALLOC BEHBEH num_powers %d\n",num_powers );
    cudaErrorCheckCall( hipMalloc( (void**)&d_components->power_comp_inds,
                        num_powers*sizeof(int) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->power_comp_inds, components->power_comp_inds,
                        num_powers*sizeof(int), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->power_ref_freqs,
                        num_powers*sizeof(double) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->power_ref_freqs, components->power_ref_freqs,
                        num_powers*sizeof(double), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->power_ref_stokesI,
                        num_powers*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->power_ref_stokesI, components->power_ref_stokesI,
                        num_powers*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->power_ref_stokesQ,
                        num_powers*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->power_ref_stokesQ, components->power_ref_stokesQ,
                        num_powers*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->power_ref_stokesU,
                        num_powers*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->power_ref_stokesU, components->power_ref_stokesU,
                        num_powers*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->power_ref_stokesV,
                        num_powers*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->power_ref_stokesV, components->power_ref_stokesV,
                        num_powers*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->power_SIs,
                        num_powers*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->power_SIs, components->power_SIs,
                        num_powers*sizeof(user_precision_t), hipMemcpyHostToDevice ) );
  }

  //CURVED_POWER_LAW things
  if (num_curves > 0) {
    // printf("MALLOC BEHBEH num_curves %d\n",num_curves );
    cudaErrorCheckCall( hipMalloc( (void**)&d_components->curve_comp_inds,
                        num_curves*sizeof(int) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->curve_comp_inds, components->curve_comp_inds,
                        num_curves*sizeof(int), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->curve_ref_freqs,
                        num_curves*sizeof(double) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->curve_ref_freqs, components->curve_ref_freqs,
                        num_curves*sizeof(double), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->curve_ref_stokesI,
                        num_curves*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->curve_ref_stokesI, components->curve_ref_stokesI,
                        num_curves*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->curve_ref_stokesQ,
                        num_curves*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->curve_ref_stokesQ, components->curve_ref_stokesQ,
                        num_curves*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->curve_ref_stokesU,
                        num_curves*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->curve_ref_stokesU, components->curve_ref_stokesU,
                        num_curves*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->curve_ref_stokesV,
                        num_curves*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->curve_ref_stokesV, components->curve_ref_stokesV,
                        num_curves*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->curve_SIs,
                        num_curves*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->curve_SIs, components->curve_SIs,
                        num_curves*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->curve_qs,
                        num_curves*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->curve_qs, components->curve_qs,
                        num_curves*sizeof(user_precision_t), hipMemcpyHostToDevice ) );
  }

  //LIST things
  if (num_lists > 0) {
    int num_list_values = components->total_num_flux_entires;

    // printf("MALLOC TIME WITH THE LISTS YES? %d\n", num_list_values);
    cudaErrorCheckCall( hipMalloc( (void**)&d_components->list_comp_inds,
                        num_lists*sizeof(int) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->list_comp_inds,
                        components->list_comp_inds,
                        num_lists*sizeof(int), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->num_list_values,
                        num_lists*sizeof(int) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->num_list_values,
                        components->num_list_values,
                        num_lists*sizeof(int), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->list_start_indexes,
                        num_lists*sizeof(int) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->list_start_indexes,
                        components->list_start_indexes,
                        num_lists*sizeof(int), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->list_freqs,
                        num_list_values*sizeof(double) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->list_freqs, components->list_freqs,
                        num_list_values*sizeof(double), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->list_stokesI,
                        num_list_values*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->list_stokesI, components->list_stokesI,
                        num_list_values*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->list_stokesQ,
                        num_list_values*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->list_stokesQ, components->list_stokesQ,
                        num_list_values*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->list_stokesU,
                        num_list_values*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->list_stokesU, components->list_stokesU,
                        num_list_values*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->list_stokesV,
                        num_list_values*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->list_stokesV, components->list_stokesV,
                        num_list_values*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

  }
}

source_t * copy_chunked_source_to_GPU(source_t *chunked_source){

  source_t *d_chunked_source = (source_t*)malloc(sizeof(source_t));

  if (chunked_source->n_points > 0) {
    copy_components_to_GPU(chunked_source, d_chunked_source, POINT);
  }
  if (chunked_source->n_gauss > 0) {
    copy_components_to_GPU(chunked_source, d_chunked_source, GAUSSIAN);
  }
  if (chunked_source->n_shapes > 0) {
    // printf("HAVE COPIED SHAPELET TINGS\n");
    copy_components_to_GPU(chunked_source, d_chunked_source, SHAPELET);
  }

  //copy across the component counters

  d_chunked_source->n_points = chunked_source->n_points;
  d_chunked_source->n_point_lists = chunked_source->n_point_lists;
  d_chunked_source->n_point_powers = chunked_source->n_point_powers;
  d_chunked_source->n_point_curves = chunked_source->n_point_curves;

  d_chunked_source->n_gauss = chunked_source->n_gauss;
  d_chunked_source->n_gauss_lists = chunked_source->n_gauss_lists;
  d_chunked_source->n_gauss_powers = chunked_source->n_gauss_powers;
  d_chunked_source->n_gauss_curves = chunked_source->n_gauss_curves;

  d_chunked_source->n_shapes = chunked_source->n_shapes;
  d_chunked_source->n_shape_lists = chunked_source->n_shape_lists;
  d_chunked_source->n_shape_powers = chunked_source->n_shape_powers;
  d_chunked_source->n_shape_curves = chunked_source->n_shape_curves;
  d_chunked_source->n_shape_coeffs = chunked_source->n_shape_coeffs;

  return d_chunked_source;
}

void free_extrapolated_flux_arrays(components_t *d_components){
  cudaErrorCheckCall( hipFree( d_components->extrap_stokesI ) );
  cudaErrorCheckCall( hipFree( d_components->extrap_stokesQ ) );
  cudaErrorCheckCall( hipFree( d_components->extrap_stokesU ) );
  cudaErrorCheckCall( hipFree( d_components->extrap_stokesV ) );
}



extern "C" void free_d_components(source_t *d_chunked_source,
                                  e_component_type comptype){
  components_t d_components;
  int n_powers = 0;
  int n_curves = 0;
  int n_lists = 0;

  if (comptype == POINT) {
    d_components = d_chunked_source->point_components;
    n_powers = d_chunked_source->n_point_powers;
    n_curves = d_chunked_source->n_point_curves;
    n_lists = d_chunked_source->n_point_lists;
  }
  else if (comptype == GAUSSIAN) {
    d_components = d_chunked_source->gauss_components;
    n_powers = d_chunked_source->n_gauss_powers;
    n_curves = d_chunked_source->n_gauss_curves;
    n_lists = d_chunked_source->n_gauss_lists;
  }
  else {
    d_components = d_chunked_source->shape_components;
    n_powers = d_chunked_source->n_shape_powers;
    n_curves = d_chunked_source->n_shape_curves;
    n_lists = d_chunked_source->n_shape_lists;
  }

  // printf("Freeing this %d %d %d\n",n_powers,n_curves,n_lists );

  cudaErrorCheckCall( hipFree( d_components.decs) );
  cudaErrorCheckCall( hipFree( d_components.ras) );

  cudaErrorCheckCall( hipFree( d_components.ls) );
  cudaErrorCheckCall( hipFree( d_components.ms) );
  cudaErrorCheckCall( hipFree( d_components.ns) );


  // printf("Did the common to all arrays\n");

  //The az,za,beam_has,beam_decs are handled by other functions

  if (n_powers > 0) {
    cudaErrorCheckCall( hipFree( d_components.power_ref_freqs ) );
    cudaErrorCheckCall( hipFree( d_components.power_ref_stokesI ) );
    cudaErrorCheckCall( hipFree( d_components.power_ref_stokesQ ) );
    cudaErrorCheckCall( hipFree( d_components.power_ref_stokesU ) );
    cudaErrorCheckCall( hipFree( d_components.power_ref_stokesV ) );
    cudaErrorCheckCall( hipFree( d_components.power_SIs ) );
    cudaErrorCheckCall( hipFree( d_components.power_comp_inds ) );
  }

  if (n_curves > 0) {
    cudaErrorCheckCall( hipFree( d_components.curve_ref_freqs ) );
    cudaErrorCheckCall( hipFree( d_components.curve_ref_stokesI ) );
    cudaErrorCheckCall( hipFree( d_components.curve_ref_stokesQ ) );
    cudaErrorCheckCall( hipFree( d_components.curve_ref_stokesU ) );
    cudaErrorCheckCall( hipFree( d_components.curve_ref_stokesV ) );
    cudaErrorCheckCall( hipFree( d_components.curve_SIs ) );
    cudaErrorCheckCall( hipFree( d_components.curve_qs ) );
    cudaErrorCheckCall( hipFree( d_components.curve_comp_inds ) );
  }
  if (n_lists > 0) {
    cudaErrorCheckCall( hipFree( d_components.list_comp_inds ) );
    cudaErrorCheckCall( hipFree( d_components.list_freqs ) );
    cudaErrorCheckCall( hipFree( d_components.list_stokesI ) );
    cudaErrorCheckCall( hipFree( d_components.list_stokesQ ) );
    cudaErrorCheckCall( hipFree( d_components.list_stokesU ) );
    cudaErrorCheckCall( hipFree( d_components.list_stokesV ) );
    cudaErrorCheckCall( hipFree( d_components.num_list_values ) );
    cudaErrorCheckCall( hipFree( d_components.list_start_indexes ) );
  }

  if (comptype == GAUSSIAN || comptype == SHAPELET) {
    cudaErrorCheckCall( hipFree( d_components.pas ) );
    cudaErrorCheckCall( hipFree( d_components.majors ) );
    cudaErrorCheckCall( hipFree( d_components.minors ) );
  }

  if (comptype == SHAPELET) {
    cudaErrorCheckCall( hipFree( d_components.shape_coeffs ) );
    cudaErrorCheckCall( hipFree( d_components.n1s ) );
    cudaErrorCheckCall( hipFree( d_components.n2s ) );
    cudaErrorCheckCall( hipFree( d_components.param_indexes ) );
  }
  // printf("DID all the freeing somehow??\n");
}

extern "C" void free_beam_gains(d_beam_gains_t d_beam_gains, e_beamtype beamtype){

  cudaErrorCheckCall( hipFree( d_beam_gains.d_gxs) );
  cudaErrorCheckCall( hipFree( d_beam_gains.d_gys) );

  if (beamtype == FEE_BEAM || beamtype == FEE_BEAM_INTERP || beamtype == MWA_ANALY){
    cudaErrorCheckCall( hipFree( d_beam_gains.d_Dxs ) );
    cudaErrorCheckCall( hipFree( d_beam_gains.d_Dys ) );
  }

}

/*******************************************************************************
                 Functions below to be used in unit tests
*******************************************************************************/

extern "C" void test_extrap_stokes_all_models(source_t *chunked_source,
           int num_extrap_freqs, double *extrap_freqs,
           user_precision_t *extrap_flux_I, user_precision_t *extrap_flux_Q,
           user_precision_t *extrap_flux_U, user_precision_t *extrap_flux_V){


  source_t *d_chunked_source = copy_chunked_source_to_GPU(chunked_source);

  double *d_extrap_freqs = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_extrap_freqs,
                                   num_extrap_freqs*sizeof(double) ));
  cudaErrorCheckCall( hipMemcpy(d_extrap_freqs, extrap_freqs,
             num_extrap_freqs*sizeof(double), hipMemcpyHostToDevice ));

  malloc_extrapolated_flux_arrays(&d_chunked_source->point_components,
                                  d_chunked_source->n_points,
                                  num_extrap_freqs);

  extrapolate_Stokes(d_chunked_source, d_extrap_freqs, num_extrap_freqs, POINT);


  components_t d_components = d_chunked_source->point_components;

  cudaErrorCheckCall( hipMemcpy(extrap_flux_I, d_components.extrap_stokesI,
            d_chunked_source->n_points*num_extrap_freqs*sizeof(user_precision_t),
                                                      hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(extrap_flux_Q, d_components.extrap_stokesQ,
            d_chunked_source->n_points*num_extrap_freqs*sizeof(user_precision_t),
                                                      hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(extrap_flux_U, d_components.extrap_stokesU,
            d_chunked_source->n_points*num_extrap_freqs*sizeof(user_precision_t),
                                                      hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(extrap_flux_V, d_components.extrap_stokesV,
            d_chunked_source->n_points*num_extrap_freqs*sizeof(user_precision_t),
                                                      hipMemcpyDeviceToHost ));
  //
  cudaErrorCheckCall( hipFree( d_extrap_freqs ) );
  free_extrapolated_flux_arrays(&d_chunked_source->point_components);
}


__global__ void kern_calc_measurement_equation(int num_components, int num_baselines,
          user_precision_t *d_us, user_precision_t *d_vs, user_precision_t *d_ws,
          double *d_ls, double *d_ms, double *d_ns, cuUserComplex *d_visis) {

  // Start by computing which baseline we're going to do
  const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);
  const int iComponent = threadIdx.y + (blockDim.y*blockIdx.y);

  if(iComponent < num_components && iBaseline < num_baselines) {

    cuUserComplex visi;
    visi = calc_measurement_equation(d_us, d_vs, d_ws, d_ls, d_ms, d_ns,
                                     iBaseline, iComponent);

    int visi_ind = num_components*iBaseline + iComponent;
    d_visis[visi_ind] = visi;

  }
}

extern "C" void test_kern_calc_measurement_equation(int num_components,
          int num_baselines,
          user_precision_t *us, user_precision_t *vs, user_precision_t *ws,
          double *ls, double *ms, double *ns, user_precision_complex_t *visis){

  user_precision_t *d_us = NULL;
  user_precision_t *d_vs = NULL;
  user_precision_t *d_ws = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_us, num_baselines*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_vs, num_baselines*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_ws, num_baselines*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMemcpy(d_us, us, num_baselines*sizeof(user_precision_t),
                                                        hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_vs, vs, num_baselines*sizeof(user_precision_t),
                                                        hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_ws, ws, num_baselines*sizeof(user_precision_t),
                                                        hipMemcpyHostToDevice ));

  double *d_ls = NULL;
  double *d_ms = NULL;
  double *d_ns = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_ls, num_components*sizeof(double) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_ms, num_components*sizeof(double) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_ns, num_components*sizeof(double) ));
  cudaErrorCheckCall( hipMemcpy(d_ls, ls, num_components*sizeof(double),
                                                      hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_ms, ms, num_components*sizeof(double),
                                                      hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_ns, ns, num_components*sizeof(double),
                                                      hipMemcpyHostToDevice ));

  user_precision_complex_t *d_visis = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_visis, num_baselines*num_components*sizeof(user_precision_complex_t) ));

  dim3 grid, threads;

  threads.x = 16;
  threads.y = 16;
  grid.x = (int)ceilf( (float)num_baselines / (float)threads.x );
  grid.y = (int)ceilf( (float)num_components / (float)threads.y );

  cudaErrorCheckKernel("kern_calc_measurement_equation",
                      kern_calc_measurement_equation, grid, threads,
                      num_components, num_baselines,
                      d_us, d_vs, d_ws,
                      d_ls, d_ms, d_ns,
                      (cuUserComplex*)d_visis );

  cudaErrorCheckCall( hipMemcpy(visis, (user_precision_complex_t*)d_visis, num_components*num_baselines*sizeof(user_precision_complex_t),hipMemcpyDeviceToHost ));

  cudaErrorCheckCall( hipFree( d_us ) );
  cudaErrorCheckCall( hipFree( d_vs ) );
  cudaErrorCheckCall( hipFree( d_ws ) );
  cudaErrorCheckCall( hipFree( d_ls ) );
  cudaErrorCheckCall( hipFree( d_ms ) );
  cudaErrorCheckCall( hipFree( d_ns ) );
  cudaErrorCheckCall( hipFree(d_visis ) );

}

__global__ void kern_apply_beam_gains(int num_gains, cuUserComplex *d_g1xs,
          cuUserComplex *d_D1xs,
          cuUserComplex *d_D1ys, cuUserComplex *d_g1ys,
          cuUserComplex *d_g2xs, cuUserComplex *d_D2xs,
          cuUserComplex *d_D2ys, cuUserComplex *d_g2ys,
          user_precision_t *d_flux_Is, user_precision_t *d_flux_Qs,
          user_precision_t *d_flux_Us, user_precision_t *d_flux_Vs,
          cuUserComplex *d_visi_components,
          cuUserComplex *d_visi_XXs, cuUserComplex *d_visi_XYs,
          cuUserComplex *d_visi_YXs, cuUserComplex *d_visi_YYs) {

  const int iGain = threadIdx.x + (blockDim.x*blockIdx.x);
  // const int iComponent = threadIdx.y + (blockDim.y*blockIdx.y);
  // if(iBaseline < num_visis && iComponent < num_points) {
  if (iGain < num_gains) {

    cuUserComplex visi_XX;
    cuUserComplex visi_XY;
    cuUserComplex visi_YX;
    cuUserComplex visi_YY;

    apply_beam_gains(d_g1xs[iGain], d_D1xs[iGain],
             d_D1ys[iGain], d_g1ys[iGain],
             d_g2xs[iGain], d_D2xs[iGain],
             d_D2ys[iGain], d_g2ys[iGain],
             d_flux_Is[iGain], d_flux_Qs[iGain],
             d_flux_Us[iGain], d_flux_Vs[iGain],
             d_visi_components[iGain],
             &visi_XX, &visi_XY,
             &visi_YX, &visi_YY);

    d_visi_XXs[iGain] = visi_XX;
    d_visi_XYs[iGain] = visi_XY;
    d_visi_YXs[iGain] = visi_YX;
    d_visi_YYs[iGain] = visi_YY;

  }
}

extern "C" void test_kern_apply_beam_gains(int num_gains, user_precision_complex_t *g1xs,
          user_precision_complex_t *D1xs,
          user_precision_complex_t *D1ys, user_precision_complex_t *g1ys,
          user_precision_complex_t *g2xs, user_precision_complex_t *D2xs,
          user_precision_complex_t *D2ys, user_precision_complex_t *g2ys,
          user_precision_t *flux_Is, user_precision_t *flux_Qs,
          user_precision_t *flux_Us, user_precision_t *flux_Vs,
          user_precision_complex_t *visi_components,
          user_precision_complex_t *visi_XXs, user_precision_complex_t *visi_XYs,
          user_precision_complex_t *visi_YXs, user_precision_complex_t *visi_YYs){

  user_precision_complex_t *d_g1xs = NULL;
  user_precision_complex_t *d_D1xs = NULL;
  user_precision_complex_t *d_D1ys = NULL;
  user_precision_complex_t *d_g1ys = NULL;
  user_precision_complex_t *d_g2xs = NULL;
  user_precision_complex_t *d_D2xs = NULL;
  user_precision_complex_t *d_D2ys = NULL;
  user_precision_complex_t *d_g2ys = NULL;
  user_precision_t *d_flux_Is = NULL;
  user_precision_t *d_flux_Qs = NULL;
  user_precision_t *d_flux_Us = NULL;
  user_precision_t *d_flux_Vs = NULL;
  user_precision_complex_t *d_visi_components = NULL;
  user_precision_complex_t *d_visi_XXs = NULL;
  user_precision_complex_t *d_visi_XYs = NULL;
  user_precision_complex_t *d_visi_YXs = NULL;
  user_precision_complex_t *d_visi_YYs = NULL;

  cudaErrorCheckCall( hipMalloc( (void**)&d_g1xs,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_D1xs,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_D1ys,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_g1ys,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_g2xs,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_D2xs,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_D2ys,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_g2ys,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_flux_Is,
                                          num_gains*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_flux_Qs,
                                          num_gains*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_flux_Us,
                                          num_gains*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_flux_Vs,
                                          num_gains*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_visi_components,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_visi_XXs,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_visi_XYs,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_visi_YXs,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_visi_YYs,
                                  num_gains*sizeof(user_precision_complex_t) ));

  cudaErrorCheckCall( hipMemcpy(d_g1xs, g1xs,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_D1xs, D1xs,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_D1ys, D1ys,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_g1ys, g1ys,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_g2xs, g2xs,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_D2xs, D2xs,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_D2ys, D2ys,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_g2ys, g2ys,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_visi_components, visi_components,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_visi_XXs, visi_XXs,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_visi_XYs, visi_XYs,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_visi_YXs, visi_YXs,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_visi_YYs, visi_YYs,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));

  cudaErrorCheckCall( hipMemcpy(d_flux_Is, flux_Is,
                             num_gains*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_flux_Qs, flux_Qs,
                             num_gains*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_flux_Us, flux_Us,
                             num_gains*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_flux_Vs, flux_Vs,
                             num_gains*sizeof(user_precision_t), hipMemcpyHostToDevice ));

  dim3 grid, threads;

  threads.x = 128;
  grid.x = (int)ceil( (user_precision_t)num_gains / (user_precision_t)threads.x );

  cudaErrorCheckKernel("kern_apply_beam_gains",
                      kern_apply_beam_gains, grid, threads,
                      num_gains,
                      (cuUserComplex *)d_g1xs, (cuUserComplex *)d_D1xs,
                      (cuUserComplex *)d_D1ys, (cuUserComplex *)d_g1ys,
                      (cuUserComplex *)d_g2xs, (cuUserComplex *)d_D2xs,
                      (cuUserComplex *)d_D2ys, (cuUserComplex *)d_g2ys,
                      d_flux_Is, d_flux_Qs,
                      d_flux_Us, d_flux_Vs,
                      (cuUserComplex *)d_visi_components,
                      (cuUserComplex *)d_visi_XXs, (cuUserComplex *)d_visi_XYs,
                      (cuUserComplex *)d_visi_YXs, (cuUserComplex *)d_visi_YYs );

  cudaErrorCheckCall( hipMemcpy(visi_XXs, d_visi_XXs,
           num_gains*sizeof(user_precision_complex_t),hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(visi_XYs, d_visi_XYs,
           num_gains*sizeof(user_precision_complex_t),hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(visi_YXs, d_visi_YXs,
           num_gains*sizeof(user_precision_complex_t),hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(visi_YYs, d_visi_YYs,
           num_gains*sizeof(user_precision_complex_t),hipMemcpyDeviceToHost ));

  cudaErrorCheckCall( hipFree( d_g1xs ) );
  cudaErrorCheckCall( hipFree( d_D1xs ) );
  cudaErrorCheckCall( hipFree( d_D1ys ) );
  cudaErrorCheckCall( hipFree( d_g1ys ) );
  cudaErrorCheckCall( hipFree( d_g2xs ) );
  cudaErrorCheckCall( hipFree( d_D2xs ) );
  cudaErrorCheckCall( hipFree( d_D2ys ) );
  cudaErrorCheckCall( hipFree( d_g2ys ) );
  cudaErrorCheckCall( hipFree( d_flux_Is ) );
  cudaErrorCheckCall( hipFree( d_flux_Qs ) );
  cudaErrorCheckCall( hipFree( d_flux_Us ) );
  cudaErrorCheckCall( hipFree( d_flux_Vs ) );
  cudaErrorCheckCall( hipFree( d_visi_components ) );
  cudaErrorCheckCall( hipFree( d_visi_XXs ) );
  cudaErrorCheckCall( hipFree( d_visi_XYs ) );
  cudaErrorCheckCall( hipFree( d_visi_YXs ) );
  cudaErrorCheckCall( hipFree( d_visi_YYs ) );

}

__global__ void kern_get_beam_gains(int num_components, int num_baselines,
           int num_freqs, int num_visis, int num_times, int beamtype,
           cuUserComplex *d_primay_beam_J00, cuUserComplex *d_primay_beam_J01,
           cuUserComplex *d_primay_beam_J10, cuUserComplex *d_primay_beam_J11,
           cuUserComplex *d_recov_g1x, cuUserComplex *d_recov_D1x,
           cuUserComplex *d_recov_D1y, cuUserComplex *d_recov_g1y,
           cuUserComplex *d_recov_g2x, cuUserComplex *d_recov_D2x,
           cuUserComplex *d_recov_D2y, cuUserComplex *d_recov_g2y) {

  // Start by computing which baseline we're going to do
  const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);
  // const int iComponent = threadIdx.y + (blockDim.y*blockIdx.y);
  // if(iBaseline < num_visis && iComponent < num_points) {
  if(iBaseline < num_visis) {

    for (int iComponent = 0; iComponent < num_components; iComponent++) {

      cuUserComplex g1x;
      cuUserComplex D1x;
      cuUserComplex D1y;
      cuUserComplex g1y;
      cuUserComplex g2x;
      cuUserComplex D2x;
      cuUserComplex D2y;
      cuUserComplex g2y;

      get_beam_gains(iBaseline, iComponent, num_freqs,
                 num_baselines, num_components, num_times, beamtype,
                 d_primay_beam_J00, d_primay_beam_J01,
                 d_primay_beam_J10, d_primay_beam_J11,
                 &g1x, &D1x, &D1y, &g1y, &g2x, &D2x, &D2y, &g2y);

      int out_ind = num_visis*iComponent + iBaseline;

      d_recov_g1x[out_ind] = g1x;
      d_recov_D1x[out_ind] = D1x;
      d_recov_D1y[out_ind] = D1y;
      d_recov_g1y[out_ind] = g1y;
      d_recov_g2x[out_ind] = g2x;
      d_recov_D2x[out_ind] = D2x;
      d_recov_D2y[out_ind] = D2y;
      d_recov_g2y[out_ind] = g2y;

    }
  }
}

extern "C" void test_kern_get_beam_gains(int num_freqs, int num_visis,
          int num_baselines, int num_components, int num_times, int beamtype,
          user_precision_complex_t *primay_beam_J00, user_precision_complex_t *primay_beam_J01,
          user_precision_complex_t *primay_beam_J10, user_precision_complex_t *primay_beam_J11,
          user_precision_complex_t *recover_g1x, user_precision_complex_t *recover_D1x,
          user_precision_complex_t *recover_D1y, user_precision_complex_t *recover_g1y,
          user_precision_complex_t *recover_g2x, user_precision_complex_t *recover_D2x,
          user_precision_complex_t *recover_D2y, user_precision_complex_t *recover_g2y){

  user_precision_complex_t *d_recover_g1x = NULL;
  user_precision_complex_t *d_recover_D1x = NULL;
  user_precision_complex_t *d_recover_D1y = NULL;
  user_precision_complex_t *d_recover_g1y = NULL;
  user_precision_complex_t *d_recover_g2x = NULL;
  user_precision_complex_t *d_recover_D2x = NULL;
  user_precision_complex_t *d_recover_D2y = NULL;
  user_precision_complex_t *d_recover_g2y = NULL;

  user_precision_complex_t *d_primay_beam_J00 = NULL;
  user_precision_complex_t *d_primay_beam_J01 = NULL;
  user_precision_complex_t *d_primay_beam_J10 = NULL;
  user_precision_complex_t *d_primay_beam_J11 = NULL;

  cudaErrorCheckCall( hipMalloc( (void**)&d_recover_g1x, num_components*num_visis*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_recover_D1x, num_components*num_visis*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_recover_D1y, num_components*num_visis*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_recover_g1y, num_components*num_visis*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_recover_g2x, num_components*num_visis*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_recover_D2x, num_components*num_visis*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_recover_D2y, num_components*num_visis*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_recover_g2y, num_components*num_visis*sizeof(user_precision_complex_t) ));

  cudaErrorCheckCall( hipMalloc( (void**)&d_primay_beam_J00, num_freqs*num_times*num_components*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_primay_beam_J01, num_freqs*num_times*num_components*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_primay_beam_J10, num_freqs*num_times*num_components*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_primay_beam_J11, num_freqs*num_times*num_components*sizeof(user_precision_complex_t) ));

  cudaErrorCheckCall( hipMemcpy(d_primay_beam_J00, primay_beam_J00, num_freqs*num_times*num_components*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_primay_beam_J01, primay_beam_J01, num_freqs*num_times*num_components*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_primay_beam_J10, primay_beam_J10, num_freqs*num_times*num_components*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_primay_beam_J11, primay_beam_J11, num_freqs*num_times*num_components*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));

  dim3 grid, threads;

  threads.x = 128;
  grid.x = (int)ceil( (user_precision_t)num_visis / (user_precision_t)threads.x );

  cudaErrorCheckKernel("kern_get_beam_gains",
                      kern_get_beam_gains, grid, threads,
                      num_components, num_baselines,
                      num_freqs, num_visis, num_times, beamtype,
                      (cuUserComplex *)d_primay_beam_J00,
                      (cuUserComplex *)d_primay_beam_J01,
                      (cuUserComplex *)d_primay_beam_J10,
                      (cuUserComplex *)d_primay_beam_J11,
                      (cuUserComplex *)d_recover_g1x, (cuUserComplex *)d_recover_D1x,
                      (cuUserComplex *)d_recover_D1y, (cuUserComplex *)d_recover_g1y,
                      (cuUserComplex *)d_recover_g2x, (cuUserComplex *)d_recover_D2x,
                      (cuUserComplex *)d_recover_D2y, (cuUserComplex *)d_recover_g2y );

  cudaErrorCheckCall( hipMemcpy(recover_g1x, d_recover_g1x, num_components*num_visis*sizeof(user_precision_complex_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(recover_D1x, d_recover_D1x, num_components*num_visis*sizeof(user_precision_complex_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(recover_D1y, d_recover_D1y, num_components*num_visis*sizeof(user_precision_complex_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(recover_g1y, d_recover_g1y, num_components*num_visis*sizeof(user_precision_complex_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(recover_g2x, d_recover_g2x, num_components*num_visis*sizeof(user_precision_complex_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(recover_D2x, d_recover_D2x, num_components*num_visis*sizeof(user_precision_complex_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(recover_D2y, d_recover_D2y, num_components*num_visis*sizeof(user_precision_complex_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(recover_g2y, d_recover_g2y, num_components*num_visis*sizeof(user_precision_complex_t), hipMemcpyDeviceToHost ));

  cudaErrorCheckCall( hipFree( d_recover_g1x ) );
  cudaErrorCheckCall( hipFree( d_recover_D1x ) );
  cudaErrorCheckCall( hipFree( d_recover_D1y ) );
  cudaErrorCheckCall( hipFree( d_recover_g1y ) );
  cudaErrorCheckCall( hipFree( d_recover_g2x ) );
  cudaErrorCheckCall( hipFree( d_recover_D2x ) );
  cudaErrorCheckCall( hipFree( d_recover_D2y ) );
  cudaErrorCheckCall( hipFree( d_recover_g2y ) );

  cudaErrorCheckCall( hipFree( d_primay_beam_J00 ) );
  cudaErrorCheckCall( hipFree( d_primay_beam_J01 ) );
  cudaErrorCheckCall( hipFree( d_primay_beam_J10 ) );
  cudaErrorCheckCall( hipFree( d_primay_beam_J11 ) );

}

__global__ void kern_update_sum_visis(int num_freqs,
     int num_baselines, int num_components, int num_times, int beamtype,
     cuUserComplex *d_primay_beam_J00, cuUserComplex *d_primay_beam_J01,
     cuUserComplex *d_primay_beam_J10, cuUserComplex *d_primay_beam_J11,
     cuUserComplex *d_visi_components,
     user_precision_t *d_flux_I, user_precision_t *d_flux_Q,
     user_precision_t *d_flux_U, user_precision_t *d_flux_V,
     user_precision_t *d_sum_visi_XX_real, user_precision_t *d_sum_visi_XX_imag,
     user_precision_t *d_sum_visi_XY_real, user_precision_t *d_sum_visi_XY_imag,
     user_precision_t *d_sum_visi_YX_real, user_precision_t *d_sum_visi_YX_imag,
     user_precision_t *d_sum_visi_YY_real, user_precision_t *d_sum_visi_YY_imag) {

  // Start by computing which baseline we're going to do
  const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);

  if(iBaseline < num_freqs*num_baselines*num_times) {

    int time_ind = (int)floorf( (user_precision_t)iBaseline / ((user_precision_t)num_baselines * (user_precision_t)num_freqs));
    int freq_ind = (int)floorf( ((user_precision_t)iBaseline - ((user_precision_t)time_ind*(user_precision_t)num_baselines * (user_precision_t)num_freqs)) / (user_precision_t)num_baselines);

    for (int iComponent = 0; iComponent < num_components; iComponent++) {

      //There is a flux for every frequnecy and component
      int flux_ind = num_components*freq_ind + iComponent;

      update_sum_visis(iBaseline, iComponent, num_freqs,
             num_baselines, num_components, num_times, beamtype,
             d_primay_beam_J00, d_primay_beam_J01,
             d_primay_beam_J10, d_primay_beam_J11,
             d_visi_components[iBaseline],
             d_flux_I[flux_ind], d_flux_Q[flux_ind],
             d_flux_U[flux_ind], d_flux_V[flux_ind],
             d_sum_visi_XX_real, d_sum_visi_XX_imag,
             d_sum_visi_XY_real, d_sum_visi_XY_imag,
             d_sum_visi_YX_real, d_sum_visi_YX_imag,
             d_sum_visi_YY_real, d_sum_visi_YY_imag);

    }
  }
}

extern "C" void test_kern_update_sum_visis(int num_freqs, int num_visis,
          int num_baselines, int num_components, int num_times, int beamtype,
          user_precision_complex_t *primay_beam_J00,
          user_precision_complex_t *primay_beam_J01,
          user_precision_complex_t *primay_beam_J10,
          user_precision_complex_t *primay_beam_J11,
          user_precision_complex_t *visi_components,
          user_precision_t *flux_I, user_precision_t *flux_Q,
          user_precision_t *flux_U, user_precision_t *flux_V,
          user_precision_t *sum_visi_XX_real, user_precision_t *sum_visi_XX_imag,
          user_precision_t *sum_visi_XY_real, user_precision_t *sum_visi_XY_imag,
          user_precision_t *sum_visi_YX_real, user_precision_t *sum_visi_YX_imag,
          user_precision_t *sum_visi_YY_real, user_precision_t *sum_visi_YY_imag){

  user_precision_complex_t *d_primay_beam_J00 = NULL;
  user_precision_complex_t *d_primay_beam_J01 = NULL;
  user_precision_complex_t *d_primay_beam_J10 = NULL;
  user_precision_complex_t *d_primay_beam_J11 = NULL;
  user_precision_complex_t *d_visi_components = NULL;

  cudaErrorCheckCall( hipMalloc( (void**)&d_primay_beam_J00,
                    num_components*num_times*num_freqs*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_primay_beam_J01,
                    num_components*num_times*num_freqs*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_primay_beam_J10,
                    num_components*num_times*num_freqs*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_primay_beam_J11,
                    num_components*num_times*num_freqs*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_visi_components,
                    num_visis*sizeof(user_precision_complex_t) ));

  cudaErrorCheckCall( hipMemcpy(d_primay_beam_J00, primay_beam_J00,
            num_components*num_times*num_freqs*sizeof(user_precision_complex_t),
            hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_primay_beam_J01, primay_beam_J01,
            num_components*num_times*num_freqs*sizeof(user_precision_complex_t),
            hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_primay_beam_J10, primay_beam_J10,
            num_components*num_times*num_freqs*sizeof(user_precision_complex_t),
            hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_primay_beam_J11, primay_beam_J11,
            num_components*num_times*num_freqs*sizeof(user_precision_complex_t),
            hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_visi_components, visi_components,
                                     num_visis*sizeof(user_precision_complex_t),
                                     hipMemcpyHostToDevice ));

  user_precision_t *d_flux_I = NULL;
  user_precision_t *d_flux_Q = NULL;
  user_precision_t *d_flux_U = NULL;
  user_precision_t *d_flux_V = NULL;

  cudaErrorCheckCall( hipMalloc( (void**)&d_flux_I, num_components*num_times*num_freqs*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_flux_Q, num_components*num_times*num_freqs*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_flux_U, num_components*num_times*num_freqs*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_flux_V, num_components*num_times*num_freqs*sizeof(user_precision_t) ));

  cudaErrorCheckCall( hipMemcpy(d_flux_I, flux_I,
                    num_components*num_times*num_freqs*sizeof(user_precision_t),    hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_flux_Q, flux_Q,
                    num_components*num_times*num_freqs*sizeof(user_precision_t),    hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_flux_U, flux_U,
                    num_components*num_times*num_freqs*sizeof(user_precision_t),    hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_flux_V, flux_V,
                    num_components*num_times*num_freqs*sizeof(user_precision_t),    hipMemcpyHostToDevice ));

  user_precision_t *d_sum_visi_XX_real = NULL;
  user_precision_t *d_sum_visi_XY_real = NULL;
  user_precision_t *d_sum_visi_YX_real = NULL;
  user_precision_t *d_sum_visi_YY_real = NULL;
  user_precision_t *d_sum_visi_XX_imag = NULL;
  user_precision_t *d_sum_visi_XY_imag = NULL;
  user_precision_t *d_sum_visi_YX_imag = NULL;
  user_precision_t *d_sum_visi_YY_imag = NULL;

  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_XX_real,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_XY_real,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_YX_real,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_YY_real,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_XX_imag,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_XY_imag,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_YX_imag,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_YY_imag,
                                          num_visis*sizeof(user_precision_t) ));

  dim3 grid, threads;

  threads.x = 128;
  grid.x = (int)ceil( (user_precision_t)num_visis / (user_precision_t)threads.x );

  cudaErrorCheckKernel("kern_update_sum_visis",
                      kern_update_sum_visis, grid, threads,
                      num_freqs, num_baselines, num_components, num_times, beamtype,
                      (cuUserComplex *)d_primay_beam_J00, (cuUserComplex *)d_primay_beam_J01,
                      (cuUserComplex *)d_primay_beam_J10, (cuUserComplex *)d_primay_beam_J11,
                      (cuUserComplex *)d_visi_components,
                      d_flux_I, d_flux_Q, d_flux_U, d_flux_V,
                      d_sum_visi_XX_real, d_sum_visi_XX_imag,
                      d_sum_visi_XY_real, d_sum_visi_XY_imag,
                      d_sum_visi_YX_real, d_sum_visi_YX_imag,
                      d_sum_visi_YY_real, d_sum_visi_YY_imag );

  cudaErrorCheckCall( hipMemcpy(sum_visi_XX_real, d_sum_visi_XX_real,
                  num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_XY_real, d_sum_visi_XY_real,
                  num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_YX_real, d_sum_visi_YX_real,
                  num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_YY_real, d_sum_visi_YY_real,
                  num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_XX_imag, d_sum_visi_XX_imag,
                  num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_XY_imag, d_sum_visi_XY_imag,
                  num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_YX_imag, d_sum_visi_YX_imag,
                  num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_YY_imag, d_sum_visi_YY_imag,
                  num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));

  cudaErrorCheckCall( hipFree( d_primay_beam_J00 ) );
  cudaErrorCheckCall( hipFree( d_primay_beam_J01 ) );
  cudaErrorCheckCall( hipFree( d_primay_beam_J10 ) );
  cudaErrorCheckCall( hipFree( d_primay_beam_J11 ) );
  cudaErrorCheckCall( hipFree( d_visi_components ) );
  cudaErrorCheckCall( hipFree( d_flux_I ) );
  cudaErrorCheckCall( hipFree( d_flux_Q ) );
  cudaErrorCheckCall( hipFree( d_flux_U ) );
  cudaErrorCheckCall( hipFree( d_flux_V ) );
  cudaErrorCheckCall( hipFree( d_sum_visi_XX_real ) );
  cudaErrorCheckCall( hipFree( d_sum_visi_XY_real ) );
  cudaErrorCheckCall( hipFree( d_sum_visi_YX_real ) );
  cudaErrorCheckCall( hipFree( d_sum_visi_YY_real ) );
  cudaErrorCheckCall( hipFree( d_sum_visi_XX_imag ) );
  cudaErrorCheckCall( hipFree( d_sum_visi_XY_imag ) );
  cudaErrorCheckCall( hipFree( d_sum_visi_YX_imag ) );
  cudaErrorCheckCall( hipFree( d_sum_visi_YY_imag ) );

}


extern "C" void test_source_component_common(int num_of_each_flux_type,
           components_t components,
           double *freqs, woden_settings_t *woden_settings,
           beam_settings_t *beam_settings,
           user_precision_complex_t *gxs, user_precision_complex_t *Dxs,
           user_precision_complex_t *Dys, user_precision_complex_t *gys,
           user_precision_t *extrap_flux_I, user_precision_t *extrap_flux_Q,
           user_precision_t *extrap_flux_U, user_precision_t *extrap_flux_V,
           double *ls, double *ms, double *ns,
           e_component_type comptype){

  source_t *chunked_source = (source_t *)malloc(sizeof(source_t));

  //TODODOD have a if (comptype == POINT) etc here so we can use same
  //componenets to test all POINT, GAUSSIAN, SHAPELET

  int NUM_FLUX_TYPES = 3;

  if (comptype == POINT) {
    chunked_source->point_components = components;
    chunked_source->n_points = NUM_FLUX_TYPES*num_of_each_flux_type;
    chunked_source->n_point_powers = num_of_each_flux_type;
    chunked_source->n_point_curves = num_of_each_flux_type;
    chunked_source->n_point_lists = num_of_each_flux_type;

    chunked_source->n_gauss = 0;
    chunked_source->n_gauss_lists = 0;
    chunked_source->n_gauss_powers = 0;
    chunked_source->n_gauss_curves = 0;
    chunked_source->n_shapes = 0;
    chunked_source->n_shape_lists = 0;
    chunked_source->n_shape_powers = 0;
    chunked_source->n_shape_curves = 0;
    chunked_source->n_shape_coeffs = 0;
  }
  else if (comptype == GAUSSIAN) {
    chunked_source->gauss_components = components;
    chunked_source->n_gauss = NUM_FLUX_TYPES*num_of_each_flux_type;
    chunked_source->n_gauss_powers = num_of_each_flux_type;
    chunked_source->n_gauss_curves = num_of_each_flux_type;
    chunked_source->n_gauss_lists = num_of_each_flux_type;

    chunked_source->n_points = 0;
    chunked_source->n_point_lists = 0;
    chunked_source->n_point_powers = 0;
    chunked_source->n_point_curves = 0;
    chunked_source->n_shapes = 0;
    chunked_source->n_shape_lists = 0;
    chunked_source->n_shape_powers = 0;
    chunked_source->n_shape_curves = 0;
    chunked_source->n_shape_coeffs = 0;
  }
  else if (comptype == SHAPELET) {
    chunked_source->shape_components = components;
    chunked_source->n_shapes = NUM_FLUX_TYPES*num_of_each_flux_type;
    chunked_source->n_shape_powers = num_of_each_flux_type;
    chunked_source->n_shape_curves = num_of_each_flux_type;
    chunked_source->n_shape_lists = num_of_each_flux_type;
    chunked_source->n_shape_coeffs = num_of_each_flux_type;

    chunked_source->n_points = 0;
    chunked_source->n_point_lists = 0;
    chunked_source->n_point_powers = 0;
    chunked_source->n_point_curves = 0;
    chunked_source->n_gauss = 0;
    chunked_source->n_gauss_lists = 0;
    chunked_source->n_gauss_powers = 0;
    chunked_source->n_gauss_curves = 0;

  }

  source_t *d_chunked_source = copy_chunked_source_to_GPU(chunked_source);

  double *d_freqs = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_freqs,
                                     woden_settings->num_freqs*sizeof(double) ));
  cudaErrorCheckCall( hipMemcpy( d_freqs, freqs,
             woden_settings->num_freqs*sizeof(double), hipMemcpyHostToDevice) );

  d_beam_gains_t d_beam_gains;

  source_component_common(woden_settings, beam_settings, d_freqs,
       chunked_source, d_chunked_source, &d_beam_gains, comptype);

  int num_beam_values = NUM_FLUX_TYPES*num_of_each_flux_type*woden_settings->num_freqs*woden_settings->num_time_steps;

  cudaErrorCheckCall( hipMemcpy(gxs, (user_precision_complex_t*)d_beam_gains.d_gxs,
              num_beam_values*sizeof(cuUserComplex), hipMemcpyDeviceToHost ));

  cudaErrorCheckCall( hipMemcpy(gys, (user_precision_complex_t*)d_beam_gains.d_gys,
              num_beam_values*sizeof(cuUserComplex), hipMemcpyDeviceToHost ));

  if (beam_settings->beamtype == FEE_BEAM || beam_settings->beamtype == FEE_BEAM_INTERP || beam_settings->beamtype == MWA_ANALY) {
    cudaErrorCheckCall( hipMemcpy(Dxs, (user_precision_complex_t*)d_beam_gains.d_Dxs,
                num_beam_values*sizeof(cuUserComplex), hipMemcpyDeviceToHost ));
    cudaErrorCheckCall( hipMemcpy(Dys, (user_precision_complex_t*)d_beam_gains.d_Dys,
                num_beam_values*sizeof(cuUserComplex), hipMemcpyDeviceToHost ));
  }

  //Just a little shorthand so don't have to keep writing out as much in the
  //memcpy below

  components_t d_components;

  if (comptype == POINT) {
    d_components = d_chunked_source->point_components;
  }
  else if (comptype == GAUSSIAN) {
    d_components = d_chunked_source->gauss_components;
  }
  else {
    d_components = d_chunked_source->shape_components;
  }


  cudaErrorCheckCall( hipMemcpy(ls, d_components.ls,
                            NUM_FLUX_TYPES*num_of_each_flux_type*sizeof(double),
                            hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(ms, d_components.ms,
                            NUM_FLUX_TYPES*num_of_each_flux_type*sizeof(double),
                            hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(ns, d_components.ns,
                            NUM_FLUX_TYPES*num_of_each_flux_type*sizeof(double),
                            hipMemcpyDeviceToHost ));

  cudaErrorCheckCall( hipMemcpy(extrap_flux_I, d_components.extrap_stokesI,
  NUM_FLUX_TYPES*num_of_each_flux_type*woden_settings->num_freqs*sizeof(user_precision_t),
                                                      hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(extrap_flux_Q, d_components.extrap_stokesQ,
  NUM_FLUX_TYPES*num_of_each_flux_type*woden_settings->num_freqs*sizeof(user_precision_t),
                                                      hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(extrap_flux_U, d_components.extrap_stokesU,
  NUM_FLUX_TYPES*num_of_each_flux_type*woden_settings->num_freqs*sizeof(user_precision_t),
                                                      hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(extrap_flux_V, d_components.extrap_stokesV,
  NUM_FLUX_TYPES*num_of_each_flux_type*woden_settings->num_freqs*sizeof(user_precision_t),
                                                      hipMemcpyDeviceToHost ));

  cudaErrorCheckCall( hipFree( d_freqs ) );
  free_extrapolated_flux_arrays(&d_components);
  free_d_components(d_chunked_source, comptype);
  free_beam_gains(d_beam_gains, beam_settings->beamtype);
}


void malloc_lmn_arrays(source_t *d_chunked_source, components_t *components,
                        int num_components, e_component_type comptype){
  components_t *d_components;
  if (comptype == POINT) {
    d_components = &d_chunked_source->point_components;
  } else if (comptype == GAUSSIAN) {
    d_components = &d_chunked_source->gauss_components;
  } else if (comptype == SHAPELET) {
    d_components = &d_chunked_source->shape_components;
  }

  cudaErrorCheckCall( hipMalloc( (void**)&d_components->ls,
                                          num_components*sizeof(double) ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_components->ms,
                                          num_components*sizeof(double) ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_components->ns,
                                          num_components*sizeof(double) ) );

  cudaErrorCheckCall( hipMemcpy(d_components->ls, components->ls, num_components*sizeof(double),
                                           hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_components->ms, components->ms, num_components*sizeof(double),
                                           hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_components->ns, components->ns, num_components*sizeof(double),
                                           hipMemcpyHostToDevice ));

}

extern "C" void test_kern_calc_visi_all(int n_powers, int n_curves, int n_lists,
          int num_baselines, int num_shape_coeffs,
          int num_freqs, int num_visis, int num_times,
          e_beamtype beamtype, e_component_type comptype,
          components_t components, double *extrap_freqs,
          user_precision_t *us, user_precision_t *vs, user_precision_t *ws,
          user_precision_t *u_shapes, user_precision_t *v_shapes, user_precision_t *w_shapes,
          user_precision_t *sum_visi_XX_real, user_precision_t *sum_visi_XX_imag,
          user_precision_t *sum_visi_XY_real, user_precision_t *sum_visi_XY_imag,
          user_precision_t *sum_visi_YX_real, user_precision_t *sum_visi_YX_imag,
          user_precision_t *sum_visi_YY_real, user_precision_t *sum_visi_YY_imag,
          user_precision_t *allsteps_wavelengths, user_precision_t *sbf,
          user_precision_complex_t *gxs, user_precision_complex_t *Dxs,
          user_precision_complex_t *Dys, user_precision_complex_t *gys){

  int num_components = n_powers + n_curves + n_lists;

  user_precision_t *d_us = NULL;
  user_precision_t *d_vs = NULL;
  user_precision_t *d_ws = NULL;
  user_precision_t *d_allsteps_wavelengths = NULL;

  cudaErrorCheckCall( hipMalloc( (void**)&d_us, num_visis*sizeof(user_precision_t) ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_vs, num_visis*sizeof(user_precision_t) ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_ws, num_visis*sizeof(user_precision_t) ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_allsteps_wavelengths, num_visis*sizeof(user_precision_t) ) );

  cudaErrorCheckCall( hipMemcpy(d_us, us,
                             num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_vs, vs,
                             num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_ws, ws,
                             num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_allsteps_wavelengths, allsteps_wavelengths,
                             num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ));

  //Here are many things that would have been done by source_component_common
  source_t *chunked_source = (source_t *)malloc(sizeof(source_t));

  double *d_extrap_freqs = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_extrap_freqs,
                                   num_freqs*sizeof(double) ));
  cudaErrorCheckCall( hipMemcpy(d_extrap_freqs, extrap_freqs,
             num_freqs*sizeof(double), hipMemcpyHostToDevice ));

  source_t *d_chunked_source = NULL;
  components_t d_components;

  if (comptype == POINT) {

    chunked_source->point_components = components;
    chunked_source->n_points = n_powers + n_curves + n_lists;
    chunked_source->n_point_powers = n_powers;
    chunked_source->n_point_curves = n_curves;
    chunked_source->n_point_lists = n_lists;

    chunked_source->n_gauss = 0;
    chunked_source->n_gauss_lists = 0;
    chunked_source->n_gauss_powers = 0;
    chunked_source->n_gauss_curves = 0;
    chunked_source->n_shapes = 0;
    chunked_source->n_shape_lists = 0;
    chunked_source->n_shape_powers = 0;
    chunked_source->n_shape_curves = 0;
    chunked_source->n_shape_coeffs = 0;

    // source_t *remap_source = (source_t *)malloc(sizeof(source_t));

    // remap_source_for_gpu(remap_source, chunked_source,
                         // num_times, beamtype);

    d_chunked_source = copy_chunked_source_to_GPU(chunked_source);
    malloc_lmn_arrays(d_chunked_source, &components, num_components, comptype);

    malloc_extrapolated_flux_arrays(&d_chunked_source->point_components,
                                    d_chunked_source->n_points,
                                    num_freqs);
    extrapolate_Stokes(d_chunked_source, d_extrap_freqs, num_freqs, POINT);
    d_components = d_chunked_source->point_components;
  }
  else if (comptype == GAUSSIAN) {

    chunked_source->gauss_components = components;
    chunked_source->n_gauss = n_powers + n_curves + n_lists;
    chunked_source->n_gauss_powers = n_powers;
    chunked_source->n_gauss_curves = n_curves;
    chunked_source->n_gauss_lists = n_lists;

    chunked_source->n_points = 0;
    chunked_source->n_point_lists = 0;
    chunked_source->n_point_powers = 0;
    chunked_source->n_point_curves = 0;
    chunked_source->n_shapes = 0;
    chunked_source->n_shape_lists = 0;
    chunked_source->n_shape_powers = 0;
    chunked_source->n_shape_curves = 0;
    chunked_source->n_shape_coeffs = 0;

    d_chunked_source = copy_chunked_source_to_GPU(chunked_source);
    malloc_lmn_arrays(d_chunked_source, &components, num_components, comptype);

    malloc_extrapolated_flux_arrays(&d_chunked_source->gauss_components,
                                    d_chunked_source->n_gauss,
                                    num_freqs);
    extrapolate_Stokes(d_chunked_source, d_extrap_freqs, num_freqs, GAUSSIAN);
    d_components = d_chunked_source->gauss_components;
  }
  else if (comptype == SHAPELET) {
    chunked_source->shape_components = components;
    chunked_source->n_shapes = n_powers + n_curves + n_lists;
    chunked_source->n_shape_powers = n_powers;
    chunked_source->n_shape_curves = n_curves;
    chunked_source->n_shape_lists = n_lists;
    chunked_source->n_shape_coeffs = num_shape_coeffs;

    chunked_source->n_points = 0;
    chunked_source->n_point_lists = 0;
    chunked_source->n_point_powers = 0;
    chunked_source->n_point_curves = 0;
    chunked_source->n_gauss = 0;
    chunked_source->n_gauss_lists = 0;
    chunked_source->n_gauss_powers = 0;
    chunked_source->n_gauss_curves = 0;

    d_chunked_source = copy_chunked_source_to_GPU(chunked_source);
    malloc_lmn_arrays(d_chunked_source, &components, num_components, comptype);

    malloc_extrapolated_flux_arrays(&d_chunked_source->shape_components,
                                    d_chunked_source->n_shapes,
                                    num_freqs);
    extrapolate_Stokes(d_chunked_source, d_extrap_freqs, num_freqs, SHAPELET);
    d_components = d_chunked_source->shape_components;
  }

  //Something to store the primary beam gains (all 4 pols) in
  d_beam_gains_t d_beam_gains;
  int num_beam_values = num_components*num_freqs*num_times;

  cudaErrorCheckCall( hipMalloc( (void**)&d_beam_gains.d_gxs,
                                      num_beam_values*sizeof(cuUserComplex) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_beam_gains.d_Dxs,
                                      num_beam_values*sizeof(cuUserComplex) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_beam_gains.d_Dys,
                                      num_beam_values*sizeof(cuUserComplex) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_beam_gains.d_gys,
                                      num_beam_values*sizeof(cuUserComplex) ));

  cudaErrorCheckCall( hipMemcpy(d_beam_gains.d_gxs, (cuUserComplex *)gxs,
              num_beam_values*sizeof(cuUserComplex), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_beam_gains.d_Dxs, (cuUserComplex *)Dxs,
              num_beam_values*sizeof(cuUserComplex), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_beam_gains.d_Dys, (cuUserComplex *)Dys,
              num_beam_values*sizeof(cuUserComplex), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_beam_gains.d_gys, (cuUserComplex *)gys,
              num_beam_values*sizeof(cuUserComplex), hipMemcpyHostToDevice ));

  user_precision_t *d_sum_visi_XX_real = NULL;
  user_precision_t *d_sum_visi_XY_real = NULL;
  user_precision_t *d_sum_visi_YX_real = NULL;
  user_precision_t *d_sum_visi_YY_real = NULL;
  user_precision_t *d_sum_visi_XX_imag = NULL;
  user_precision_t *d_sum_visi_XY_imag = NULL;
  user_precision_t *d_sum_visi_YX_imag = NULL;
  user_precision_t *d_sum_visi_YY_imag = NULL;

  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_XX_real,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_XY_real,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_YX_real,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_YY_real,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_XX_imag,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_XY_imag,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_YX_imag,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_YY_imag,
                                          num_visis*sizeof(user_precision_t) ));

  //Make sure the visis start at zero by copying across host versions, which
  //should be set to zero already
  cudaErrorCheckCall( hipMemcpy( d_sum_visi_XX_real, sum_visi_XX_real,
    num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMemcpy( d_sum_visi_XY_real, sum_visi_XY_real,
    num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMemcpy( d_sum_visi_YX_real, sum_visi_YX_real,
    num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMemcpy( d_sum_visi_YY_real, sum_visi_YY_real,
    num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMemcpy( d_sum_visi_XX_imag, sum_visi_XX_imag,
    num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMemcpy( d_sum_visi_XY_imag, sum_visi_XY_imag,
    num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMemcpy( d_sum_visi_YX_imag, sum_visi_YX_imag,
    num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMemcpy( d_sum_visi_YY_imag, sum_visi_YY_imag,
    num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

  dim3 grid, threads;

  threads.x = 128;
  grid.x = (int)ceil( (float)num_visis / (float)threads.x );

  //Shapelets need many many extra things

  user_precision_t *d_sbf=NULL;
  user_precision_t *d_u_shapes = NULL;
  user_precision_t *d_v_shapes = NULL;
  user_precision_t *d_w_shapes = NULL;

  if (comptype == SHAPELET) {

    cudaErrorCheckCall( hipMalloc( (void**)&d_u_shapes,
                                     num_components*num_visis*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMalloc( (void**)&d_v_shapes,
                                     num_components*num_visis*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMalloc( (void**)&d_w_shapes,
                                     num_components*num_visis*sizeof(user_precision_t) ) );

    cudaErrorCheckCall( hipMemcpy(d_u_shapes, u_shapes,
               num_components*num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ));
    cudaErrorCheckCall( hipMemcpy(d_v_shapes, v_shapes,
               num_components*num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ));
    cudaErrorCheckCall( hipMemcpy(d_w_shapes, w_shapes,
               num_components*num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ));

    cudaErrorCheckCall( hipMalloc( (void**)&d_components.shape_coeffs,
                                                num_shape_coeffs*sizeof(user_precision_t) ));
    cudaErrorCheckCall( hipMalloc( (void**)&d_components.n1s,
                                                num_shape_coeffs*sizeof(user_precision_t) ));
    cudaErrorCheckCall( hipMalloc( (void**)&d_components.n2s,
                                                num_shape_coeffs*sizeof(user_precision_t) ));
    cudaErrorCheckCall( hipMalloc( (void**)&d_components.param_indexes,
                                                num_shape_coeffs*sizeof(user_precision_t) ));

    cudaErrorCheckCall( hipMemcpy(d_components.shape_coeffs,
                          components.shape_coeffs, num_shape_coeffs*sizeof(user_precision_t),
                          hipMemcpyHostToDevice ));
    cudaErrorCheckCall( hipMemcpy(d_components.n1s,
                          components.n1s, num_shape_coeffs*sizeof(user_precision_t),
                          hipMemcpyHostToDevice ));
    cudaErrorCheckCall( hipMemcpy(d_components.n2s,
                          components.n2s, num_shape_coeffs*sizeof(user_precision_t),
                          hipMemcpyHostToDevice ));
    cudaErrorCheckCall( hipMemcpy(d_components.param_indexes,
                          components.param_indexes, num_shape_coeffs*sizeof(user_precision_t),
                          hipMemcpyHostToDevice ));
    cudaErrorCheckCall( hipMalloc( (void**)&(d_sbf), sbf_N*sbf_L*sizeof(user_precision_t) ));
    cudaErrorCheckCall( hipMemcpy( d_sbf, sbf, sbf_N*sbf_L*sizeof(user_precision_t),
                        hipMemcpyHostToDevice ));
  }

  if (comptype == POINT || comptype == GAUSSIAN ) {

    cudaErrorCheckKernel("kern_calc_visi_point_or_gauss",
                  kern_calc_visi_point_or_gauss, grid, threads,
                  d_components, d_beam_gains,
                  d_us, d_vs, d_ws,
                  d_sum_visi_XX_real, d_sum_visi_XX_imag,
                  d_sum_visi_XY_real, d_sum_visi_XY_imag,
                  d_sum_visi_YX_real, d_sum_visi_YX_imag,
                  d_sum_visi_YY_real, d_sum_visi_YY_imag,
                  num_components, num_baselines, num_freqs, num_visis,
                  num_times, beamtype, comptype);
  }
  else if (comptype == SHAPELET) {
    cudaErrorCheckKernel("kern_calc_visi_shapelets",
                  kern_calc_visi_shapelets, grid, threads,
                  d_components, d_beam_gains,
                  d_us, d_vs, d_ws,
                  d_allsteps_wavelengths,
                  d_u_shapes, d_v_shapes, d_w_shapes,
                  d_sum_visi_XX_real, d_sum_visi_XX_imag,
                  d_sum_visi_XY_real, d_sum_visi_XY_imag,
                  d_sum_visi_YX_real, d_sum_visi_YX_imag,
                  d_sum_visi_YY_real, d_sum_visi_YY_imag,
                  d_sbf,  num_components,
                  num_baselines, num_freqs, num_visis,
                  num_shape_coeffs, num_times, beamtype);
  }

  cudaErrorCheckCall( hipMemcpy(sum_visi_XX_real, d_sum_visi_XX_real,
                             num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_XY_real, d_sum_visi_XY_real,
                             num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_YX_real, d_sum_visi_YX_real,
                             num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_YY_real, d_sum_visi_YY_real,
                             num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_XX_imag, d_sum_visi_XX_imag,
                             num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_XY_imag, d_sum_visi_XY_imag,
                             num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_YX_imag, d_sum_visi_YX_imag,
                             num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_YY_imag, d_sum_visi_YY_imag,
                             num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));


  free_d_components(d_chunked_source, comptype);

  cudaErrorCheckCall(  hipFree( d_sum_visi_XX_real ) );
  cudaErrorCheckCall(  hipFree( d_sum_visi_XX_imag ) );
  cudaErrorCheckCall(  hipFree( d_sum_visi_XY_real ) );
  cudaErrorCheckCall(  hipFree( d_sum_visi_XY_imag ) );
  cudaErrorCheckCall(  hipFree( d_sum_visi_YX_real ) );
  cudaErrorCheckCall(  hipFree( d_sum_visi_YX_imag ) );
  cudaErrorCheckCall(  hipFree( d_sum_visi_YY_real ) );
  cudaErrorCheckCall(  hipFree( d_sum_visi_YY_imag ) );
  cudaErrorCheckCall(  hipFree( d_allsteps_wavelengths ) );



  free_beam_gains(d_beam_gains, beamtype);

  cudaErrorCheckCall(  hipFree( d_us ) );
  cudaErrorCheckCall(  hipFree( d_vs ) );
  cudaErrorCheckCall(  hipFree( d_ws ) );

  cudaErrorCheckCall( hipFree( d_extrap_freqs ) );

  if (comptype == POINT) {
    free_extrapolated_flux_arrays(&d_chunked_source->point_components);
  }
  else if (comptype == GAUSSIAN) {
    free_extrapolated_flux_arrays(&d_chunked_source->gauss_components);
  }
  if (comptype == SHAPELET){
    free_extrapolated_flux_arrays(&d_chunked_source->shape_components);
    cudaErrorCheckCall(  hipFree( d_sbf) );
    cudaErrorCheckCall(  hipFree( d_u_shapes) );
    cudaErrorCheckCall(  hipFree( d_v_shapes) );
    cudaErrorCheckCall(  hipFree( d_w_shapes) );
  }
}
