#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <complex.h>
#include <math.h>
#include "cudacomplex.h"
#include "fundamental_coords.h"
#include "constants.h"
#include "shapelet_basis.h"
#include "source_components.h"
#include "cudacheck.h"
#include "woden_struct_defs.h"
#include "primary_beam_cuda.h"
#include "FEE_primary_beam_cuda.h"

__device__ void extrap_stokes(float *d_allsteps_wavelengths, float *d_ref_freqs,
           float *d_ref_stokesI, float *d_ref_stokesQ,
           float *d_ref_stokesU, float *d_ref_stokesV,
           float *d_SIs, int iComponent, int iBaseline,
           float * flux_I, float * flux_Q, float * flux_U, float * flux_V){

  float d_freq = VELC / d_allsteps_wavelengths[iBaseline];
  float d_ref_freq = d_ref_freqs[iComponent];

  float flux_ratio = powf(d_freq / d_ref_freq, d_SIs[iComponent]);

  * flux_I = d_ref_stokesI[iComponent] * flux_ratio;
  * flux_Q = d_ref_stokesQ[iComponent] * flux_ratio;
  * flux_U = d_ref_stokesU[iComponent] * flux_ratio;
  * flux_V = d_ref_stokesV[iComponent] * flux_ratio;

}

__device__ hipFloatComplex calc_measurement_equation(float *d_us,
           float *d_vs, float *d_ws, float *d_ls, float *d_ms, float *d_ns,
           const int iBaseline, const int iComponent){

  float u, v, w;
  float l, m, n;

  u = d_us[iBaseline];
  v = d_vs[iBaseline];
  w = d_ws[iBaseline];

  l = d_ls[iComponent];
  m = d_ms[iComponent];
  n = d_ns[iComponent];

  hipFloatComplex visi;

  //Not sure why, but get match with OSKAR/RTS sims, and correct location
  //on sky through WSClean, without negative infront on 2pi
  float temp = 2*M_PI*( u*l + v*m + w*(n-1) );
  sincosf(temp, &(visi.y), &(visi.x));

  return visi;
}

__device__ void apply_beam_gains(hipFloatComplex g1x, hipFloatComplex D1x,
          hipFloatComplex D1y, hipFloatComplex g1yy,
          hipFloatComplex g2x, hipFloatComplex D2x,
          hipFloatComplex D2y, hipFloatComplex g2y,
          float flux_I, float flux_Q,
          float flux_U, float flux_V,
          hipFloatComplex visi_component,
          hipFloatComplex * visi_XX, hipFloatComplex * visi_XY,
          hipFloatComplex * visi_YX, hipFloatComplex * visi_YY) {

  //Conjugate the second beam gains
  hipFloatComplex g2x_conj = make_hipFloatComplex(g2x.x,-g2x.y);
  hipFloatComplex D2x_conj = make_hipFloatComplex(D2x.x,-D2x.y);
  hipFloatComplex D2y_conj = make_hipFloatComplex(D2y.x,-D2y.y);
  hipFloatComplex g2y_conj = make_hipFloatComplex(g2y.x,-g2y.y);

  //Create the Stokes visibilities
  hipFloatComplex visi_I = hipCmulf(make_hipComplex(flux_I,0.0),visi_component );
  hipFloatComplex visi_Q = hipCmulf(make_hipComplex(flux_Q,0.0),visi_component );
  hipFloatComplex visi_U = hipCmulf(make_hipComplex(flux_U,0.0),visi_component );
  hipFloatComplex visi_V = hipCmulf(make_hipComplex(flux_V,0.0),visi_component );

  hipFloatComplex this_XX;
  hipFloatComplex this_XY;
  hipFloatComplex this_YX;
  hipFloatComplex this_YY;

  //Convert the Stokes into instrumental visibilities
  this_XX = hipCmulf(hipCmulf(g1x,g2x_conj) + hipCmulf(D1x,D2x_conj),visi_I);
  this_XX += hipCmulf(hipCmulf(g1x,g2x_conj) - hipCmulf(D1x,D2x_conj),visi_Q);
  this_XX += hipCmulf(hipCmulf(g1x,D2x_conj) + hipCmulf(D1x,g2x_conj),visi_U);
  this_XX += hipCmulf(hipCmulf(make_hipFloatComplex(0.0,1.0),visi_V), hipCmulf(g1x,D2x_conj) - hipCmulf(D1x,g2x_conj) );

  this_XY = hipCmulf(hipCmulf(g1x,D2y_conj) + hipCmulf(D1x,g2y_conj),visi_I);
  this_XY += hipCmulf(hipCmulf(g1x,D2y_conj) - hipCmulf(D1x,g2y_conj),visi_Q);
  this_XY += hipCmulf(hipCmulf(g1x,g2y_conj) + hipCmulf(D1x,D2y_conj),visi_U);
  this_XY += hipCmulf(hipCmulf(make_hipFloatComplex(0.0,1.0),visi_V), hipCmulf(g1x,g2y_conj) - hipCmulf(D1x,D2y_conj) );

  this_YX = hipCmulf(hipCmulf(D1y,g2x_conj) + hipCmulf(g1yy,D2x_conj),visi_I);
  this_YX += hipCmulf(hipCmulf(D1y,g2x_conj) - hipCmulf(g1yy,D2x_conj),visi_Q);
  this_YX += hipCmulf(hipCmulf(D1y,D2x_conj) + hipCmulf(g1yy,g2x_conj),visi_U);
  this_YX += hipCmulf(hipCmulf(make_hipFloatComplex(0.0,1.0),visi_V), hipCmulf(D1y,D2x_conj) - hipCmulf(g1yy,g2x_conj) );

  this_YY = hipCmulf(hipCmulf(D1y,D2y_conj) + hipCmulf(g1yy,g2y_conj),visi_I);
  this_YY += hipCmulf(hipCmulf(D1y,D2y_conj) - hipCmulf(g1yy,g2y_conj),visi_Q);
  this_YY += hipCmulf(hipCmulf(D1y,g2y_conj) + hipCmulf(g1yy,D2y_conj),visi_U);
  this_YY += hipCmulf(hipCmulf(make_hipFloatComplex(0.0,1.0),visi_V), hipCmulf(D1y,g2y_conj) - hipCmulf(g1yy,D2y_conj) );

  * visi_XX = this_XX;
  * visi_XY = this_XY;
  * visi_YX = this_YX;
  * visi_YY = this_YY;

}

__device__ void get_beam_gains(int iBaseline, int iComponent, int num_freqs,
           int num_baselines, int num_components, int num_times, int beamtype,
           hipFloatComplex *d_primay_beam_J00, hipFloatComplex *d_primay_beam_J01,
           hipFloatComplex *d_primay_beam_J10, hipFloatComplex *d_primay_beam_J11,
           hipFloatComplex * g1x, hipFloatComplex * D1x,
           hipFloatComplex * D1y, hipFloatComplex * g1yy,
           hipFloatComplex * g2x, hipFloatComplex * D2x,
           hipFloatComplex * D2y, hipFloatComplex * g2y){

  int beam_ind = 0;
  int time_ind = 0;
  int freq_ind = 0;

  time_ind = (int)floorf( (float)iBaseline / ((float)num_baselines * (float)num_freqs));
  freq_ind = (int)floorf( ((float)iBaseline - ((float)time_ind*(float)num_baselines * (float)num_freqs)) / (float)num_baselines);
  beam_ind = num_freqs*time_ind*num_components + (num_components*freq_ind) + iComponent;

  //Get XX,YY if using a beam
  if (beamtype == FEE_BEAM || beamtype == ANALY_DIPOLE || beamtype == GAUSS_BEAM) {
    * g1x = d_primay_beam_J00[beam_ind];
    * g2x = d_primay_beam_J00[beam_ind];
    * g1yy = d_primay_beam_J11[beam_ind];
    * g2y = d_primay_beam_J11[beam_ind];
  }
  else { //Set beam gains to 1.0 if not
    * g1x = make_hipComplex(1.0, 0.0);
    * g2x = make_hipComplex(1.0, 0.0);
    * g1yy = make_hipComplex(1.0, 0.0);
    * g2y = make_hipComplex(1.0, 0.0);
  }

  //Only FEE model has XY and YX at the moment
  if (beamtype == FEE_BEAM) {
    * D1x = d_primay_beam_J01[beam_ind];
    * D2x = d_primay_beam_J01[beam_ind];
    * D1y = d_primay_beam_J10[beam_ind];
    * D2y = d_primay_beam_J10[beam_ind];
  }
  else {
    * D1x = make_hipComplex(0.0, 0.0);
    * D2x = make_hipComplex(0.0, 0.0);
    * D1y = make_hipComplex(0.0, 0.0);
    * D2y = make_hipComplex(0.0, 0.0);
  }
} //end __device__ get_beam_gains

__device__ void update_sum_visis(int iBaseline, int iComponent, int num_freqs,
           int num_baselines, int num_components, int num_times, int beamtype,
           hipFloatComplex *d_primay_beam_J00, hipFloatComplex *d_primay_beam_J01,
           hipFloatComplex *d_primay_beam_J10, hipFloatComplex *d_primay_beam_J11,
           hipFloatComplex visi_component,
           float flux_I, float flux_Q, float flux_U, float flux_V,
           float *d_sum_visi_XX_real, float *d_sum_visi_XX_imag,
           float *d_sum_visi_XY_real, float *d_sum_visi_XY_imag,
           float *d_sum_visi_YX_real, float *d_sum_visi_YX_imag,
           float *d_sum_visi_YY_real, float *d_sum_visi_YY_imag){

    hipFloatComplex g1x;
    hipFloatComplex D1x;
    hipFloatComplex D1y;
    hipFloatComplex g1yy;
    hipFloatComplex g2x;
    hipFloatComplex D2x;
    hipFloatComplex D2y;
    hipFloatComplex g2y;

    get_beam_gains(iBaseline, iComponent, num_freqs,
               num_baselines, num_components, num_times, beamtype,
               d_primay_beam_J00, d_primay_beam_J01,
               d_primay_beam_J10, d_primay_beam_J11,
               &g1x, &D1x, &D1y, &g1yy, &g2x, &D2x, &D2y, &g2y);

    hipFloatComplex visi_XX;
    hipFloatComplex visi_XY;
    hipFloatComplex visi_YX;
    hipFloatComplex visi_YY;

    apply_beam_gains(g1x, D1x, D1y, g1yy, g2x, D2x, D2y, g2y,
                    flux_I, flux_Q, flux_U, flux_V,
                    visi_component, &visi_XX, &visi_XY, &visi_YX, &visi_YY);

    atomicAdd(&d_sum_visi_XX_real[iBaseline],visi_XX.x);
    atomicAdd(&d_sum_visi_XX_imag[iBaseline],visi_XX.y);

    atomicAdd(&d_sum_visi_XY_real[iBaseline],visi_XY.x);
    atomicAdd(&d_sum_visi_XY_imag[iBaseline],visi_XY.y);

    atomicAdd(&d_sum_visi_YX_real[iBaseline],visi_YX.x);
    atomicAdd(&d_sum_visi_YX_imag[iBaseline],visi_YX.y);

    atomicAdd(&d_sum_visi_YY_real[iBaseline],visi_YY.x);
    atomicAdd(&d_sum_visi_YY_imag[iBaseline],visi_YY.y);

}

void source_component_common(int num_components,
           hipFloatComplex *d_primay_beam_J00, hipFloatComplex *d_primay_beam_J01,
           hipFloatComplex *d_primay_beam_J10, hipFloatComplex *d_primay_beam_J11,
           float *d_freqs, float *d_ls, float *d_ms, float *d_ns,
           float *d_ras, float *d_decs, float *azs, float *zas,
           float *sin_para_angs, float *cos_para_angs,
           float *beam_has, float *beam_decs,
           woden_settings_t *woden_settings,
           beam_settings_t beam_settings,
           RTS_MWA_FEE_beam_t *FEE_beam){

  dim3 grid, threads;

  threads.x = 128;
  threads.y = 1;
  threads.z = 1;
  grid.x = (int)ceil( (float)num_components / (float)threads.x );
  grid.y = 1;
  grid.z = 1;

  cudaErrorCheckKernel("kern_calc_lmn",
                        kern_calc_lmn, grid, threads,
                        woden_settings->ra0,
                        woden_settings->sdec0, woden_settings->cdec0,
                        d_ras, d_decs,
                        d_ls, d_ms, d_ns, num_components)

  //If using a gaussian primary beam, calculate beam values for all freqs,
  //lsts and point component locations
  if (beam_settings.beamtype == GAUSS_BEAM) {

    //TODO currently hardcoded to have beam position angle = 0.
    //Should this change with az/za?
    float cos_theta = 1.0;
    float sin_theta = 0.0;
    float sin_2theta = 0.0;
    float fwhm_lm = sinf(beam_settings.beam_FWHM_rad);

    printf("\tDoing gaussian beam tings\n");

    calculate_gaussian_beam(num_components,
         woden_settings->num_time_steps, woden_settings->num_freqs,
         beam_settings.gauss_ha, beam_settings.gauss_sdec,
         beam_settings.gauss_cdec,
         fwhm_lm, cos_theta, sin_theta, sin_2theta,
         beam_settings.beam_ref_freq, d_freqs,
         beam_has, beam_decs,
         d_primay_beam_J00, d_primay_beam_J11);

  }// end if beam == GAUSS

  else if (beam_settings.beamtype == FEE_BEAM) {

    //Rotate FEE beam by parallactic angle
    int rotation = 1;
    //Normalise FEE beam to zenith
    int scaling = 1;

    calc_CUDA_FEE_beam(azs, zas, sin_para_angs, cos_para_angs,
           num_components, woden_settings->num_time_steps, FEE_beam,
           rotation, scaling);

    threads.x = 64;
    threads.y = 4;
    grid.x = (int)ceil( (float)woden_settings->num_visis / (float)threads.x );
    grid.y = (int)ceil( ((float)num_components) / ((float)threads.y) );

    cudaErrorCheckKernel("kern_map_FEE_beam_gains",
              kern_map_FEE_beam_gains, grid, threads,
              (hipFloatComplex *)FEE_beam->d_FEE_beam_gain_matrices,
              d_primay_beam_J00, d_primay_beam_J01,
              d_primay_beam_J10, d_primay_beam_J11,
              woden_settings->num_freqs, num_components,
              woden_settings->num_visis, woden_settings->num_baselines,
              woden_settings->num_time_steps);
  }

  else if (beam_settings.beamtype == ANALY_DIPOLE) {
    printf("\tDoing analytic_dipole (EDA2 beam)\n");

    calculate_analytic_dipole_beam(num_components,
         woden_settings->num_time_steps, woden_settings->num_freqs,
         azs, zas, d_freqs, d_primay_beam_J00, d_primay_beam_J11);
  }
}

__global__ void kern_calc_visi_point(float *d_point_ras, float *d_point_decs,
           float *d_point_freqs, float *d_point_stokesI, float *d_point_stokesQ,
           float *d_point_stokesU, float *d_point_stokesV, float *d_point_SIs,
           float *d_us, float *d_vs, float *d_ws,
           float *d_sum_visi_XX_real, float *d_sum_visi_XX_imag,
           float *d_sum_visi_XY_real, float *d_sum_visi_XY_imag,
           float *d_sum_visi_YX_real, float *d_sum_visi_YX_imag,
           float *d_sum_visi_YY_real, float *d_sum_visi_YY_imag,
           float *d_allsteps_wavelengths,
           float *d_ls, float *d_ms, float *d_ns,
           int num_points, int num_baselines, int num_freqs, int num_visis,
           int num_times, int beamtype,
           hipFloatComplex *d_primay_beam_J00, hipFloatComplex *d_primay_beam_J01,
           hipFloatComplex *d_primay_beam_J10, hipFloatComplex *d_primay_beam_J11) {

  // Start by computing which baseline we're going to do
  const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);
  const int iComponent = threadIdx.y + (blockDim.y*blockIdx.y);

  if(iBaseline < num_visis && iComponent < num_points) {

    float point_flux_I;
    float point_flux_Q;
    float point_flux_U;
    float point_flux_V;

    extrap_stokes(d_allsteps_wavelengths, d_point_freqs,
                 d_point_stokesI, d_point_stokesQ,
                 d_point_stokesU, d_point_stokesV,
                 d_point_SIs, iComponent, iBaseline,
                 &point_flux_I, &point_flux_Q, &point_flux_U, &point_flux_V);

    hipFloatComplex visi_point;
    visi_point = calc_measurement_equation(d_us, d_vs, d_ws,
                           d_ls, d_ms, d_ns,
                           iBaseline, iComponent);

    update_sum_visis(iBaseline, iComponent, num_freqs,
           num_baselines, num_points, num_times, beamtype,
           d_primay_beam_J00, d_primay_beam_J01,
           d_primay_beam_J10, d_primay_beam_J11,
           visi_point,
           point_flux_I, point_flux_Q, point_flux_U, point_flux_V,
           d_sum_visi_XX_real, d_sum_visi_XX_imag,
           d_sum_visi_XY_real, d_sum_visi_XY_imag,
           d_sum_visi_YX_real, d_sum_visi_YX_imag,
           d_sum_visi_YY_real, d_sum_visi_YY_imag);

  }
}

__global__ void kern_calc_visi_gaussian(float *d_gauss_ras, float *d_gauss_decs,
           float *d_gauss_freqs, float *d_gauss_stokesI, float *d_gauss_stokesQ,
           float *d_gauss_stokesU, float *d_gauss_stokesV, float *d_gauss_SIs,
           float *d_us, float *d_vs, float *d_ws,
           float *d_sum_visi_XX_real, float *d_sum_visi_XX_imag,
           float *d_sum_visi_XY_real, float *d_sum_visi_XY_imag,
           float *d_sum_visi_YX_real, float *d_sum_visi_YX_imag,
           float *d_sum_visi_YY_real, float *d_sum_visi_YY_imag,
           float *d_allsteps_wavelengths,
           float *d_ls, float *d_ms, float *d_ns,
           float *d_gauss_pas, float *d_gauss_majors, float *d_gauss_minors,
           int num_gauss, int num_baselines, int num_freqs, int num_visis,
           int num_times, int beamtype,
           hipFloatComplex *d_primay_beam_J00, hipFloatComplex *d_primay_beam_J01,
           hipFloatComplex *d_primay_beam_J10, hipFloatComplex *d_primay_beam_J11) {

  // Start by computing which baseline we're going to do
  const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);
  const int iComponent = threadIdx.y + (blockDim.y*blockIdx.y);

  if(iBaseline < num_visis && iComponent < num_gauss) {

    float gauss_flux_I;
    float gauss_flux_Q;
    float gauss_flux_U;
    float gauss_flux_V;

    extrap_stokes(d_allsteps_wavelengths, d_gauss_freqs,
                 d_gauss_stokesI, d_gauss_stokesQ,
                 d_gauss_stokesU, d_gauss_stokesV,
                 d_gauss_SIs, iComponent, iBaseline,
                 &gauss_flux_I, &gauss_flux_Q, &gauss_flux_U, &gauss_flux_V);

    hipFloatComplex visi_gauss;
    visi_gauss = calc_measurement_equation(d_us, d_vs, d_ws,
                           d_ls, d_ms, d_ns,
                           iBaseline, iComponent);

    hipFloatComplex V_envelop = make_hipFloatComplex( 1.0, 0.0 );

    float pa = d_gauss_pas[iComponent];
    float u = d_us[iBaseline];
    float v = d_vs[iBaseline];
    float sinpa = sin(pa);
    float cospa = cos(pa);

    float x =  cospa*v + sinpa*u; // major axis
    float y = -sinpa*v + cospa*u; // minor axis
    float invsig_x = d_gauss_majors[iComponent];
    float invsig_y = d_gauss_minors[iComponent];

    V_envelop = make_hipFloatComplex( exp( -0.5 * ( x*x*invsig_x*invsig_x*M_PI_2_2_LN_2 + y*y*invsig_y*invsig_y*M_PI_2_2_LN_2 ) ), 0.0 );

    visi_gauss = hipCmulf(visi_gauss, V_envelop);

    update_sum_visis(iBaseline, iComponent, num_freqs,
           num_baselines, num_gauss, num_times, beamtype,
           d_primay_beam_J00, d_primay_beam_J01,
           d_primay_beam_J10, d_primay_beam_J11,
           visi_gauss,
           gauss_flux_I, gauss_flux_Q, gauss_flux_U, gauss_flux_V,
           d_sum_visi_XX_real, d_sum_visi_XX_imag,
           d_sum_visi_XY_real, d_sum_visi_XY_imag,
           d_sum_visi_YX_real, d_sum_visi_YX_imag,
           d_sum_visi_YY_real, d_sum_visi_YY_imag);
  }
}

__global__ void kern_calc_visi_shapelets(float *d_shape_freqs,
      float *d_shape_stokesI, float *d_shape_stokesQ,
      float *d_shape_stokesU, float *d_shape_stokesV, float *d_shape_SIs,
      float *d_us, float *d_vs, float *d_ws,
      float *d_allsteps_wavelengths,
      float *d_u_s_metres, float *d_v_s_metres, float *d_w_s_metres,
      float *d_sum_visi_XX_real, float *d_sum_visi_XX_imag,
      float *d_sum_visi_XY_real, float *d_sum_visi_XY_imag,
      float *d_sum_visi_YX_real, float *d_sum_visi_YX_imag,
      float *d_sum_visi_YY_real, float *d_sum_visi_YY_imag,
      float *d_shape_pas, float *d_shape_majors,
      float *d_shape_minors,
      float *d_shape_n1s, float *d_shape_n2s, float *d_shape_coeffs,
      float *d_shape_param_indexes,
      float *d_shape_ls, float *d_shape_ms, float *d_shape_ns,
      float *d_sbf,
      int num_shapes, int num_baselines, int num_freqs, int num_visis,
      const int num_coeffs, int num_times, int beamtype,
      hipFloatComplex *d_primay_beam_J00, hipFloatComplex *d_primay_beam_J01,
      hipFloatComplex *d_primay_beam_J10, hipFloatComplex *d_primay_beam_J11) {

  // Start by computing which baseline we're going to do
  const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);
  const int iCoeff = threadIdx.y + (blockDim.y*blockIdx.y);

  if (iBaseline < num_visis && iCoeff < num_coeffs) {
    int iComponent = d_shape_param_indexes[iCoeff];

    float shape_flux_I;
    float shape_flux_Q;
    float shape_flux_U;
    float shape_flux_V;

    extrap_stokes(d_allsteps_wavelengths, d_shape_freqs,
                 d_shape_stokesI, d_shape_stokesQ,
                 d_shape_stokesU, d_shape_stokesV,
                 d_shape_SIs, iComponent, iBaseline,
                 &shape_flux_I, &shape_flux_Q, &shape_flux_U, &shape_flux_V);

    hipFloatComplex visi_shape;
    visi_shape = calc_measurement_equation(d_us, d_vs, d_ws,
                          d_shape_ls, d_shape_ms, d_shape_ns,
                          iBaseline, iComponent);

    float pa = d_shape_pas[iComponent];
    float sinpa = sin(pa);
    float cospa = cos(pa);

    float d_wavelength = d_allsteps_wavelengths[iBaseline];

    float u_s = d_u_s_metres[iComponent*num_visis + iBaseline] / d_wavelength;
    float v_s = d_v_s_metres[iComponent*num_visis + iBaseline] / d_wavelength;
    //
    float x = (cospa*v_s + sinpa*u_s); // major axis
    float y = (-sinpa*v_s + cospa*u_s); // minor axis

    //Scales the FWHM to std to match basis functions, and account for the
    //basis functions being stored with beta = 1.0
    //Basis functions have been stored in such a way that x is in the same
    //direction as on sky, but y is opposite, so include negative here
    float const_x = (d_shape_majors[iComponent]*SQRT_M_PI_2_2_LN_2)/sbf_dx;
    float const_y = -(d_shape_minors[iComponent]*SQRT_M_PI_2_2_LN_2)/sbf_dx;

    // I^(n1+n2) = Ipow_lookup[(n1+n2) % 4]
    hipFloatComplex Ipow_lookup[] = { make_hipFloatComplex(  1.0,  0.0 ),
                                     make_hipFloatComplex(  0.0,  1.0 ),
                                     make_hipFloatComplex( -1.0,  0.0 ),
                                     make_hipFloatComplex(  0.0, -1.0 ) };

    float xlow, xhigh, ylow, yhigh, u_value, v_value, f_hat, *sbf_n;

    // find the indices in the basis functions for u*beta_u and v*beta_v

    float xpos = x*const_x + sbf_c;
    float ypos = y*const_y + sbf_c;

    int xindex = (int)floor(xpos);
    int yindex = (int)floor(ypos);
    //
    int n1 = (int)d_shape_n1s[iCoeff];
    int n2 = (int)d_shape_n2s[iCoeff];

    // if ( n1 < 0 || n2 < 0 || n1 >= sbf_N || n2 >= sbf_N ) continue;

    f_hat = d_shape_coeffs[iCoeff];
    //
    sbf_n = &d_sbf[n1*sbf_L];
    xlow  = sbf_n[xindex];
    xhigh = sbf_n[xindex+1];
    u_value = xlow + (xhigh-xlow)*(xpos-xindex);

    sbf_n = &d_sbf[n2*sbf_L];
    ylow  = sbf_n[yindex];
    yhigh = sbf_n[yindex+1];
    v_value = ylow + (yhigh-ylow)*(ypos-yindex);

    // accumulate the intensity model for baseline pair (u,v)
    hipFloatComplex V_envelop = make_hipFloatComplex( 0.0, 0.0 );
    V_envelop = V_envelop + Ipow_lookup[(n1+n2) % 4] * f_hat * u_value*v_value;

    visi_shape = hipCmulf(visi_shape, V_envelop);

    update_sum_visis(iBaseline, iComponent, num_freqs,
           num_baselines, num_shapes, num_times, beamtype,
           d_primay_beam_J00, d_primay_beam_J01,
           d_primay_beam_J10, d_primay_beam_J11,
           visi_shape,
           shape_flux_I, shape_flux_Q, shape_flux_U, shape_flux_V,
           d_sum_visi_XX_real, d_sum_visi_XX_imag,
           d_sum_visi_XY_real, d_sum_visi_XY_imag,
           d_sum_visi_YX_real, d_sum_visi_YX_imag,
           d_sum_visi_YY_real, d_sum_visi_YY_imag);

  }
}



//TODO use in future for testing
// __global__ void kern_extrap_stokes(int num_visis, int num_components,
//            float *d_allsteps_wavelengths, float *d_ref_freqs, float *d_SIs,
//            float *d_ref_stokesI, float *d_ref_stokesQ,
//            float *d_ref_stokesU, float *d_ref_stokesV,
//            float *d_flux_I, float *d_flux_Q,
//            float *d_flux_U, float *d_flux_V ) {
//
//   // Start by computing which baseline we're going to do
//   const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);
//   const int iComponent = threadIdx.y + (blockDim.y*blockIdx.y);
//
//   if(iBaseline < num_visis && iComponent < num_components) {
//
//     float flux_I, flux_Q, flux_U, flux_V;
//
//     extrap_stokes(d_allsteps_wavelengths, d_ref_freqs,
//                  d_ref_stokesI, d_ref_stokesQ,
//                  d_ref_stokesU, d_ref_stokesV,
//                  d_SIs, iComponent, iBaseline,
//                  &flux_I, &flux_Q, &flux_U, &flux_V);
//
//       d_flux_I[iComponent] = flux_I;
//       d_flux_Q[iComponent] = flux_Q;
//       d_flux_U[iComponent] = flux_U;
//       d_flux_V[iComponent] = flux_V;
//
//   }
// }

// TODO replace this with a test function that uses extrap_stokes
// extern "C" void test_extrap_flux(catsource_t *catsource,
//            int num_visis, int num_components,
//            float *allsteps_wavelengths, float *flux_I, float *flux_Q,
//            float *flux_U, float *flux_V ){
//
//   float *d_allsteps_wavelengths = NULL;
//   float *d_ref_freqs = NULL;
//   float *d_SIs = NULL;
//   float *d_ref_stokesI = NULL;
//   float *d_ref_stokesQ = NULL;
//   float *d_ref_stokesU = NULL;
//   float *d_ref_stokesV = NULL;
//   float *d_flux_I = NULL;
//   float *d_flux_Q = NULL;
//   float *d_flux_U = NULL;
//   float *d_flux_V = NULL;
//
//   hipMalloc( (void**)&d_allsteps_wavelengths, num_visis*sizeof(float));
//   hipMalloc( (void**)&d_ref_freqs, num_components*sizeof(float));
//   hipMalloc( (void**)&d_SIs, num_components*sizeof(float));
//   hipMalloc( (void**)&d_ref_stokesI, num_components*sizeof(float));
//   hipMalloc( (void**)&d_ref_stokesQ, num_components*sizeof(float));
//   hipMalloc( (void**)&d_ref_stokesU, num_components*sizeof(float));
//   hipMalloc( (void**)&d_ref_stokesV, num_components*sizeof(float));
//   hipMalloc( (void**)&d_flux_I, num_components*sizeof(float));
//   hipMalloc( (void**)&d_flux_Q, num_components*sizeof(float));
//   hipMalloc( (void**)&d_flux_U, num_components*sizeof(float));
//   hipMalloc( (void**)&d_flux_V, num_components*sizeof(float));
//
//   // printf("CUDA error 1: %s\n", hipGetErrorString( hipGetLastError() ) );
//   hipMemcpy(d_allsteps_wavelengths, allsteps_wavelengths, num_visis*sizeof(float), hipMemcpyHostToDevice );
//
//   // printf("CUDA error 2: %s\n", hipGetErrorString( hipGetLastError() ) );
//   hipMemcpy(d_ref_stokesI, catsource->point_ref_stokesI, num_components*sizeof(float), hipMemcpyHostToDevice );
//   hipMemcpy(d_ref_stokesQ, catsource->point_ref_stokesQ, num_components*sizeof(float), hipMemcpyHostToDevice );
//   hipMemcpy(d_ref_stokesU, catsource->point_ref_stokesU, num_components*sizeof(float), hipMemcpyHostToDevice );
//   hipMemcpy(d_ref_stokesV, catsource->point_ref_stokesV, num_components*sizeof(float), hipMemcpyHostToDevice );
//   hipMemcpy(d_ref_freqs, catsource->point_ref_freqs, num_components*sizeof(float), hipMemcpyHostToDevice );
//   hipMemcpy(d_SIs, catsource->point_SIs, num_components*sizeof(float), hipMemcpyHostToDevice );
//
//   dim3 grid, threads;
//
//   threads.x = 64;
//   threads.y = 2;
//   grid.x = (int)ceil( (float)num_visis / (float)threads.x );
//   grid.y = (int)ceil( (float)num_components / (float)threads.y );
//
//   kern_extrap_stokes<<< grid, threads >>>(num_visis, num_components,
//                      d_allsteps_wavelengths, d_ref_freqs, d_SIs,
//                      d_ref_stokesI, d_ref_stokesQ,
//                      d_ref_stokesU, d_ref_stokesV,
//                      d_flux_I, d_flux_Q,
//                      d_flux_U, d_flux_V);
//
//   hipMemcpy(flux_I, d_flux_I, num_components*sizeof(float),hipMemcpyDeviceToHost);
//   hipMemcpy(flux_Q, d_flux_Q, num_components*sizeof(float),hipMemcpyDeviceToHost);
//   hipMemcpy(flux_U, d_flux_U, num_components*sizeof(float),hipMemcpyDeviceToHost);
//   hipMemcpy(flux_V, d_flux_V, num_components*sizeof(float),hipMemcpyDeviceToHost);
//
//   hipFree( d_allsteps_wavelengths );
//   hipFree( d_ref_freqs );
//   hipFree( d_SIs );
//   hipFree( d_ref_stokesI );
//   hipFree( d_ref_stokesQ );
//   hipFree( d_ref_stokesU );
//   hipFree( d_ref_stokesV );
//   hipFree( d_flux_I );
//   hipFree( d_flux_Q );
//   hipFree( d_flux_U );
//   hipFree( d_flux_V );
// }
