#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <complex.h>
#include <math.h>
#include "cudacomplex.h"
#include "fundamental_coords.h"
#include "constants.h"
#include "shapelet_basis.h"
#include "source_components.h"
#include "cudacheck.h"
#include "woden_struct_defs.h"
#include "primary_beam_cuda.h"
#include "woden_precision_defs.h"

__device__ void extrap_stokes(user_precision_t *d_allsteps_wavelengths,
           double *d_ref_freqs,
           user_precision_t *d_ref_stokesI, user_precision_t *d_ref_stokesQ,
           user_precision_t *d_ref_stokesU, user_precision_t *d_ref_stokesV,
           user_precision_t *d_SIs, int iComponent, int iBaseline,
           user_precision_t * flux_I, user_precision_t * flux_Q,
           user_precision_t * flux_U, user_precision_t * flux_V){

  double d_freq = VELC / d_allsteps_wavelengths[iBaseline];
  double d_ref_freq = d_ref_freqs[iComponent];

  user_precision_t flux_ratio = pow(d_freq / d_ref_freq, d_SIs[iComponent]);

  * flux_I = d_ref_stokesI[iComponent] * flux_ratio;
  * flux_Q = d_ref_stokesQ[iComponent] * flux_ratio;
  * flux_U = d_ref_stokesU[iComponent] * flux_ratio;
  * flux_V = d_ref_stokesV[iComponent] * flux_ratio;

}

__device__  cuUserComplex calc_measurement_equation(user_precision_t *d_us,
           user_precision_t *d_vs, user_precision_t *d_ws,
           double *d_ls, double *d_ms, double *d_ns,
           const int iBaseline, const int iComponent){

  cuUserComplex visi;

  double u, v, w;
  double l, m, n;

  u = (double)d_us[iBaseline];
  v = (double)d_vs[iBaseline];
  w = (double)d_ws[iBaseline];

  l = d_ls[iComponent];
  m = d_ms[iComponent];
  n = d_ns[iComponent];

  //Not sure why, but get match with OSKAR/RTS sims, and correct location
  //on sky through WSClean, without negative infront on 2pi
  double temp = 2*M_PI*( u*l + v*m + w*(n-1) );

  visi.y = (user_precision_t)sin(temp);
  visi.x = (user_precision_t)cos(temp);

  return visi;
}

__device__ void apply_beam_gains(cuUserComplex g1x, cuUserComplex D1x,
          cuUserComplex D1y, cuUserComplex g1y,
          cuUserComplex g2x, cuUserComplex D2x,
          cuUserComplex D2y, cuUserComplex g2y,
          user_precision_t flux_I, user_precision_t flux_Q,
          user_precision_t flux_U, user_precision_t flux_V,
          cuUserComplex visi_component,
          cuUserComplex * visi_XX, cuUserComplex * visi_XY,
          cuUserComplex * visi_YX, cuUserComplex * visi_YY) {

  //Conjugate the second beam gains
  cuUserComplex g2x_conj = make_cuUserComplex(g2x.x,-g2x.y);
  cuUserComplex D2x_conj = make_cuUserComplex(D2x.x,-D2x.y);
  cuUserComplex D2y_conj = make_cuUserComplex(D2y.x,-D2y.y);
  cuUserComplex g2y_conj = make_cuUserComplex(g2y.x,-g2y.y);

  //Create the Stokes visibilities
  cuUserComplex visi_I = make_cuUserComplex(flux_I, 0.0)*visi_component;
  cuUserComplex visi_Q = make_cuUserComplex(flux_Q, 0.0)*visi_component;
  cuUserComplex visi_U = make_cuUserComplex(flux_U, 0.0)*visi_component;
  cuUserComplex visi_V = make_cuUserComplex(flux_V, 0.0)*visi_component;

  cuUserComplex this_XX;
  cuUserComplex this_XY;
  cuUserComplex this_YX;
  cuUserComplex this_YY;

  // this_XX = (g1x*g2x_conj + D1x*D2x_conj);
  // this_XY = (g1x*D2y_conj + D1x*g2y_conj);
  // this_YX = (D1y*g2x_conj + g1y*D2x_conj);
  // this_YY = (D1y*D2y_conj + g1y*g2y_conj);
  //
  // printf("XX %.16f %.16f\n",this_XX.x, this_XX.y );
  // printf("XY %.16f %.16f\n",this_XY.x, this_XY.y );
  // printf("YX %.16f %.16f\n",this_YX.x, this_YX.y );
  // printf("YY %.16f %.16f\n",this_YY.x, this_YY.y );

  this_XX = (g1x*g2x_conj + D1x*D2x_conj)*visi_I;
  this_XX += (g1x*g2x_conj - D1x*D2x_conj)*visi_Q;
  this_XX += (g1x*D2x_conj + D1x*g2x_conj)*visi_U;
  this_XX += (make_cuUserComplex(0.0,1.0)*visi_V)*(g1x*D2x_conj - D1x*g2x_conj);

  this_XY = (g1x*D2y_conj + D1x*g2y_conj)*visi_I;
  this_XY += (g1x*D2y_conj - D1x*g2y_conj)*visi_Q;
  this_XY += (g1x*g2y_conj + D1x*D2y_conj)*visi_U;
  this_XY += (make_cuUserComplex(0.0,1.0)*visi_V)* (g1x*g2y_conj - D1x*D2y_conj);

  this_YX = (D1y*g2x_conj + g1y*D2x_conj)*visi_I;
  this_YX += (D1y*g2x_conj - g1y*D2x_conj)*visi_Q;
  this_YX += (D1y*D2x_conj + g1y*g2x_conj)*visi_U;
  this_YX += (make_cuUserComplex(0.0,1.0)*visi_V)* (D1y*D2x_conj - g1y*g2x_conj);

  this_YY = (D1y*D2y_conj + g1y*g2y_conj)*visi_I;
  this_YY += (D1y*D2y_conj - g1y*g2y_conj)*visi_Q;
  this_YY += (D1y*g2y_conj + g1y*D2y_conj)*visi_U;
  this_YY += (make_cuUserComplex(0.0,1.0)*visi_V)* (D1y*g2y_conj - g1y*D2y_conj);

  * visi_XX = this_XX;
  * visi_XY = this_XY;
  * visi_YX = this_YX;
  * visi_YY = this_YY;

}

__device__ void get_beam_gains(int iBaseline, int iComponent, int num_freqs,
           int num_baselines, int num_components, int num_times, int beamtype,
           cuUserComplex *d_primay_beam_J00, cuUserComplex *d_primay_beam_J01,
           cuUserComplex *d_primay_beam_J10, cuUserComplex *d_primay_beam_J11,
           cuUserComplex * g1x, cuUserComplex * D1x,
           cuUserComplex * D1y, cuUserComplex * g1y,
           cuUserComplex * g2x, cuUserComplex * D2x,
           cuUserComplex * D2y, cuUserComplex * g2y){

  int beam_ind = 0;
  int time_ind = 0;
  int freq_ind = 0;

  time_ind = (int)floorf( (user_precision_t)iBaseline / ((user_precision_t)num_baselines * (user_precision_t)num_freqs));
  freq_ind = (int)floorf( ((user_precision_t)iBaseline - ((user_precision_t)time_ind*(user_precision_t)num_baselines * (user_precision_t)num_freqs)) / (user_precision_t)num_baselines);
  beam_ind = num_freqs*time_ind*num_components + (num_components*freq_ind) + iComponent;

    //Set gains to one if no beam
  if (beamtype == NO_BEAM) {
    * g1x = make_cuUserComplex(1.0, 0.0);
    * g2x = make_cuUserComplex(1.0, 0.0);
    * g1y = make_cuUserComplex(1.0, 0.0);
    * g2y = make_cuUserComplex(1.0, 0.0);
  }

  //Get gains if using a beam
  else {
    * g1x = d_primay_beam_J00[beam_ind];
    * g2x = d_primay_beam_J00[beam_ind];
    * g1y = d_primay_beam_J11[beam_ind];
    * g2y = d_primay_beam_J11[beam_ind];

  }

  //Only MWA models have leakge terms at the moment
  if (beamtype == FEE_BEAM || beamtype == FEE_BEAM_INTERP || beamtype == MWA_ANALY) {
    * D1x = d_primay_beam_J01[beam_ind];
    * D2x = d_primay_beam_J01[beam_ind];
    * D1y = d_primay_beam_J10[beam_ind];
    * D2y = d_primay_beam_J10[beam_ind];
  }
  // Set leakage to zero if no leakage
  else {
    * D1x = make_cuUserComplex(0.0, 0.0);
    * D2x = make_cuUserComplex(0.0, 0.0);
    * D1y = make_cuUserComplex(0.0, 0.0);
    * D2y = make_cuUserComplex(0.0, 0.0);
  }
} //end __device__ get_beam_gains

__device__ void update_sum_visis(int iBaseline, int iComponent, int num_freqs,
    int num_baselines, int num_components, int num_times, int beamtype,
    cuUserComplex *d_primay_beam_J00, cuUserComplex *d_primay_beam_J01,
    cuUserComplex *d_primay_beam_J10, cuUserComplex *d_primay_beam_J11,
    cuUserComplex visi_component,
    user_precision_t flux_I, user_precision_t flux_Q,
    user_precision_t flux_U, user_precision_t flux_V,
    user_precision_t *d_sum_visi_XX_real, user_precision_t *d_sum_visi_XX_imag,
    user_precision_t *d_sum_visi_XY_real, user_precision_t *d_sum_visi_XY_imag,
    user_precision_t *d_sum_visi_YX_real, user_precision_t *d_sum_visi_YX_imag,
    user_precision_t *d_sum_visi_YY_real, user_precision_t *d_sum_visi_YY_imag){

    cuUserComplex g1x;
    cuUserComplex D1x;
    cuUserComplex D1y;
    cuUserComplex g1y;
    cuUserComplex g2x;
    cuUserComplex D2x;
    cuUserComplex D2y;
    cuUserComplex g2y;

    get_beam_gains(iBaseline, iComponent, num_freqs,
               num_baselines, num_components, num_times, beamtype,
               d_primay_beam_J00, d_primay_beam_J01,
               d_primay_beam_J10, d_primay_beam_J11,
               &g1x, &D1x, &D1y, &g1y, &g2x, &D2x, &D2y, &g2y);

    cuUserComplex visi_XX;
    cuUserComplex visi_XY;
    cuUserComplex visi_YX;
    cuUserComplex visi_YY;

    apply_beam_gains(g1x, D1x, D1y, g1y, g2x, D2x, D2y, g2y,
                    flux_I, flux_Q, flux_U, flux_V,
                    visi_component, &visi_XX, &visi_XY, &visi_YX, &visi_YY);

    d_sum_visi_XX_real[iBaseline] += visi_XX.x;
    d_sum_visi_XX_imag[iBaseline] += visi_XX.y;

    d_sum_visi_XY_real[iBaseline] += visi_XY.x;
    d_sum_visi_XY_imag[iBaseline] += visi_XY.y;

    d_sum_visi_YX_real[iBaseline] += visi_YX.x;
    d_sum_visi_YX_imag[iBaseline] += visi_YX.y;

    d_sum_visi_YY_real[iBaseline] += visi_YY.x;
    d_sum_visi_YY_imag[iBaseline] += visi_YY.y;

}

extern "C" void source_component_common(int num_components,
           int num_shape_coeffs, components_t *components,
           double *d_freqs, woden_settings_t *woden_settings,
           beam_settings_t *beam_settings, e_component_type comptype,
           components_t *d_components, d_beam_gains_t *d_component_beam_gains){

  // //Copy across many things from host to Device
  cudaErrorCheckCall( hipMalloc( (void**)&d_components->ras,
                      num_components*sizeof(double) ) );
  cudaErrorCheckCall( hipMemcpy( d_components->ras, components->ras,
                      num_components*sizeof(double), hipMemcpyHostToDevice ) );

  cudaErrorCheckCall( hipMalloc( (void**)&d_components->decs,
                      num_components*sizeof(double) ) );
  cudaErrorCheckCall( hipMemcpy( d_components->decs, components->decs,
                      num_components*sizeof(double), hipMemcpyHostToDevice ) );

  cudaErrorCheckCall( hipMalloc( (void**)&d_components->ref_freqs,
                      num_components*sizeof(double) ) );
  cudaErrorCheckCall( hipMemcpy( d_components->ref_freqs, components->ref_freqs,
                      num_components*sizeof(double), hipMemcpyHostToDevice ) );

  cudaErrorCheckCall( hipMalloc( (void**)&d_components->ref_stokesI,
                      num_components*sizeof(user_precision_t) ) );
  cudaErrorCheckCall( hipMemcpy( d_components->ref_stokesI, components->ref_stokesI,
                      num_components*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

  cudaErrorCheckCall( hipMalloc( (void**)&d_components->ref_stokesQ,
                      num_components*sizeof(user_precision_t) ) );
  cudaErrorCheckCall( hipMemcpy( d_components->ref_stokesQ, components->ref_stokesQ,
                      num_components*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

  cudaErrorCheckCall( hipMalloc( (void**)&d_components->ref_stokesU,
                      num_components*sizeof(user_precision_t) ) );
  cudaErrorCheckCall( hipMemcpy( d_components->ref_stokesU, components->ref_stokesU,
                      num_components*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

  cudaErrorCheckCall( hipMalloc( (void**)&d_components->ref_stokesV,
                      num_components*sizeof(user_precision_t) ) );
  cudaErrorCheckCall( hipMemcpy( d_components->ref_stokesV, components->ref_stokesV,
                      num_components*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

  cudaErrorCheckCall( hipMalloc( (void**)&d_components->SIs,
                      num_components*sizeof(user_precision_t) ) );
  cudaErrorCheckCall( hipMemcpy( d_components->SIs, components->SIs,
                      num_components*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

  if (comptype == GAUSSIAN || comptype == SHAPELET ) {
    cudaErrorCheckCall( hipMalloc( (void**)&d_components->pas,
                        num_components*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->pas, components->pas,
                        num_components*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->majors,
                        num_components*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->majors, components->majors,
                        num_components*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->minors,
                        num_components*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->minors, components->minors,
                        num_components*sizeof(user_precision_t), hipMemcpyHostToDevice ) );
  }

  if (comptype == SHAPELET) {
    cudaErrorCheckCall( hipMalloc( (void**)&d_components->shape_coeffs,
                        num_shape_coeffs*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->shape_coeffs, components->shape_coeffs,
                        num_shape_coeffs*sizeof(user_precision_t),
                        hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->n1s,
                        num_shape_coeffs*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->n1s, components->n1s,
                        num_shape_coeffs*sizeof(user_precision_t),
                        hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->n2s,
                        num_shape_coeffs*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->n2s, components->n2s,
                        num_shape_coeffs*sizeof(user_precision_t),
                        hipMemcpyHostToDevice ) );

    cudaErrorCheckCall( hipMalloc( (void**)&d_components->param_indexes,
                        num_shape_coeffs*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMemcpy( d_components->param_indexes, components->param_indexes,
                        num_shape_coeffs*sizeof(user_precision_t),
                        hipMemcpyHostToDevice ) );
  }

  //Only the MWA beams currently yields cross pol values, so only malloc what
  //we need here
  //TODO in the future, this might need to be a loop over all primary beams,
  //if we have different beams for different tiles
  if (beam_settings->beamtype == FEE_BEAM || beam_settings->beamtype == MWA_ANALY || beam_settings->beamtype == FEE_BEAM_INTERP) {
    cudaErrorCheckCall( hipMalloc( (void**)&d_component_beam_gains->d_Dxs,
                    components->num_primarybeam_values*sizeof(cuUserComplex) ));
    cudaErrorCheckCall( hipMalloc( (void**)&d_component_beam_gains->d_Dys,
                    components->num_primarybeam_values*sizeof(cuUserComplex) ));
  }
  cudaErrorCheckCall( hipMalloc( (void**)&d_component_beam_gains->d_gxs,
                    components->num_primarybeam_values*sizeof(cuUserComplex) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_component_beam_gains->d_gys,
                    components->num_primarybeam_values*sizeof(cuUserComplex) ));
  //
  cudaErrorCheckCall( hipMalloc( (void**)&d_components->ls,
                                               num_components*sizeof(double) ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_components->ms,
                                               num_components*sizeof(double) ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_components->ns,
                                               num_components*sizeof(double) ) );


  dim3 grid, threads;

  threads.x = 128;
  threads.y = 1;
  threads.z = 1;
  grid.x = (int)ceil( (float)num_components / (float)threads.x );
  grid.y = 1;
  grid.z = 1;

  cudaErrorCheckKernel("kern_calc_lmn",
                        kern_calc_lmn, grid, threads,
                        woden_settings->ra0,
                        woden_settings->sdec0, woden_settings->cdec0,
                        d_components->ras, d_components->decs,
                        d_components->ls, d_components->ms, d_components->ns, num_components);

  //If using a gaussian primary beam, calculate beam values for all freqs,
  //lsts and point component locations
  if (beam_settings->beamtype == GAUSS_BEAM) {

    //TODO currently hardcoded to have beam position angle = 0.
    //Should this change with az/za?
    user_precision_t cos_theta = 1.0;
    user_precision_t sin_theta = 0.0;
    user_precision_t sin_2theta = 0.0;
    user_precision_t fwhm_lm = sin(beam_settings->beam_FWHM_rad);

    printf("\tDoing Gaussian Beam\n");

    calculate_gaussian_beam(num_components,
         woden_settings->num_time_steps, woden_settings->num_freqs,
         beam_settings->gauss_ha, beam_settings->gauss_sdec,
         beam_settings->gauss_cdec,
         fwhm_lm, cos_theta, sin_theta, sin_2theta,
         beam_settings->beam_ref_freq, d_freqs,
         components->beam_has,
         components->beam_decs,
         d_component_beam_gains->d_gxs, d_component_beam_gains->d_gys);

  }// end if beam == GAUSS

  else if (beam_settings->beamtype == FEE_BEAM || beam_settings->beamtype == FEE_BEAM_INTERP) {

    int num_azza = woden_settings->num_time_steps*num_components;

    double *double_azs = (double*)malloc(num_azza*sizeof(double));
    double *double_zas = (double*)malloc(num_azza*sizeof(double));

    for (int i = 0; i < num_azza; i++) {
      double_azs[i] = (double)components->azs[i];
      double_zas[i] = (double)components->zas[i];
    }

    if (beam_settings->beamtype == FEE_BEAM_INTERP) {
      printf("\tDoing the hyperbeam (interpolated)\n");
    } else {
      printf("\tDoing the hyperbeam\n");
    }


    uint8_t parallactic = 1;
    // int num_freqs = 3;
    run_hyperbeam_cuda(num_components,
           woden_settings->num_time_steps, woden_settings->num_freqs,
           parallactic,
           beam_settings->cuda_fee_beam,
           double_azs, double_zas,
           d_component_beam_gains->d_gxs, d_component_beam_gains->d_Dxs,
           d_component_beam_gains->d_Dys, d_component_beam_gains->d_gys);

    free(double_azs);
    free(double_zas);

  }

  else if (beam_settings->beamtype == ANALY_DIPOLE) {
    printf("\tDoing analytic_dipole (EDA2 beam)\n");

    calculate_analytic_dipole_beam(num_components,
         woden_settings->num_time_steps, woden_settings->num_freqs,
         components->azs, components->zas, d_freqs,
         d_component_beam_gains->d_gxs, d_component_beam_gains->d_gys);
  }

  else if (beam_settings->beamtype == MWA_ANALY) {

    //Always normalise to zenith
    int norm = 1;

    printf("\tDoing analytic MWA beam\n");

    calculate_RTS_MWA_analytic_beam(num_components,
         woden_settings->num_time_steps, woden_settings->num_freqs,
         components->azs, components->zas,
         woden_settings->FEE_ideal_delays, woden_settings->latitude,
         norm, components->beam_has, components->beam_decs,
         d_freqs, d_component_beam_gains->d_gxs, d_component_beam_gains->d_Dxs,
         d_component_beam_gains->d_Dys, d_component_beam_gains->d_gys);
  }

} //END source_component_common


__global__ void kern_calc_visi_point_or_gauss(components_t d_components,
           d_beam_gains_t d_component_beam_gains,
           user_precision_t *d_us, user_precision_t *d_vs, user_precision_t *d_ws,
           user_precision_t *d_sum_visi_XX_real, user_precision_t *d_sum_visi_XX_imag,
           user_precision_t *d_sum_visi_XY_real, user_precision_t *d_sum_visi_XY_imag,
           user_precision_t *d_sum_visi_YX_real, user_precision_t *d_sum_visi_YX_imag,
           user_precision_t *d_sum_visi_YY_real, user_precision_t *d_sum_visi_YY_imag,
           user_precision_t *d_allsteps_wavelengths, int num_components,
           int num_baselines, int num_freqs, int num_visis,
           int num_times, e_beamtype beamtype, e_component_type comptype) {

  // Start by computing which baseline we're going to do
  const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);
  if(iBaseline < num_visis) {

    user_precision_t flux_I;
    user_precision_t flux_Q;
    user_precision_t flux_U;
    user_precision_t flux_V;

    cuUserComplex visi_comp;
    cuUserComplex V_envelop;

    user_precision_t pa, sinpa, cospa, u, v, x, y, invsig_x, invsig_y;

    for (size_t iComponent = 0; iComponent < num_components; iComponent++) {

      extrap_stokes(d_allsteps_wavelengths, d_components.ref_freqs,
                   d_components.ref_stokesI, d_components.ref_stokesQ,
                   d_components.ref_stokesU, d_components.ref_stokesV,
                   d_components.SIs, iComponent, iBaseline,
                   &flux_I, &flux_Q, &flux_U, &flux_V);

      visi_comp = calc_measurement_equation(d_us, d_vs, d_ws,
                             d_components.ls, d_components.ms, d_components.ns,
                             iBaseline, iComponent);

      if (comptype == GAUSSIAN) {

        V_envelop = make_cuUserComplex( 1.0, 0.0 );

        pa = d_components.pas[iComponent];
        sinpa = sin(pa);
        cospa = cos(pa);
        u = d_us[iBaseline];
        v = d_vs[iBaseline];

        x =  cospa*v + sinpa*u; // major axis
        y = -sinpa*v + cospa*u; // minor axis
        invsig_x = d_components.majors[iComponent];
        invsig_y = d_components.minors[iComponent];

        V_envelop = make_cuUserComplex( exp( -0.5 * ( x*x*invsig_x*invsig_x*M_PI_2_2_LN_2 + y*y*invsig_y*invsig_y*M_PI_2_2_LN_2 ) ), 0.0 );

        visi_comp = visi_comp*V_envelop;
      }

      update_sum_visis(iBaseline, iComponent, num_freqs,
             num_baselines, num_components, num_times, beamtype,
             d_component_beam_gains.d_gxs, d_component_beam_gains.d_Dxs,
             d_component_beam_gains.d_Dys, d_component_beam_gains.d_gys,
             visi_comp, flux_I, flux_Q, flux_U, flux_V,
             d_sum_visi_XX_real, d_sum_visi_XX_imag,
             d_sum_visi_XY_real, d_sum_visi_XY_imag,
             d_sum_visi_YX_real, d_sum_visi_YX_imag,
             d_sum_visi_YY_real, d_sum_visi_YY_imag);
    }
  }
}

__global__ void kern_calc_visi_shapelets(components_t d_components,
      d_beam_gains_t d_component_beam_gains,
      user_precision_t *d_us, user_precision_t *d_vs, user_precision_t *d_ws,
      user_precision_t *d_allsteps_wavelengths,
      user_precision_t *d_u_shapes, user_precision_t *d_v_shapes,
      user_precision_t *d_w_shapes,
      user_precision_t *d_sum_visi_XX_real, user_precision_t *d_sum_visi_XX_imag,
      user_precision_t *d_sum_visi_XY_real, user_precision_t *d_sum_visi_XY_imag,
      user_precision_t *d_sum_visi_YX_real, user_precision_t *d_sum_visi_YX_imag,
      user_precision_t *d_sum_visi_YY_real, user_precision_t *d_sum_visi_YY_imag,
      user_precision_t *d_sbf,
      int num_shapes, int num_baselines, int num_freqs, int num_visis,
      const int num_coeffs, int num_times, e_beamtype beamtype) {

  // Start by computing which baseline we're going to do
  const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);

  if (iBaseline < num_visis) {

    user_precision_t shape_flux_I;
    user_precision_t shape_flux_Q;
    user_precision_t shape_flux_U;
    user_precision_t shape_flux_V;
    cuUserComplex visi_shape;

    for (int iCoeff = 0; iCoeff < num_coeffs; iCoeff++) {

      int iComponent = d_components.param_indexes[iCoeff];

      extrap_stokes(d_allsteps_wavelengths,  d_components.ref_freqs,
                   d_components.ref_stokesI, d_components.ref_stokesQ,
                   d_components.ref_stokesU, d_components.ref_stokesV,
                   d_components.SIs, iComponent, iBaseline,
                   &shape_flux_I, &shape_flux_Q, &shape_flux_U, &shape_flux_V);

      visi_shape = calc_measurement_equation(d_us, d_vs, d_ws,
                            d_components.ls, d_components.ms, d_components.ns,
                            iBaseline, iComponent);

      user_precision_t pa = d_components.pas[iComponent];
      user_precision_t sinpa = sin(pa);
      user_precision_t cospa = cos(pa);

      user_precision_t u_shape = d_u_shapes[iComponent*num_visis + iBaseline];
      user_precision_t v_shape = d_v_shapes[iComponent*num_visis + iBaseline];

      user_precision_t x = (cospa*v_shape + sinpa*u_shape); // major axis
      user_precision_t y = (-sinpa*v_shape + cospa*u_shape); // minor axis

      //Scales the FWHM to std to match basis functions, and account for the
      //basis functions being stored with beta = 1.0
      //Basis functions have been stored in such a way that x is in the same
      //direction as on sky, but y is opposite, so include negative here
      user_precision_t const_x = (d_components.majors[iComponent]*SQRT_M_PI_2_2_LN_2)/sbf_dx;
      user_precision_t const_y = -(d_components.minors[iComponent]*SQRT_M_PI_2_2_LN_2)/sbf_dx;

      // I^(n1+n2) = Ipow_lookup[(n1+n2) % 4]
      cuUserComplex Ipow_lookup[] = { make_cuUserComplex(  1.0,  0.0 ),
                                       make_cuUserComplex(  0.0,  1.0 ),
                                       make_cuUserComplex( -1.0,  0.0 ),
                                       make_cuUserComplex(  0.0, -1.0 ) };

      user_precision_t xlow, xhigh, ylow, yhigh, u_value, v_value, f_hat, *sbf_n;

      // find the indices in the basis functions for u*beta_u and v*beta_v

      user_precision_t xpos = x*const_x + sbf_c;
      user_precision_t ypos = y*const_y + sbf_c;

      int xindex = (int)floor(xpos);
      int yindex = (int)floor(ypos);
      //
      int n1 = (int)d_components.n1s[iCoeff];
      int n2 = (int)d_components.n2s[iCoeff];

      f_hat = d_components.shape_coeffs[iCoeff];

      sbf_n = &d_sbf[n1*sbf_L];
      xlow  = sbf_n[xindex];
      xhigh = sbf_n[xindex+1];
      u_value = xlow + (xhigh-xlow)*(xpos-xindex);

      sbf_n = &d_sbf[n2*sbf_L];
      ylow  = sbf_n[yindex];
      yhigh = sbf_n[yindex+1];
      v_value = ylow + (yhigh-ylow)*(ypos-yindex);

      // accumulate the intensity model for baseline pair (u,v)
      cuUserComplex V_envelop = make_cuUserComplex( 0.0, 0.0 );
      V_envelop = V_envelop + Ipow_lookup[(n1+n2) % 4] * f_hat * u_value*v_value;

      visi_shape = visi_shape*V_envelop;

      // printf("V_envelop %.5f %.5f\n",V_envelop.x, V_envelop.y );

      update_sum_visis(iBaseline, iComponent, num_freqs,
             num_baselines, num_shapes, num_times, beamtype,
             d_component_beam_gains.d_gxs, d_component_beam_gains.d_Dxs,
             d_component_beam_gains.d_Dys, d_component_beam_gains.d_gys,
             visi_shape,
             shape_flux_I, shape_flux_Q, shape_flux_U, shape_flux_V,
             d_sum_visi_XX_real, d_sum_visi_XX_imag,
             d_sum_visi_XY_real, d_sum_visi_XY_imag,
             d_sum_visi_YX_real, d_sum_visi_YX_imag,
             d_sum_visi_YY_real, d_sum_visi_YY_imag);
    }
  }
}



extern "C" void free_d_components(components_t d_components,
                                  e_component_type comptype){

  cudaErrorCheckCall( hipFree( d_components.ns) );
  cudaErrorCheckCall( hipFree( d_components.ms) );
  cudaErrorCheckCall( hipFree( d_components.ls) );
  cudaErrorCheckCall( hipFree( d_components.ref_freqs ) );
  cudaErrorCheckCall( hipFree( d_components.ref_stokesI ) );
  cudaErrorCheckCall( hipFree( d_components.ref_stokesQ ) );
  cudaErrorCheckCall( hipFree( d_components.ref_stokesU ) );
  cudaErrorCheckCall( hipFree( d_components.ref_stokesV ) );
  cudaErrorCheckCall( hipFree( d_components.SIs ) );
  cudaErrorCheckCall( hipFree( d_components.decs) );
  cudaErrorCheckCall( hipFree( d_components.ras) );

  if (comptype == GAUSSIAN || comptype == SHAPELET) {
    cudaErrorCheckCall( hipFree( d_components.pas ) );
    cudaErrorCheckCall( hipFree( d_components.majors ) );
    cudaErrorCheckCall( hipFree( d_components.minors ) );
  }

  if (comptype == SHAPELET) {
    cudaErrorCheckCall( hipFree( d_components.shape_coeffs ) );
    cudaErrorCheckCall( hipFree( d_components.n1s ) );
    cudaErrorCheckCall( hipFree( d_components.n2s ) );
    cudaErrorCheckCall( hipFree( d_components.param_indexes ) );
  }
}

extern "C" void free_beam_gains(d_beam_gains_t d_beam_gains, e_beamtype beamtype){

  cudaErrorCheckCall( hipFree( d_beam_gains.d_gxs) );
  cudaErrorCheckCall( hipFree( d_beam_gains.d_gys) );

  if (beamtype == FEE_BEAM){
    cudaErrorCheckCall( hipFree( d_beam_gains.d_Dxs ) );
    cudaErrorCheckCall( hipFree( d_beam_gains.d_Dys ) );
  }

}


/*******************************************************************************
                 Functions below to be used in unit tests
*******************************************************************************/

__global__ void kern_extrap_stokes(int num_extrap_freqs, int num_components,
           user_precision_t *d_extrap_wavelengths, double *d_ref_freqs,
           user_precision_t *d_SIs,
           user_precision_t *d_ref_stokesI, user_precision_t *d_ref_stokesQ,
           user_precision_t *d_ref_stokesU, user_precision_t *d_ref_stokesV,
           user_precision_t *d_flux_I, user_precision_t *d_flux_Q,
           user_precision_t *d_flux_U, user_precision_t *d_flux_V ) {

  // Start by computing which baseline we're going to do
  const int iComponent = threadIdx.x + (blockDim.x*blockIdx.x);
  const int iFreq = threadIdx.y + (blockDim.y*blockIdx.y);
  // if(iBaseline < num_visis && iComponent < num_points) {
  if(iComponent < num_components && iFreq < num_extrap_freqs) {

    user_precision_t flux_I;
    user_precision_t flux_Q;
    user_precision_t flux_U;
    user_precision_t flux_V;

    extrap_stokes(d_extrap_wavelengths, d_ref_freqs,
                 d_ref_stokesI, d_ref_stokesQ,
                 d_ref_stokesU, d_ref_stokesV,
                 d_SIs, iComponent, iFreq,
                 &flux_I, &flux_Q, &flux_U, &flux_V);

    int extrap_ind = num_components*iComponent + iFreq;

    d_flux_I[extrap_ind] = flux_I;
    d_flux_Q[extrap_ind] = flux_Q;
    d_flux_U[extrap_ind] = flux_U;
    d_flux_V[extrap_ind] = flux_V;

  }
}

extern "C" void test_kern_extrap_stokes(int num_extrap_freqs, int num_components,
           user_precision_t *extrap_wavelengths, double *ref_freqs,
           user_precision_t *SIs,
           user_precision_t *ref_stokesI, user_precision_t *ref_stokesQ,
           user_precision_t *ref_stokesU, user_precision_t *ref_stokesV,
           user_precision_t *flux_I, user_precision_t *flux_Q,
           user_precision_t *flux_U, user_precision_t *flux_V){

  user_precision_t *d_extrap_wavelengths = NULL;
  double *d_ref_freqs = NULL;
  user_precision_t *d_SIs = NULL;
  user_precision_t *d_ref_stokesI = NULL;
  user_precision_t *d_ref_stokesQ = NULL;
  user_precision_t *d_ref_stokesU = NULL;
  user_precision_t *d_ref_stokesV = NULL;
  user_precision_t *d_flux_I = NULL;
  user_precision_t *d_flux_Q = NULL;
  user_precision_t *d_flux_U = NULL;
  user_precision_t *d_flux_V = NULL;

  cudaErrorCheckCall( hipMalloc( (void**)&d_extrap_wavelengths,
                                   num_extrap_freqs*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_ref_freqs,
                                     num_components*sizeof(double) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_SIs,
                                     num_components*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_ref_stokesI,
                                     num_components*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_ref_stokesQ,
                                     num_components*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_ref_stokesU,
                                     num_components*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_ref_stokesV,
                                     num_components*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_flux_I,
                    num_components*num_extrap_freqs*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_flux_Q,
                    num_components*num_extrap_freqs*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_flux_U,
                    num_components*num_extrap_freqs*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_flux_V,
                    num_components*num_extrap_freqs*sizeof(user_precision_t) ));

  cudaErrorCheckCall( hipMemcpy(d_extrap_wavelengths, extrap_wavelengths,
           num_extrap_freqs*sizeof(user_precision_t), hipMemcpyHostToDevice ));

  cudaErrorCheckCall( hipMemcpy(d_ref_stokesI, ref_stokesI,
             num_components*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_ref_stokesQ, ref_stokesQ,
             num_components*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_ref_stokesU, ref_stokesU,
             num_components*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_ref_stokesV, ref_stokesV,
             num_components*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_ref_freqs, ref_freqs,
             num_components*sizeof(double), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_SIs, SIs,
             num_components*sizeof(user_precision_t), hipMemcpyHostToDevice ));

  dim3 grid, threads;

  threads.x = 16;
  threads.y = 16;
  grid.x = (int)ceilf( (float)num_components / (float)threads.x );
  grid.y = (int)ceilf( (float)num_extrap_freqs / (float)threads.y );

  cudaErrorCheckKernel("kern_extrap_stokes",
                        kern_extrap_stokes, grid, threads,
                        num_extrap_freqs, num_components,
                        d_extrap_wavelengths, d_ref_freqs, d_SIs,
                        d_ref_stokesI, d_ref_stokesQ,
                        d_ref_stokesU, d_ref_stokesV,
                        d_flux_I, d_flux_Q,
                        d_flux_U, d_flux_V);

  cudaErrorCheckCall( hipMemcpy(flux_I, d_flux_I,
                      num_components*num_extrap_freqs*sizeof(user_precision_t),
                                                      hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(flux_Q, d_flux_Q,
                      num_components*num_extrap_freqs*sizeof(user_precision_t),
                                                      hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(flux_U, d_flux_U,
                      num_components*num_extrap_freqs*sizeof(user_precision_t),
                                                      hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(flux_V, d_flux_V,
                      num_components*num_extrap_freqs*sizeof(user_precision_t),
                                                      hipMemcpyDeviceToHost ));

  cudaErrorCheckCall( hipFree( d_extrap_wavelengths ) );
  cudaErrorCheckCall( hipFree( d_ref_freqs ) );
  cudaErrorCheckCall( hipFree( d_SIs ) );
  cudaErrorCheckCall( hipFree( d_ref_stokesI ) );
  cudaErrorCheckCall( hipFree( d_ref_stokesQ ) );
  cudaErrorCheckCall( hipFree( d_ref_stokesU ) );
  cudaErrorCheckCall( hipFree( d_ref_stokesV ) );
  cudaErrorCheckCall( hipFree( d_flux_I ) );
  cudaErrorCheckCall( hipFree( d_flux_Q ) );
  cudaErrorCheckCall( hipFree( d_flux_U ) );
  cudaErrorCheckCall( hipFree( d_flux_V ) );
}


__global__ void kern_calc_measurement_equation(int num_components, int num_baselines,
          user_precision_t *d_us, user_precision_t *d_vs, user_precision_t *d_ws,
          double *d_ls, double *d_ms, double *d_ns, cuUserComplex *d_visis) {

  // Start by computing which baseline we're going to do
  const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);
  const int iComponent = threadIdx.y + (blockDim.y*blockIdx.y);

  if(iComponent < num_components && iBaseline < num_baselines) {

    cuUserComplex visi;
    visi = calc_measurement_equation(d_us, d_vs, d_ws, d_ls, d_ms, d_ns,
                                     iBaseline, iComponent);

    int visi_ind = num_components*iBaseline + iComponent;
    d_visis[visi_ind] = visi;

  }
}

extern "C" void test_kern_calc_measurement_equation(int num_components,
          int num_baselines,
          user_precision_t *us, user_precision_t *vs, user_precision_t *ws,
          double *ls, double *ms, double *ns, user_precision_complex_t *visis){

  user_precision_t *d_us = NULL;
  user_precision_t *d_vs = NULL;
  user_precision_t *d_ws = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_us, num_baselines*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_vs, num_baselines*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_ws, num_baselines*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMemcpy(d_us, us, num_baselines*sizeof(user_precision_t),
                                                        hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_vs, vs, num_baselines*sizeof(user_precision_t),
                                                        hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_ws, ws, num_baselines*sizeof(user_precision_t),
                                                        hipMemcpyHostToDevice ));

  double *d_ls = NULL;
  double *d_ms = NULL;
  double *d_ns = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_ls, num_components*sizeof(double) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_ms, num_components*sizeof(double) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_ns, num_components*sizeof(double) ));
  cudaErrorCheckCall( hipMemcpy(d_ls, ls, num_components*sizeof(double),
                                                      hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_ms, ms, num_components*sizeof(double),
                                                      hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_ns, ns, num_components*sizeof(double),
                                                      hipMemcpyHostToDevice ));

  user_precision_complex_t *d_visis = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_visis, num_baselines*num_components*sizeof(user_precision_complex_t) ));

  dim3 grid, threads;

  threads.x = 16;
  threads.y = 16;
  grid.x = (int)ceilf( (float)num_baselines / (float)threads.x );
  grid.y = (int)ceilf( (float)num_components / (float)threads.y );

  cudaErrorCheckKernel("kern_calc_measurement_equation",
                      kern_calc_measurement_equation, grid, threads,
                      num_components, num_baselines,
                      d_us, d_vs, d_ws,
                      d_ls, d_ms, d_ns,
                      (cuUserComplex*)d_visis );

  cudaErrorCheckCall( hipMemcpy(visis, (user_precision_complex_t*)d_visis, num_components*num_baselines*sizeof(user_precision_complex_t),hipMemcpyDeviceToHost ));

  cudaErrorCheckCall( hipFree( d_us ) );
  cudaErrorCheckCall( hipFree( d_vs ) );
  cudaErrorCheckCall( hipFree( d_ws ) );
  cudaErrorCheckCall( hipFree( d_ls ) );
  cudaErrorCheckCall( hipFree( d_ms ) );
  cudaErrorCheckCall( hipFree( d_ns ) );
  cudaErrorCheckCall( hipFree(d_visis ) );

}

__global__ void kern_apply_beam_gains(int num_gains, cuUserComplex *d_g1xs,
          cuUserComplex *d_D1xs,
          cuUserComplex *d_D1ys, cuUserComplex *d_g1ys,
          cuUserComplex *d_g2xs, cuUserComplex *d_D2xs,
          cuUserComplex *d_D2ys, cuUserComplex *d_g2ys,
          user_precision_t *d_flux_Is, user_precision_t *d_flux_Qs,
          user_precision_t *d_flux_Us, user_precision_t *d_flux_Vs,
          cuUserComplex *d_visi_components,
          cuUserComplex *d_visi_XXs, cuUserComplex *d_visi_XYs,
          cuUserComplex *d_visi_YXs, cuUserComplex *d_visi_YYs) {

  const int iGain = threadIdx.x + (blockDim.x*blockIdx.x);
  // const int iComponent = threadIdx.y + (blockDim.y*blockIdx.y);
  // if(iBaseline < num_visis && iComponent < num_points) {
  if (iGain < num_gains) {

    cuUserComplex visi_XX;
    cuUserComplex visi_XY;
    cuUserComplex visi_YX;
    cuUserComplex visi_YY;

    apply_beam_gains(d_g1xs[iGain], d_D1xs[iGain],
             d_D1ys[iGain], d_g1ys[iGain],
             d_g2xs[iGain], d_D2xs[iGain],
             d_D2ys[iGain], d_g2ys[iGain],
             d_flux_Is[iGain], d_flux_Qs[iGain],
             d_flux_Us[iGain], d_flux_Vs[iGain],
             d_visi_components[iGain],
             &visi_XX, &visi_XY,
             &visi_YX, &visi_YY);

    d_visi_XXs[iGain] = visi_XX;
    d_visi_XYs[iGain] = visi_XY;
    d_visi_YXs[iGain] = visi_YX;
    d_visi_YYs[iGain] = visi_YY;

  }
}

extern "C" void test_kern_apply_beam_gains(int num_gains, user_precision_complex_t *g1xs,
          user_precision_complex_t *D1xs,
          user_precision_complex_t *D1ys, user_precision_complex_t *g1ys,
          user_precision_complex_t *g2xs, user_precision_complex_t *D2xs,
          user_precision_complex_t *D2ys, user_precision_complex_t *g2ys,
          user_precision_t *flux_Is, user_precision_t *flux_Qs,
          user_precision_t *flux_Us, user_precision_t *flux_Vs,
          user_precision_complex_t *visi_components,
          user_precision_complex_t *visi_XXs, user_precision_complex_t *visi_XYs,
          user_precision_complex_t *visi_YXs, user_precision_complex_t *visi_YYs){

  user_precision_complex_t *d_g1xs = NULL;
  user_precision_complex_t *d_D1xs = NULL;
  user_precision_complex_t *d_D1ys = NULL;
  user_precision_complex_t *d_g1ys = NULL;
  user_precision_complex_t *d_g2xs = NULL;
  user_precision_complex_t *d_D2xs = NULL;
  user_precision_complex_t *d_D2ys = NULL;
  user_precision_complex_t *d_g2ys = NULL;
  user_precision_t *d_flux_Is = NULL;
  user_precision_t *d_flux_Qs = NULL;
  user_precision_t *d_flux_Us = NULL;
  user_precision_t *d_flux_Vs = NULL;
  user_precision_complex_t *d_visi_components = NULL;
  user_precision_complex_t *d_visi_XXs = NULL;
  user_precision_complex_t *d_visi_XYs = NULL;
  user_precision_complex_t *d_visi_YXs = NULL;
  user_precision_complex_t *d_visi_YYs = NULL;

  cudaErrorCheckCall( hipMalloc( (void**)&d_g1xs,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_D1xs,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_D1ys,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_g1ys,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_g2xs,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_D2xs,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_D2ys,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_g2ys,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_flux_Is,
                                          num_gains*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_flux_Qs,
                                          num_gains*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_flux_Us,
                                          num_gains*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_flux_Vs,
                                          num_gains*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_visi_components,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_visi_XXs,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_visi_XYs,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_visi_YXs,
                                  num_gains*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_visi_YYs,
                                  num_gains*sizeof(user_precision_complex_t) ));

  cudaErrorCheckCall( hipMemcpy(d_g1xs, g1xs,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_D1xs, D1xs,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_D1ys, D1ys,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_g1ys, g1ys,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_g2xs, g2xs,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_D2xs, D2xs,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_D2ys, D2ys,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_g2ys, g2ys,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_visi_components, visi_components,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_visi_XXs, visi_XXs,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_visi_XYs, visi_XYs,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_visi_YXs, visi_YXs,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_visi_YYs, visi_YYs,
          num_gains*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));

  cudaErrorCheckCall( hipMemcpy(d_flux_Is, flux_Is,
                             num_gains*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_flux_Qs, flux_Qs,
                             num_gains*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_flux_Us, flux_Us,
                             num_gains*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_flux_Vs, flux_Vs,
                             num_gains*sizeof(user_precision_t), hipMemcpyHostToDevice ));

  dim3 grid, threads;

  threads.x = 128;
  grid.x = (int)ceil( (user_precision_t)num_gains / (user_precision_t)threads.x );

  cudaErrorCheckKernel("kern_apply_beam_gains",
                      kern_apply_beam_gains, grid, threads,
                      num_gains,
                      (cuUserComplex *)d_g1xs, (cuUserComplex *)d_D1xs,
                      (cuUserComplex *)d_D1ys, (cuUserComplex *)d_g1ys,
                      (cuUserComplex *)d_g2xs, (cuUserComplex *)d_D2xs,
                      (cuUserComplex *)d_D2ys, (cuUserComplex *)d_g2ys,
                      d_flux_Is, d_flux_Qs,
                      d_flux_Us, d_flux_Vs,
                      (cuUserComplex *)d_visi_components,
                      (cuUserComplex *)d_visi_XXs, (cuUserComplex *)d_visi_XYs,
                      (cuUserComplex *)d_visi_YXs, (cuUserComplex *)d_visi_YYs );

  cudaErrorCheckCall( hipMemcpy(visi_XXs, d_visi_XXs,
           num_gains*sizeof(user_precision_complex_t),hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(visi_XYs, d_visi_XYs,
           num_gains*sizeof(user_precision_complex_t),hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(visi_YXs, d_visi_YXs,
           num_gains*sizeof(user_precision_complex_t),hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(visi_YYs, d_visi_YYs,
           num_gains*sizeof(user_precision_complex_t),hipMemcpyDeviceToHost ));

  cudaErrorCheckCall( hipFree( d_g1xs ) );
  cudaErrorCheckCall( hipFree( d_D1xs ) );
  cudaErrorCheckCall( hipFree( d_D1ys ) );
  cudaErrorCheckCall( hipFree( d_g1ys ) );
  cudaErrorCheckCall( hipFree( d_g2xs ) );
  cudaErrorCheckCall( hipFree( d_D2xs ) );
  cudaErrorCheckCall( hipFree( d_D2ys ) );
  cudaErrorCheckCall( hipFree( d_g2ys ) );
  cudaErrorCheckCall( hipFree( d_flux_Is ) );
  cudaErrorCheckCall( hipFree( d_flux_Qs ) );
  cudaErrorCheckCall( hipFree( d_flux_Us ) );
  cudaErrorCheckCall( hipFree( d_flux_Vs ) );
  cudaErrorCheckCall( hipFree( d_visi_components ) );
  cudaErrorCheckCall( hipFree( d_visi_XXs ) );
  cudaErrorCheckCall( hipFree( d_visi_XYs ) );
  cudaErrorCheckCall( hipFree( d_visi_YXs ) );
  cudaErrorCheckCall( hipFree( d_visi_YYs ) );

}

__global__ void kern_get_beam_gains(int num_components, int num_baselines,
           int num_freqs, int num_visis, int num_times, int beamtype,
           cuUserComplex *d_primay_beam_J00, cuUserComplex *d_primay_beam_J01,
           cuUserComplex *d_primay_beam_J10, cuUserComplex *d_primay_beam_J11,
           cuUserComplex *d_recov_g1x, cuUserComplex *d_recov_D1x,
           cuUserComplex *d_recov_D1y, cuUserComplex *d_recov_g1y,
           cuUserComplex *d_recov_g2x, cuUserComplex *d_recov_D2x,
           cuUserComplex *d_recov_D2y, cuUserComplex *d_recov_g2y) {

  // Start by computing which baseline we're going to do
  const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);
  // const int iComponent = threadIdx.y + (blockDim.y*blockIdx.y);
  // if(iBaseline < num_visis && iComponent < num_points) {
  if(iBaseline < num_visis) {

    for (int iComponent = 0; iComponent < num_components; iComponent++) {

      cuUserComplex g1x;
      cuUserComplex D1x;
      cuUserComplex D1y;
      cuUserComplex g1y;
      cuUserComplex g2x;
      cuUserComplex D2x;
      cuUserComplex D2y;
      cuUserComplex g2y;

      get_beam_gains(iBaseline, iComponent, num_freqs,
                 num_baselines, num_components, num_times, beamtype,
                 d_primay_beam_J00, d_primay_beam_J01,
                 d_primay_beam_J10, d_primay_beam_J11,
                 &g1x, &D1x, &D1y, &g1y, &g2x, &D2x, &D2y, &g2y);

      int out_ind = num_visis*iComponent + iBaseline;

      d_recov_g1x[out_ind] = g1x;
      d_recov_D1x[out_ind] = D1x;
      d_recov_D1y[out_ind] = D1y;
      d_recov_g1y[out_ind] = g1y;
      d_recov_g2x[out_ind] = g2x;
      d_recov_D2x[out_ind] = D2x;
      d_recov_D2y[out_ind] = D2y;
      d_recov_g2y[out_ind] = g2y;

    }
  }
}

extern "C" void test_kern_get_beam_gains(int num_freqs, int num_visis,
          int num_baselines, int num_components, int num_times, int beamtype,
          user_precision_complex_t *primay_beam_J00, user_precision_complex_t *primay_beam_J01,
          user_precision_complex_t *primay_beam_J10, user_precision_complex_t *primay_beam_J11,
          user_precision_complex_t *recover_g1x, user_precision_complex_t *recover_D1x,
          user_precision_complex_t *recover_D1y, user_precision_complex_t *recover_g1y,
          user_precision_complex_t *recover_g2x, user_precision_complex_t *recover_D2x,
          user_precision_complex_t *recover_D2y, user_precision_complex_t *recover_g2y){

  user_precision_complex_t *d_recover_g1x = NULL;
  user_precision_complex_t *d_recover_D1x = NULL;
  user_precision_complex_t *d_recover_D1y = NULL;
  user_precision_complex_t *d_recover_g1y = NULL;
  user_precision_complex_t *d_recover_g2x = NULL;
  user_precision_complex_t *d_recover_D2x = NULL;
  user_precision_complex_t *d_recover_D2y = NULL;
  user_precision_complex_t *d_recover_g2y = NULL;

  user_precision_complex_t *d_primay_beam_J00 = NULL;
  user_precision_complex_t *d_primay_beam_J01 = NULL;
  user_precision_complex_t *d_primay_beam_J10 = NULL;
  user_precision_complex_t *d_primay_beam_J11 = NULL;

  cudaErrorCheckCall( hipMalloc( (void**)&d_recover_g1x, num_components*num_visis*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_recover_D1x, num_components*num_visis*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_recover_D1y, num_components*num_visis*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_recover_g1y, num_components*num_visis*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_recover_g2x, num_components*num_visis*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_recover_D2x, num_components*num_visis*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_recover_D2y, num_components*num_visis*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_recover_g2y, num_components*num_visis*sizeof(user_precision_complex_t) ));

  cudaErrorCheckCall( hipMalloc( (void**)&d_primay_beam_J00, num_freqs*num_times*num_components*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_primay_beam_J01, num_freqs*num_times*num_components*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_primay_beam_J10, num_freqs*num_times*num_components*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_primay_beam_J11, num_freqs*num_times*num_components*sizeof(user_precision_complex_t) ));

  cudaErrorCheckCall( hipMemcpy(d_primay_beam_J00, primay_beam_J00, num_freqs*num_times*num_components*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_primay_beam_J01, primay_beam_J01, num_freqs*num_times*num_components*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_primay_beam_J10, primay_beam_J10, num_freqs*num_times*num_components*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_primay_beam_J11, primay_beam_J11, num_freqs*num_times*num_components*sizeof(user_precision_complex_t), hipMemcpyHostToDevice ));

  dim3 grid, threads;

  threads.x = 128;
  grid.x = (int)ceil( (user_precision_t)num_visis / (user_precision_t)threads.x );

  cudaErrorCheckKernel("kern_get_beam_gains",
                      kern_get_beam_gains, grid, threads,
                      num_components, num_baselines,
                      num_freqs, num_visis, num_times, beamtype,
                      (cuUserComplex *)d_primay_beam_J00,
                      (cuUserComplex *)d_primay_beam_J01,
                      (cuUserComplex *)d_primay_beam_J10,
                      (cuUserComplex *)d_primay_beam_J11,
                      (cuUserComplex *)d_recover_g1x, (cuUserComplex *)d_recover_D1x,
                      (cuUserComplex *)d_recover_D1y, (cuUserComplex *)d_recover_g1y,
                      (cuUserComplex *)d_recover_g2x, (cuUserComplex *)d_recover_D2x,
                      (cuUserComplex *)d_recover_D2y, (cuUserComplex *)d_recover_g2y );

  cudaErrorCheckCall( hipMemcpy(recover_g1x, d_recover_g1x, num_components*num_visis*sizeof(user_precision_complex_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(recover_D1x, d_recover_D1x, num_components*num_visis*sizeof(user_precision_complex_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(recover_D1y, d_recover_D1y, num_components*num_visis*sizeof(user_precision_complex_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(recover_g1y, d_recover_g1y, num_components*num_visis*sizeof(user_precision_complex_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(recover_g2x, d_recover_g2x, num_components*num_visis*sizeof(user_precision_complex_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(recover_D2x, d_recover_D2x, num_components*num_visis*sizeof(user_precision_complex_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(recover_D2y, d_recover_D2y, num_components*num_visis*sizeof(user_precision_complex_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(recover_g2y, d_recover_g2y, num_components*num_visis*sizeof(user_precision_complex_t), hipMemcpyDeviceToHost ));

  cudaErrorCheckCall( hipFree( d_recover_g1x ) );
  cudaErrorCheckCall( hipFree( d_recover_D1x ) );
  cudaErrorCheckCall( hipFree( d_recover_D1y ) );
  cudaErrorCheckCall( hipFree( d_recover_g1y ) );
  cudaErrorCheckCall( hipFree( d_recover_g2x ) );
  cudaErrorCheckCall( hipFree( d_recover_D2x ) );
  cudaErrorCheckCall( hipFree( d_recover_D2y ) );
  cudaErrorCheckCall( hipFree( d_recover_g2y ) );

  cudaErrorCheckCall( hipFree( d_primay_beam_J00 ) );
  cudaErrorCheckCall( hipFree( d_primay_beam_J01 ) );
  cudaErrorCheckCall( hipFree( d_primay_beam_J10 ) );
  cudaErrorCheckCall( hipFree( d_primay_beam_J11 ) );

}

__global__ void kern_update_sum_visis(int num_freqs,
     int num_baselines, int num_components, int num_times, int beamtype,
     cuUserComplex *d_primay_beam_J00, cuUserComplex *d_primay_beam_J01,
     cuUserComplex *d_primay_beam_J10, cuUserComplex *d_primay_beam_J11,
     cuUserComplex *d_visi_components,
     user_precision_t *d_flux_I, user_precision_t *d_flux_Q,
     user_precision_t *d_flux_U, user_precision_t *d_flux_V,
     user_precision_t *d_sum_visi_XX_real, user_precision_t *d_sum_visi_XX_imag,
     user_precision_t *d_sum_visi_XY_real, user_precision_t *d_sum_visi_XY_imag,
     user_precision_t *d_sum_visi_YX_real, user_precision_t *d_sum_visi_YX_imag,
     user_precision_t *d_sum_visi_YY_real, user_precision_t *d_sum_visi_YY_imag) {

  // Start by computing which baseline we're going to do
  const int iBaseline = threadIdx.x + (blockDim.x*blockIdx.x);

  if(iBaseline < num_freqs*num_baselines*num_times) {

    int time_ind = (int)floorf( (user_precision_t)iBaseline / ((user_precision_t)num_baselines * (user_precision_t)num_freqs));
    int freq_ind = (int)floorf( ((user_precision_t)iBaseline - ((user_precision_t)time_ind*(user_precision_t)num_baselines * (user_precision_t)num_freqs)) / (user_precision_t)num_baselines);

    for (int iComponent = 0; iComponent < num_components; iComponent++) {

      //There is a flux for every frequnecy and component
      int flux_ind = num_components*freq_ind + iComponent;

      update_sum_visis(iBaseline, iComponent, num_freqs,
             num_baselines, num_components, num_times, beamtype,
             d_primay_beam_J00, d_primay_beam_J01,
             d_primay_beam_J10, d_primay_beam_J11,
             d_visi_components[iBaseline],
             d_flux_I[flux_ind], d_flux_Q[flux_ind],
             d_flux_U[flux_ind], d_flux_V[flux_ind],
             d_sum_visi_XX_real, d_sum_visi_XX_imag,
             d_sum_visi_XY_real, d_sum_visi_XY_imag,
             d_sum_visi_YX_real, d_sum_visi_YX_imag,
             d_sum_visi_YY_real, d_sum_visi_YY_imag);

    }
  }
}

extern "C" void test_kern_update_sum_visis(int num_freqs, int num_visis,
          int num_baselines, int num_components, int num_times, int beamtype,
          user_precision_complex_t *primay_beam_J00,
          user_precision_complex_t *primay_beam_J01,
          user_precision_complex_t *primay_beam_J10,
          user_precision_complex_t *primay_beam_J11,
          user_precision_complex_t *visi_components,
          user_precision_t *flux_I, user_precision_t *flux_Q,
          user_precision_t *flux_U, user_precision_t *flux_V,
          user_precision_t *sum_visi_XX_real, user_precision_t *sum_visi_XX_imag,
          user_precision_t *sum_visi_XY_real, user_precision_t *sum_visi_XY_imag,
          user_precision_t *sum_visi_YX_real, user_precision_t *sum_visi_YX_imag,
          user_precision_t *sum_visi_YY_real, user_precision_t *sum_visi_YY_imag){

  user_precision_complex_t *d_primay_beam_J00 = NULL;
  user_precision_complex_t *d_primay_beam_J01 = NULL;
  user_precision_complex_t *d_primay_beam_J10 = NULL;
  user_precision_complex_t *d_primay_beam_J11 = NULL;
  user_precision_complex_t *d_visi_components = NULL;

  cudaErrorCheckCall( hipMalloc( (void**)&d_primay_beam_J00,
                    num_components*num_times*num_freqs*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_primay_beam_J01,
                    num_components*num_times*num_freqs*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_primay_beam_J10,
                    num_components*num_times*num_freqs*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_primay_beam_J11,
                    num_components*num_times*num_freqs*sizeof(user_precision_complex_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_visi_components,
                    num_visis*sizeof(user_precision_complex_t) ));

  cudaErrorCheckCall( hipMemcpy(d_primay_beam_J00, primay_beam_J00,
            num_components*num_times*num_freqs*sizeof(user_precision_complex_t),
            hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_primay_beam_J01, primay_beam_J01,
            num_components*num_times*num_freqs*sizeof(user_precision_complex_t),
            hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_primay_beam_J10, primay_beam_J10,
            num_components*num_times*num_freqs*sizeof(user_precision_complex_t),
            hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_primay_beam_J11, primay_beam_J11,
            num_components*num_times*num_freqs*sizeof(user_precision_complex_t),
            hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_visi_components, visi_components,
                                     num_visis*sizeof(user_precision_complex_t),
                                     hipMemcpyHostToDevice ));

  user_precision_t *d_flux_I = NULL;
  user_precision_t *d_flux_Q = NULL;
  user_precision_t *d_flux_U = NULL;
  user_precision_t *d_flux_V = NULL;

  cudaErrorCheckCall( hipMalloc( (void**)&d_flux_I, num_components*num_times*num_freqs*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_flux_Q, num_components*num_times*num_freqs*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_flux_U, num_components*num_times*num_freqs*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_flux_V, num_components*num_times*num_freqs*sizeof(user_precision_t) ));

  cudaErrorCheckCall( hipMemcpy(d_flux_I, flux_I,
                    num_components*num_times*num_freqs*sizeof(user_precision_t),    hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_flux_Q, flux_Q,
                    num_components*num_times*num_freqs*sizeof(user_precision_t),    hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_flux_U, flux_U,
                    num_components*num_times*num_freqs*sizeof(user_precision_t),    hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_flux_V, flux_V,
                    num_components*num_times*num_freqs*sizeof(user_precision_t),    hipMemcpyHostToDevice ));

  user_precision_t *d_sum_visi_XX_real = NULL;
  user_precision_t *d_sum_visi_XY_real = NULL;
  user_precision_t *d_sum_visi_YX_real = NULL;
  user_precision_t *d_sum_visi_YY_real = NULL;
  user_precision_t *d_sum_visi_XX_imag = NULL;
  user_precision_t *d_sum_visi_XY_imag = NULL;
  user_precision_t *d_sum_visi_YX_imag = NULL;
  user_precision_t *d_sum_visi_YY_imag = NULL;

  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_XX_real,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_XY_real,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_YX_real,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_YY_real,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_XX_imag,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_XY_imag,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_YX_imag,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_YY_imag,
                                          num_visis*sizeof(user_precision_t) ));

  dim3 grid, threads;

  threads.x = 128;
  grid.x = (int)ceil( (user_precision_t)num_visis / (user_precision_t)threads.x );

  cudaErrorCheckKernel("kern_update_sum_visis",
                      kern_update_sum_visis, grid, threads,
                      num_freqs, num_baselines, num_components, num_times, beamtype,
                      (cuUserComplex *)d_primay_beam_J00, (cuUserComplex *)d_primay_beam_J01,
                      (cuUserComplex *)d_primay_beam_J10, (cuUserComplex *)d_primay_beam_J11,
                      (cuUserComplex *)d_visi_components,
                      d_flux_I, d_flux_Q, d_flux_U, d_flux_V,
                      d_sum_visi_XX_real, d_sum_visi_XX_imag,
                      d_sum_visi_XY_real, d_sum_visi_XY_imag,
                      d_sum_visi_YX_real, d_sum_visi_YX_imag,
                      d_sum_visi_YY_real, d_sum_visi_YY_imag );

  cudaErrorCheckCall( hipMemcpy(sum_visi_XX_real, d_sum_visi_XX_real,
                  num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_XY_real, d_sum_visi_XY_real,
                  num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_YX_real, d_sum_visi_YX_real,
                  num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_YY_real, d_sum_visi_YY_real,
                  num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_XX_imag, d_sum_visi_XX_imag,
                  num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_XY_imag, d_sum_visi_XY_imag,
                  num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_YX_imag, d_sum_visi_YX_imag,
                  num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_YY_imag, d_sum_visi_YY_imag,
                  num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));

  cudaErrorCheckCall( hipFree( d_primay_beam_J00 ) );
  cudaErrorCheckCall( hipFree( d_primay_beam_J01 ) );
  cudaErrorCheckCall( hipFree( d_primay_beam_J10 ) );
  cudaErrorCheckCall( hipFree( d_primay_beam_J11 ) );
  cudaErrorCheckCall( hipFree( d_visi_components ) );
  cudaErrorCheckCall( hipFree( d_flux_I ) );
  cudaErrorCheckCall( hipFree( d_flux_Q ) );
  cudaErrorCheckCall( hipFree( d_flux_U ) );
  cudaErrorCheckCall( hipFree( d_flux_V ) );
  cudaErrorCheckCall( hipFree( d_sum_visi_XX_real ) );
  cudaErrorCheckCall( hipFree( d_sum_visi_XY_real ) );
  cudaErrorCheckCall( hipFree( d_sum_visi_YX_real ) );
  cudaErrorCheckCall( hipFree( d_sum_visi_YY_real ) );
  cudaErrorCheckCall( hipFree( d_sum_visi_XX_imag ) );
  cudaErrorCheckCall( hipFree( d_sum_visi_XY_imag ) );
  cudaErrorCheckCall( hipFree( d_sum_visi_YX_imag ) );
  cudaErrorCheckCall( hipFree( d_sum_visi_YY_imag ) );

}


extern "C" void test_source_component_common(int num_components,
           int num_shape_coeffs, components_t components,
           components_t d_components,
           double *freqs, woden_settings_t *woden_settings,
           beam_settings_t *beam_settings,
           user_precision_complex_t *gxs, user_precision_complex_t *Dxs,
           user_precision_complex_t *Dys, user_precision_complex_t *gys,
           double *ls, double *ms, double *ns){

  double *d_freqs = NULL;
  cudaErrorCheckCall( hipMalloc( (void**)&d_freqs,
                                     woden_settings->num_freqs*sizeof(double) ));
  cudaErrorCheckCall( hipMemcpy( d_freqs, freqs,
             woden_settings->num_freqs*sizeof(double), hipMemcpyHostToDevice) );


  d_beam_gains_t d_beam_gains;

  source_component_common(num_components, num_shape_coeffs,
             &components, d_freqs, woden_settings, beam_settings, POINT,
             &d_components, &d_beam_gains);

  int num_beam_values = num_components*woden_settings->num_freqs*woden_settings->num_time_steps;

  cudaErrorCheckCall( hipMemcpy(gxs, (user_precision_complex_t*)d_beam_gains.d_gxs,
              num_beam_values*sizeof(cuUserComplex), hipMemcpyDeviceToHost ));

  cudaErrorCheckCall( hipMemcpy(gys, (user_precision_complex_t*)d_beam_gains.d_gys,
              num_beam_values*sizeof(cuUserComplex), hipMemcpyDeviceToHost ));

  if (beam_settings->beamtype == FEE_BEAM || beam_settings->beamtype == FEE_BEAM_INTERP || beam_settings->beamtype == MWA_ANALY) {
    cudaErrorCheckCall( hipMemcpy(Dxs, (user_precision_complex_t*)d_beam_gains.d_Dxs,
                num_beam_values*sizeof(cuUserComplex), hipMemcpyDeviceToHost ));
    cudaErrorCheckCall( hipMemcpy(Dys, (user_precision_complex_t*)d_beam_gains.d_Dys,
                num_beam_values*sizeof(cuUserComplex), hipMemcpyDeviceToHost ));
  }

  cudaErrorCheckCall( hipMemcpy(ls, d_components.ls,
                        num_components*sizeof(double), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(ms, d_components.ms,
                        num_components*sizeof(double), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(ns, d_components.ns,
                        num_components*sizeof(double), hipMemcpyDeviceToHost ));

  free_d_components(d_components, POINT);
  free_beam_gains(d_beam_gains, beam_settings->beamtype);
}

extern "C" void test_kern_calc_visi_all(int num_components,
          int num_baselines, int num_shape_coeffs,
          int num_freqs, int num_visis, int num_times,
          e_beamtype beamtype, e_component_type comptype,
          components_t components,
          user_precision_t *us, user_precision_t *vs, user_precision_t *ws,
          user_precision_t *u_shapes, user_precision_t *v_shapes, user_precision_t *w_shapes,
          user_precision_t *sum_visi_XX_real, user_precision_t *sum_visi_XX_imag,
          user_precision_t *sum_visi_XY_real, user_precision_t *sum_visi_XY_imag,
          user_precision_t *sum_visi_YX_real, user_precision_t *sum_visi_YX_imag,
          user_precision_t *sum_visi_YY_real, user_precision_t *sum_visi_YY_imag,
          user_precision_t *allsteps_wavelengths, user_precision_t *sbf,
          user_precision_complex_t *gxs, user_precision_complex_t *Dxs,
          user_precision_complex_t *Dys, user_precision_complex_t *gys){

  user_precision_t *d_us = NULL;
  user_precision_t *d_vs = NULL;
  user_precision_t *d_ws = NULL;
  user_precision_t *d_allsteps_wavelengths = NULL;

  cudaErrorCheckCall( hipMalloc( (void**)&d_us, num_visis*sizeof(user_precision_t) ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_vs, num_visis*sizeof(user_precision_t) ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_ws, num_visis*sizeof(user_precision_t) ) );
  cudaErrorCheckCall( hipMalloc( (void**)&d_allsteps_wavelengths, num_visis*sizeof(user_precision_t) ) );

  cudaErrorCheckCall( hipMemcpy(d_us, us,
                             num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_vs, vs,
                             num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_ws, ws,
                             num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_allsteps_wavelengths, allsteps_wavelengths,
                             num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ));

  //Something to store the device component types in
  components_t d_components;

  //just malloc these here are the overall free-ing function later needs to free them
  cudaErrorCheckCall( hipMalloc( (void**)&d_components.ras,
                                                num_components*sizeof(double) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_components.decs,
                                                num_components*sizeof(double) ));

  cudaErrorCheckCall( hipMalloc( (void**)&d_components.ref_stokesI,
                                               num_components*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_components.ref_stokesQ,
                                               num_components*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_components.ref_stokesU,
                                               num_components*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_components.ref_stokesV,
                                               num_components*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_components.SIs,
                                               num_components*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_components.ref_freqs,
                                                num_components*sizeof(double) ));

  cudaErrorCheckCall( hipMemcpy(d_components.ref_stokesI, components.ref_stokesI,
                             num_components*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_components.ref_stokesQ, components.ref_stokesQ,
                             num_components*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_components.ref_stokesU, components.ref_stokesU,
                             num_components*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_components.ref_stokesV, components.ref_stokesV,
                             num_components*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_components.SIs, components.SIs,
                             num_components*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_components.ref_freqs, components.ref_freqs,
                        num_components*sizeof(double), hipMemcpyHostToDevice ));

  //This would be done by source_component_common
  cudaErrorCheckCall( hipMalloc( (void**)&d_components.ls, num_components*sizeof(double) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_components.ms, num_components*sizeof(double) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_components.ns, num_components*sizeof(double) ));

  cudaErrorCheckCall( hipMemcpy(d_components.ls, components.ls, num_components*sizeof(double),
                                           hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_components.ms, components.ms, num_components*sizeof(double),
                                           hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_components.ns, components.ns, num_components*sizeof(double),
                                           hipMemcpyHostToDevice ));


  //Something to store the primary beam gains (all 4 pols) in
  d_beam_gains_t d_beam_gains;
  int num_beam_values = num_components*num_freqs*num_times;

  cudaErrorCheckCall( hipMalloc( (void**)&d_beam_gains.d_gxs,
                                      num_beam_values*sizeof(cuUserComplex) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_beam_gains.d_Dxs,
                                      num_beam_values*sizeof(cuUserComplex) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_beam_gains.d_Dys,
                                      num_beam_values*sizeof(cuUserComplex) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_beam_gains.d_gys,
                                      num_beam_values*sizeof(cuUserComplex) ));

  cudaErrorCheckCall( hipMemcpy(d_beam_gains.d_gxs, (cuUserComplex *)gxs,
              num_beam_values*sizeof(cuUserComplex), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_beam_gains.d_Dxs, (cuUserComplex *)Dxs,
              num_beam_values*sizeof(cuUserComplex), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_beam_gains.d_Dys, (cuUserComplex *)Dys,
              num_beam_values*sizeof(cuUserComplex), hipMemcpyHostToDevice ));
  cudaErrorCheckCall( hipMemcpy(d_beam_gains.d_gys, (cuUserComplex *)gys,
              num_beam_values*sizeof(cuUserComplex), hipMemcpyHostToDevice ));

  user_precision_t *d_sum_visi_XX_real = NULL;
  user_precision_t *d_sum_visi_XY_real = NULL;
  user_precision_t *d_sum_visi_YX_real = NULL;
  user_precision_t *d_sum_visi_YY_real = NULL;
  user_precision_t *d_sum_visi_XX_imag = NULL;
  user_precision_t *d_sum_visi_XY_imag = NULL;
  user_precision_t *d_sum_visi_YX_imag = NULL;
  user_precision_t *d_sum_visi_YY_imag = NULL;

  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_XX_real,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_XY_real,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_YX_real,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_YY_real,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_XX_imag,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_XY_imag,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_YX_imag,
                                          num_visis*sizeof(user_precision_t) ));
  cudaErrorCheckCall( hipMalloc( (void**)&d_sum_visi_YY_imag,
                                          num_visis*sizeof(user_precision_t) ));

  //Make sure the visis start at zero by copying across host versions, which
  //should be set to zero already
  cudaErrorCheckCall( hipMemcpy( d_sum_visi_XX_real, sum_visi_XX_real,
    num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMemcpy( d_sum_visi_XY_real, sum_visi_XY_real,
    num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMemcpy( d_sum_visi_YX_real, sum_visi_YX_real,
    num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMemcpy( d_sum_visi_YY_real, sum_visi_YY_real,
    num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMemcpy( d_sum_visi_XX_imag, sum_visi_XX_imag,
    num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMemcpy( d_sum_visi_XY_imag, sum_visi_XY_imag,
    num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMemcpy( d_sum_visi_YX_imag, sum_visi_YX_imag,
    num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ) );
  cudaErrorCheckCall( hipMemcpy( d_sum_visi_YY_imag, sum_visi_YY_imag,
    num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ) );

  dim3 grid, threads;

  threads.x = 128;
  grid.x = (int)ceil( (float)num_visis / (float)threads.x );

  if (comptype == GAUSSIAN || comptype == SHAPELET ) {

    //This would be done by source_component_common
    cudaErrorCheckCall( hipMalloc( (void**)&d_components.pas,
                                                num_components*sizeof(user_precision_t) ));
    cudaErrorCheckCall( hipMalloc( (void**)&d_components.majors,
                                                num_components*sizeof(user_precision_t) ));
    cudaErrorCheckCall( hipMalloc( (void**)&d_components.minors,
                                                num_components*sizeof(user_precision_t) ));

    cudaErrorCheckCall( hipMemcpy(d_components.pas, components.pas,
                       num_components*sizeof(user_precision_t), hipMemcpyHostToDevice ));
    cudaErrorCheckCall( hipMemcpy(d_components.majors, components.majors,
                       num_components*sizeof(user_precision_t), hipMemcpyHostToDevice ));
    cudaErrorCheckCall( hipMemcpy(d_components.minors, components.minors,
                       num_components*sizeof(user_precision_t), hipMemcpyHostToDevice ));
  }

  //Shapelets need many many extra things

  user_precision_t *d_sbf=NULL;
  user_precision_t *d_u_shapes = NULL;
  user_precision_t *d_v_shapes = NULL;
  user_precision_t *d_w_shapes = NULL;

  if (comptype == SHAPELET) {

    cudaErrorCheckCall( hipMalloc( (void**)&d_u_shapes,
                                     num_components*num_visis*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMalloc( (void**)&d_v_shapes,
                                     num_components*num_visis*sizeof(user_precision_t) ) );
    cudaErrorCheckCall( hipMalloc( (void**)&d_w_shapes,
                                     num_components*num_visis*sizeof(user_precision_t) ) );

    cudaErrorCheckCall( hipMemcpy(d_u_shapes, u_shapes,
               num_components*num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ));
    cudaErrorCheckCall( hipMemcpy(d_v_shapes, v_shapes,
               num_components*num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ));
    cudaErrorCheckCall( hipMemcpy(d_w_shapes, w_shapes,
               num_components*num_visis*sizeof(user_precision_t), hipMemcpyHostToDevice ));

    cudaErrorCheckCall( hipMalloc( (void**)&d_components.shape_coeffs,
                                                num_shape_coeffs*sizeof(user_precision_t) ));
    cudaErrorCheckCall( hipMalloc( (void**)&d_components.n1s,
                                                num_shape_coeffs*sizeof(user_precision_t) ));
    cudaErrorCheckCall( hipMalloc( (void**)&d_components.n2s,
                                                num_shape_coeffs*sizeof(user_precision_t) ));
    cudaErrorCheckCall( hipMalloc( (void**)&d_components.param_indexes,
                                                num_shape_coeffs*sizeof(user_precision_t) ));

    cudaErrorCheckCall( hipMemcpy(d_components.shape_coeffs,
                          components.shape_coeffs, num_shape_coeffs*sizeof(user_precision_t),
                          hipMemcpyHostToDevice ));
    cudaErrorCheckCall( hipMemcpy(d_components.n1s,
                          components.n1s, num_shape_coeffs*sizeof(user_precision_t),
                          hipMemcpyHostToDevice ));
    cudaErrorCheckCall( hipMemcpy(d_components.n2s,
                          components.n2s, num_shape_coeffs*sizeof(user_precision_t),
                          hipMemcpyHostToDevice ));
    cudaErrorCheckCall( hipMemcpy(d_components.param_indexes,
                          components.param_indexes, num_shape_coeffs*sizeof(user_precision_t),
                          hipMemcpyHostToDevice ));
    cudaErrorCheckCall( hipMalloc( (void**)&(d_sbf), sbf_N*sbf_L*sizeof(user_precision_t) ));
    cudaErrorCheckCall( hipMemcpy( d_sbf, sbf, sbf_N*sbf_L*sizeof(user_precision_t),
                        hipMemcpyHostToDevice ));
  }

  if (comptype == POINT || comptype == GAUSSIAN ) {

    cudaErrorCheckKernel("kern_calc_visi_point_or_gauss",
                  kern_calc_visi_point_or_gauss, grid, threads,
                  d_components, d_beam_gains,
                  d_us, d_vs, d_ws,
                  d_sum_visi_XX_real, d_sum_visi_XX_imag,
                  d_sum_visi_XY_real, d_sum_visi_XY_imag,
                  d_sum_visi_YX_real, d_sum_visi_YX_imag,
                  d_sum_visi_YY_real, d_sum_visi_YY_imag,
                  d_allsteps_wavelengths, num_components,
                  num_baselines, num_freqs, num_visis,
                  num_times, beamtype, comptype);
  }
  else if (comptype == SHAPELET) {
    cudaErrorCheckKernel("kern_calc_visi_shapelets",
                  kern_calc_visi_shapelets, grid, threads,
                  d_components, d_beam_gains,
                  d_us, d_vs, d_ws,
                  d_allsteps_wavelengths,
                  d_u_shapes, d_v_shapes, d_w_shapes,
                  d_sum_visi_XX_real, d_sum_visi_XX_imag,
                  d_sum_visi_XY_real, d_sum_visi_XY_imag,
                  d_sum_visi_YX_real, d_sum_visi_YX_imag,
                  d_sum_visi_YY_real, d_sum_visi_YY_imag,
                  d_sbf,  num_components,
                  num_baselines, num_freqs, num_visis,
                  num_shape_coeffs, num_times, beamtype);
  }

  cudaErrorCheckCall( hipMemcpy(sum_visi_XX_real, d_sum_visi_XX_real,
                             num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_XY_real, d_sum_visi_XY_real,
                             num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_YX_real, d_sum_visi_YX_real,
                             num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_YY_real, d_sum_visi_YY_real,
                             num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_XX_imag, d_sum_visi_XX_imag,
                             num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_XY_imag, d_sum_visi_XY_imag,
                             num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_YX_imag, d_sum_visi_YX_imag,
                             num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));
  cudaErrorCheckCall( hipMemcpy(sum_visi_YY_imag, d_sum_visi_YY_imag,
                             num_visis*sizeof(user_precision_t), hipMemcpyDeviceToHost ));


  cudaErrorCheckCall(  hipFree( d_sum_visi_XX_real ) );
  cudaErrorCheckCall(  hipFree( d_sum_visi_XX_imag ) );
  cudaErrorCheckCall(  hipFree( d_sum_visi_XY_real ) );
  cudaErrorCheckCall(  hipFree( d_sum_visi_XY_imag ) );
  cudaErrorCheckCall(  hipFree( d_sum_visi_YX_real ) );
  cudaErrorCheckCall(  hipFree( d_sum_visi_YX_imag ) );
  cudaErrorCheckCall(  hipFree( d_sum_visi_YY_real ) );
  cudaErrorCheckCall(  hipFree( d_sum_visi_YY_imag ) );
  cudaErrorCheckCall(  hipFree( d_allsteps_wavelengths ) );

  free_d_components(d_components, comptype);
  free_beam_gains(d_beam_gains, beamtype);

  cudaErrorCheckCall(  hipFree( d_us ) );
  cudaErrorCheckCall(  hipFree( d_vs ) );
  cudaErrorCheckCall(  hipFree( d_ws ) );

  if (comptype == SHAPELET){
    cudaErrorCheckCall(  hipFree( d_sbf) );
    cudaErrorCheckCall(  hipFree( d_u_shapes) );
    cudaErrorCheckCall(  hipFree( d_v_shapes) );
    cudaErrorCheckCall(  hipFree( d_w_shapes) );
  }
}
