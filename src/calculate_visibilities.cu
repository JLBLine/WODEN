#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_complex.h>
#include <complex.h>
#include <math.h>
#include "woden.h"
#include "calculate_visibilities.h"
#include "shapelet_basis.h"
#include "cudacomplex.h"
#include "fundamental_coords.h"
#include "constants.h"
#include "source_components.h"
#include "primary_beam_cuda.h"
#include "FEE_primary_beam_cuda.h"


extern "C" void calculate_visibilities(array_layout_t * array_layout,
  source_catalogue_t *cropped_sky_models,
  float *angles_array, const int num_baselines,
  const int num_time_steps, const int num_visis, const int num_freqs,
  visibility_set_t *visibility_set, visibility_set_t *chunk_visibility_set,
  float *sbf, int num_chunks) {

  // printf("CUDA error 0: %s\n", hipGetErrorString( hipGetLastError() ) );

  /*START We should be able to do all this outside of this function and transfer in--------------*/
  //==========================================================================================
  float *d_X_diff = NULL;
  float *d_Y_diff = NULL;
  float *d_Z_diff = NULL;

  hipMalloc( (void**)&d_X_diff, num_baselines*sizeof(float) );
  hipMemcpy( d_X_diff, array_layout->X_diff_metres, num_baselines*sizeof(float), hipMemcpyHostToDevice );
  hipMalloc( (void**)&d_Y_diff, num_baselines*sizeof(float) );
  hipMemcpy( d_Y_diff, array_layout->Y_diff_metres, num_baselines*sizeof(float), hipMemcpyHostToDevice );
  hipMalloc( (void**)&d_Z_diff, num_baselines*sizeof(float) );
  hipMemcpy( d_Z_diff, array_layout->Z_diff_metres, num_baselines*sizeof(float), hipMemcpyHostToDevice );

  float *d_angles_array = NULL;
  hipMalloc( (void**)&d_angles_array, 3*sizeof(float) );
  hipMemcpy( d_angles_array, angles_array, 3*sizeof(float), hipMemcpyHostToDevice );

  float *d_sha0s = NULL;
  float *d_cha0s = NULL;
  float *d_wavelengths = NULL;
  hipMalloc( (void**)&d_sha0s, num_visis*sizeof(float) );
  hipMemcpy( d_sha0s, visibility_set->sha0s, num_visis*sizeof(float), hipMemcpyHostToDevice );
  hipMalloc( (void**)&d_cha0s, num_visis*sizeof(float) );
  hipMemcpy( d_cha0s, visibility_set->cha0s, num_visis*sizeof(float), hipMemcpyHostToDevice );
  hipMalloc( (void**)&d_wavelengths, num_visis*sizeof(float) );
  hipMemcpy( d_wavelengths, visibility_set->wavelengths, num_visis*sizeof(float), hipMemcpyHostToDevice );
  //
  /* END We should be able to do all this outside of this function and transfer in--------------*/

  float *d_u_metres = NULL;
  float *d_v_metres = NULL;
  float *d_w_metres = NULL;
  float *d_us = NULL;
  float *d_vs = NULL;
  float *d_ws = NULL;

  hipMalloc( (void**)&d_u_metres, num_visis*sizeof(float) );
  hipMalloc( (void**)&d_v_metres, num_visis*sizeof(float) );
  hipMalloc( (void**)&d_w_metres, num_visis*sizeof(float) );
  hipMalloc( (void**)&d_us, num_visis*sizeof(float) );
  hipMalloc( (void**)&d_vs, num_visis*sizeof(float) );
  hipMalloc( (void**)&d_ws, num_visis*sizeof(float) );

  float *d_sum_visi_XX_real;
  float *d_sum_visi_XX_imag;
  float *d_sum_visi_XY_real;
  float *d_sum_visi_XY_imag;
  float *d_sum_visi_YX_real;
  float *d_sum_visi_YX_imag;
  float *d_sum_visi_YY_real;
  float *d_sum_visi_YY_imag;

  hipMalloc( (void**)&d_sum_visi_XX_real, num_visis*sizeof(float) );
  hipMalloc( (void**)&d_sum_visi_XX_imag, num_visis*sizeof(float) );
  hipMalloc( (void**)&d_sum_visi_XY_real, num_visis*sizeof(float) );
  hipMalloc( (void**)&d_sum_visi_XY_imag, num_visis*sizeof(float) );
  hipMalloc( (void**)&d_sum_visi_YX_real, num_visis*sizeof(float) );
  hipMalloc( (void**)&d_sum_visi_YX_imag, num_visis*sizeof(float) );
  hipMalloc( (void**)&d_sum_visi_YY_real, num_visis*sizeof(float) );
  hipMalloc( (void**)&d_sum_visi_YY_imag, num_visis*sizeof(float) );

  float *d_freqs = NULL;
  hipMalloc( (void**)&d_freqs, num_freqs*sizeof(float) );
  hipMemcpy( d_freqs, visibility_set->channel_frequencies, num_freqs*sizeof(float), hipMemcpyHostToDevice );

  //Iterate through all sky model chunks, calculated visibilities are
  //added to chunk_visibility_set, and then summed onto visibility_set

  for (int chunk = 0; chunk < num_chunks; chunk++) {

    catsource_t catsource;
    catsource = cropped_sky_models->catsources[chunk];

    beam_settings_t beam_settings;
    beam_settings = cropped_sky_models->beam_settings[chunk];

    //Make sure the temp visis are 0 at the start of each chunk
    for (int visi = 0; visi < num_visis; visi++) {
      chunk_visibility_set->sum_visi_XX_real[visi] = 0;
      chunk_visibility_set->sum_visi_XX_imag[visi] = 0;
      chunk_visibility_set->sum_visi_XY_real[visi] = 0;
      chunk_visibility_set->sum_visi_XY_imag[visi] = 0;
      chunk_visibility_set->sum_visi_YX_real[visi] = 0;
      chunk_visibility_set->sum_visi_YX_imag[visi] = 0;
      chunk_visibility_set->sum_visi_YY_real[visi] = 0;
      chunk_visibility_set->sum_visi_YY_imag[visi] = 0;
    }

    printf("Processing chunk %d\n", chunk);
    printf("\tNumber of components in chunk are: P %d G %d S_coeffs %d\n",
              cropped_sky_models->catsources[chunk].n_points,
              cropped_sky_models->catsources[chunk].n_gauss,
              cropped_sky_models->catsources[chunk].n_shape_coeffs );

    // calculate_visibilities(arrvay_layout, cropped_sky_models->catsources[chunk],
    //             angles_array, ,
    //             woden_settings->num_baselines, woden_settings->num_time_steps,
    //             num_visis, woden_settings->num_freqs, chunk_visibility_set,
    //             sbf);



    //ensure d_sum_visi_XX_real are set entirely to zero by copying the host
    //array values, which have been set explictly to zero during chunking
    hipMemcpy(d_sum_visi_XX_real, chunk_visibility_set->sum_visi_XX_real,
               num_visis*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(d_sum_visi_XX_imag, chunk_visibility_set->sum_visi_XX_imag,
               num_visis*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(d_sum_visi_XY_real, chunk_visibility_set->sum_visi_XY_real,
               num_visis*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(d_sum_visi_XY_imag, chunk_visibility_set->sum_visi_XY_imag,
               num_visis*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(d_sum_visi_YX_real, chunk_visibility_set->sum_visi_YX_real,
               num_visis*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(d_sum_visi_YX_imag, chunk_visibility_set->sum_visi_YX_imag,
               num_visis*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(d_sum_visi_YY_real, chunk_visibility_set->sum_visi_YY_real,
               num_visis*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(d_sum_visi_YY_imag, chunk_visibility_set->sum_visi_YY_imag,
               num_visis*sizeof(float), hipMemcpyHostToDevice );

    dim3 grid, threads;

    threads.x = 128;
    threads.y = 1;
    grid.x = (int)ceil( (float)num_visis / (float)threads.x );
    grid.y = 1;

    kern_calc_uvw<<< grid, threads >>>( d_X_diff,
            d_Y_diff, d_Z_diff,
            d_u_metres, d_v_metres, d_w_metres,
            d_us, d_vs, d_ws, d_wavelengths,
            d_angles_array, d_cha0s, d_sha0s,
            num_visis, num_baselines);

    int num_points = catsource.n_points;
    int num_gauss = catsource.n_gauss;
    int num_shapes = catsource.n_shapes;



    //TODO currently hardcoded to have beam position angle = 0. Should this change with az/za?
    float cos_theta = 1.0;
    float sin_theta = 0.0;
    float sin_2theta = 0.0;
    float fwhm_lm; //= 20.0 * D2R;

    float *d_beam_ref_freq = NULL;
    float *d_beam_angles_array = NULL;
    if (beam_settings.beamtype == GAUSS_BEAM) {
      // hipMalloc( (void**)&d_beam_ref_freq, sizeof(float) );
      // hipMemcpy( d_beam_ref_freq, beam_settings.beam_ref_freq_array, sizeof(float), hipMemcpyHostToDevice );
      fwhm_lm = sinf(beam_settings.beam_FWHM_rad);

      hipMalloc( (void**)&d_beam_angles_array, 3*sizeof(float) );
      hipMemcpy( d_beam_angles_array, beam_settings.beam_angles_array, 3*sizeof(float), hipMemcpyHostToDevice );

    }

    hipFloatComplex *d_primay_beam_J00 = NULL;
    hipFloatComplex *d_primay_beam_J01 = NULL;
    hipFloatComplex *d_primay_beam_J10 = NULL;
    hipFloatComplex *d_primay_beam_J11 = NULL;

    if (num_points > 0) {
      printf("\tDoing point components\n");

      float *d_point_ras=NULL;
      float *d_point_decs=NULL;
      float *d_point_freqs=NULL;
      float *d_point_stokesI=NULL;
      float *d_point_stokesQ=NULL;
      float *d_point_stokesU=NULL;
      float *d_point_stokesV=NULL;
      float *d_point_SIs=NULL;

      hipMalloc( (void**)&(d_point_ras), num_points*sizeof(float) );
      hipMemcpy( d_point_ras, catsource.point_ras, num_points*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_point_decs), num_points*sizeof(float) );
      hipMemcpy( d_point_decs, catsource.point_decs, num_points*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_point_freqs), num_points*sizeof(float) );
      hipMemcpy( d_point_freqs, catsource.point_ref_freqs, num_points*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_point_stokesI), num_points*sizeof(float) );
      hipMemcpy( d_point_stokesI, catsource.point_ref_stokesI, num_points*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_point_stokesQ), num_points*sizeof(float) );
      hipMemcpy( d_point_stokesQ, catsource.point_ref_stokesQ, num_points*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_point_stokesU), num_points*sizeof(float) );
      hipMemcpy( d_point_stokesU, catsource.point_ref_stokesU, num_points*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_point_stokesV), num_points*sizeof(float) );
      hipMemcpy( d_point_stokesV, catsource.point_ref_stokesV, num_points*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_point_SIs), num_points*sizeof(float) );
      hipMemcpy( d_point_SIs, catsource.point_SIs, num_points*sizeof(float), hipMemcpyHostToDevice );

      //Only the FEE beam currently yields cross pol values, so only malloc what
      //we need here
      if (beam_settings.beamtype == FEE_BEAM) {
        hipMalloc( (void**)&d_primay_beam_J01, beam_settings.num_point_beam_values*sizeof(hipFloatComplex) );
        hipMalloc( (void**)&d_primay_beam_J10, beam_settings.num_point_beam_values*sizeof(hipFloatComplex) );
      }

      hipMalloc( (void**)&d_primay_beam_J00, beam_settings.num_point_beam_values*sizeof(hipFloatComplex) );
      hipMalloc( (void**)&d_primay_beam_J11, beam_settings.num_point_beam_values*sizeof(hipFloatComplex) );

      float *d_ls=NULL;
      float *d_ms=NULL;
      float *d_ns=NULL;

      hipMalloc( (void**)&d_ls, num_points*sizeof(float) );
      hipMalloc( (void**)&d_ms, num_points*sizeof(float) );
      hipMalloc( (void**)&d_ns, num_points*sizeof(float) );

      threads.x = 128;
      threads.y = 1;
      threads.z = 1;
      grid.x = (int)ceil( (float)num_points / (float)threads.x );
      grid.y = 1;
      grid.z = 1;

      kern_calc_lmn<<< grid, threads >>>(d_angles_array, d_point_ras, d_point_decs,
                                 d_ls, d_ms, d_ns, num_points);

      //If using a gaussian primary beam, calculate beam values for all freqs,
      //lsts and point component locations
      if (beam_settings.beamtype == GAUSS_BEAM) {
        printf("\tDoing gaussian beam tings\n");

        calculate_gaussian_beam(num_points, num_time_steps, num_freqs,
             fwhm_lm, cos_theta, sin_theta, sin_2theta,
             beam_settings.beam_ref_freq, d_freqs, d_beam_angles_array,
             beam_settings.beam_point_has, beam_settings.beam_point_decs,
             d_primay_beam_J00, d_primay_beam_J11);

      }// end if beam == GAUSS

      if (beam_settings.beamtype == FEE_BEAM) {

        calc_CUDA_FEE_beam(catsource.point_azs, catsource.point_zas,
               catsource.sin_point_para_angs, catsource.cos_point_para_angs,
               num_points, num_time_steps, beam_settings.FEE_beam);

        threads.x = 64;
        threads.y = 4;
        grid.x = (int)ceil( (float)num_visis / (float)threads.x );
        grid.y = (int)ceil( ((float)num_points) / ((float)threads.y) );

        kern_map_FEE_beam_gains<<< grid, threads >>>(
            (hipFloatComplex *)beam_settings.FEE_beam->d_FEE_beam_gain_matrices,
            d_primay_beam_J00, d_primay_beam_J01,
            d_primay_beam_J10, d_primay_beam_J11,
            num_freqs, num_points, num_visis,
            num_baselines, num_time_steps);

      }

      if (beam_settings.beamtype == ANALY_DIPOLE) {
        printf("\tTrying to do analytic_dipole\n");

        calculate_analytic_dipole_beam(num_points, num_time_steps, num_freqs,
             catsource.point_azs, catsource.point_zas, d_freqs,
             d_primay_beam_J00, d_primay_beam_J11);

      }

      if (num_points == 1) {
        threads.x = 128;
        threads.y = 1;
        grid.x = grid.x = (int)ceil( (float)num_visis / (float)threads.x );
        grid.y = 1;
      }
      else {
        threads.x = 64;
        threads.y = 4;
        grid.x = (int)ceil( (float)num_visis / (float)threads.x );
        grid.y = (int)ceil( ((float)num_points) / ((float)threads.y) );

      }

      kern_calc_visi_point<<<grid , threads>>>(d_point_ras, d_point_decs,
              d_point_freqs, d_point_stokesI, d_point_stokesQ,
              d_point_stokesU, d_point_stokesV, d_point_SIs,
              d_us, d_vs, d_ws,
              d_sum_visi_XX_real, d_sum_visi_XX_imag,
              d_sum_visi_XY_real, d_sum_visi_XY_imag,
              d_sum_visi_YX_real, d_sum_visi_YX_imag,
              d_sum_visi_YY_real, d_sum_visi_YY_imag,
              d_angles_array, d_wavelengths,
              d_ls, d_ms, d_ns,
              num_points, num_baselines, num_freqs, num_visis,
              num_time_steps, beam_settings.beamtype,
              d_primay_beam_J00, d_primay_beam_J01,
              d_primay_beam_J10, d_primay_beam_J11);

      hipFree( beam_settings.FEE_beam->d_FEE_beam_gain_matrices);
      hipFree( d_ns);
      hipFree( d_ms);
      hipFree( d_ls);
      hipFree( d_point_freqs );
      // hipFree( d_point_fluxes );
      hipFree( d_point_stokesI );
      hipFree( d_point_stokesQ );
      hipFree( d_point_stokesU );
      hipFree( d_point_stokesV );
      hipFree( d_point_SIs );
      hipFree( d_point_decs);
      hipFree( d_point_ras);

      hipFree( d_primay_beam_J00 );
      hipFree( d_primay_beam_J11 );

      if (beam_settings.beamtype == FEE_BEAM){
        hipFree( d_primay_beam_J01 );
        hipFree( d_primay_beam_J10 );
      }

    }//if point sources

    if (num_gauss > 0) {
      printf("\tDoing gaussian components\n");

      float *d_gauss_ras=NULL;
      float *d_gauss_decs=NULL;
      float *d_gauss_pas=NULL;
      float *d_gauss_majors=NULL;
      float *d_gauss_minors=NULL;

      float *d_gauss_freqs=NULL;
      float *d_gauss_stokesI=NULL;
      float *d_gauss_stokesQ=NULL;
      float *d_gauss_stokesU=NULL;
      float *d_gauss_stokesV=NULL;
      float *d_gauss_SIs=NULL;

      hipMalloc( (void**)&(d_gauss_freqs), num_gauss*sizeof(float) );
      hipMemcpy( d_gauss_freqs, catsource.gauss_ref_freqs, num_gauss*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_gauss_stokesI), num_gauss*sizeof(float) );
      hipMemcpy( d_gauss_stokesI, catsource.gauss_ref_stokesI, num_gauss*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_gauss_stokesQ), num_gauss*sizeof(float) );
      hipMemcpy( d_gauss_stokesQ, catsource.gauss_ref_stokesQ, num_gauss*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_gauss_stokesU), num_gauss*sizeof(float) );
      hipMemcpy( d_gauss_stokesU, catsource.gauss_ref_stokesU, num_gauss*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_gauss_stokesV), num_gauss*sizeof(float) );
      hipMemcpy( d_gauss_stokesV, catsource.gauss_ref_stokesV, num_gauss*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_gauss_SIs), num_gauss*sizeof(float) );
      hipMemcpy( d_gauss_SIs, catsource.gauss_SIs, num_gauss*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_gauss_ras), num_gauss*sizeof(float) );
      hipMemcpy( d_gauss_ras, catsource.gauss_ras, num_gauss*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_gauss_decs), num_gauss*sizeof(float) );
      hipMemcpy( d_gauss_decs, catsource.gauss_decs, num_gauss*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_gauss_pas), num_gauss*sizeof(float) );
      hipMemcpy( d_gauss_pas, catsource.gauss_pas, num_gauss*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_gauss_majors), num_gauss*sizeof(float) );
      hipMemcpy( d_gauss_majors, catsource.gauss_majors, num_gauss*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_gauss_minors), num_gauss*sizeof(float) );
      hipMemcpy( d_gauss_minors, catsource.gauss_minors, num_gauss*sizeof(float), hipMemcpyHostToDevice );

      //Only the FEE beam currently yields cross pol values, so only malloc what
      //we need here
      if (beam_settings.beamtype == FEE_BEAM) {
        hipMalloc( (void**)&d_primay_beam_J01, beam_settings.num_gausscomp_beam_values*sizeof(hipFloatComplex) );
        hipMalloc( (void**)&d_primay_beam_J10, beam_settings.num_gausscomp_beam_values*sizeof(hipFloatComplex) );
      }

      hipMalloc( (void**)&d_primay_beam_J00, beam_settings.num_gausscomp_beam_values*sizeof(hipFloatComplex) );
      hipMalloc( (void**)&d_primay_beam_J11, beam_settings.num_gausscomp_beam_values*sizeof(hipFloatComplex) );

      float *d_ls=NULL;
      float *d_ms=NULL;
      float *d_ns=NULL;

      hipMalloc( (void**)&d_ls, num_gauss*sizeof(float) );
      hipMalloc( (void**)&d_ms, num_gauss*sizeof(float) );
      hipMalloc( (void**)&d_ns, num_gauss*sizeof(float) );

      threads.x = 128;
      threads.y = 1;
      threads.z = 1;
      grid.x = (int)ceil( (float)num_gauss / (float)threads.x );
      grid.y = 1;
      grid.z = 1;

      kern_calc_lmn<<< grid , threads >>>(d_angles_array, d_gauss_ras, d_gauss_decs,
                                 d_ls, d_ms, d_ns, num_gauss);


      if (beam_settings.beamtype == GAUSS_BEAM) {
        calculate_gaussian_beam(num_gauss, num_time_steps, num_freqs,
             fwhm_lm, cos_theta, sin_theta, sin_2theta,
             beam_settings.beam_ref_freq, d_freqs, d_beam_angles_array,
             beam_settings.beam_gausscomp_has, beam_settings.beam_gausscomp_decs,
             d_primay_beam_J00, d_primay_beam_J11);

      }// end if beam == GAUSS

      if (beam_settings.beamtype == FEE_BEAM) {
        calc_CUDA_FEE_beam(catsource.gauss_azs, catsource.gauss_zas,
               catsource.sin_gauss_para_angs, catsource.cos_gauss_para_angs,
               num_gauss, num_time_steps, beam_settings.FEE_beam);

        threads.x = 64;
        threads.y = 4;
        grid.x = (int)ceil( (float)num_visis / (float)threads.x );
        grid.y = (int)ceil( ((float)num_gauss) / ((float)threads.y) );

        kern_map_FEE_beam_gains<<< grid, threads >>>(
            (hipFloatComplex *)beam_settings.FEE_beam->d_FEE_beam_gain_matrices,
            d_primay_beam_J00, d_primay_beam_J01,
            d_primay_beam_J10, d_primay_beam_J11,
            num_freqs, num_gauss, num_visis,
            num_baselines, num_time_steps);
      }

      if (beam_settings.beamtype == ANALY_DIPOLE) {
      printf("\tTrying to do analytic_dipole\n");

      calculate_analytic_dipole_beam(num_gauss, num_time_steps, num_freqs,
           catsource.gauss_azs, catsource.gauss_zas, d_freqs,
           d_primay_beam_J00, d_primay_beam_J11);

      }

      if (num_gauss == 1) {
       threads.x = 128;
       threads.y = 1;
       grid.x = grid.x = (int)ceil( (float)num_visis / (float)threads.x );
       grid.y = 1;
      }
      else {
       threads.x = 64;
       threads.y = 4;
       grid.x = (int)ceil( (float)num_visis / (float)threads.x );
       grid.y = (int)ceil( ((float)num_gauss) / ((float)threads.y) );
      }

      kern_calc_visi_gaussian<<<grid , threads>>>(d_gauss_ras, d_gauss_decs,
              d_gauss_freqs, d_gauss_stokesI, d_gauss_stokesQ,
              d_gauss_stokesU, d_gauss_stokesV, d_gauss_SIs,
              d_us, d_vs, d_ws,
              d_sum_visi_XX_real, d_sum_visi_XX_imag,
              d_sum_visi_XY_real, d_sum_visi_XY_imag,
              d_sum_visi_YX_real, d_sum_visi_YX_imag,
              d_sum_visi_YY_real, d_sum_visi_YY_imag,
              d_angles_array, d_wavelengths,
              d_ls, d_ms, d_ns,
              d_gauss_pas, d_gauss_majors, d_gauss_minors,
              num_gauss, num_baselines, num_freqs, num_visis,
              num_time_steps, beam_settings.beamtype,
              d_primay_beam_J00, d_primay_beam_J01,
              d_primay_beam_J10, d_primay_beam_J11);

      hipFree( beam_settings.FEE_beam->d_FEE_beam_gain_matrices);
      hipFree( d_primay_beam_J00 );
      hipFree( d_primay_beam_J11 );

      if (beam_settings.beamtype == FEE_BEAM){
        hipFree( d_primay_beam_J01 );
        hipFree( d_primay_beam_J10 );
      }

      hipFree( d_ns);
      hipFree( d_ms);
      hipFree( d_ls);
      hipFree( d_gauss_minors );
      hipFree( d_gauss_majors);
      hipFree( d_gauss_pas);
      hipFree( d_gauss_decs);
      hipFree( d_gauss_ras);
      hipFree( d_gauss_freqs );
      hipFree( d_gauss_stokesI );
      hipFree( d_gauss_stokesQ );
      hipFree( d_gauss_stokesU );
      hipFree( d_gauss_stokesV );
      hipFree( d_gauss_SIs );

    }//if gauss sources

    if (num_shapes > 0) {
      printf("\tDoing shapelet components\n");

      float *d_shape_ras=NULL;
      float *d_shape_decs=NULL;
      float *d_shape_fluxes=NULL;
      float *d_shape_freqs=NULL;
      float *d_shape_pas=NULL;
      float *d_shape_majors=NULL;
      float *d_shape_minors=NULL;

      float *d_shape_coeffs=NULL;
      float *d_shape_n1s=NULL;
      float *d_shape_n2s=NULL;
      float *d_shape_param_indexes=NULL;

      float *d_sbf=NULL;
      float *d_lsts=NULL;

      float *d_u_s_metres = NULL;
      float *d_v_s_metres = NULL;
      float *d_w_s_metres = NULL;

      float *d_shape_ls=NULL;
      float *d_shape_ms=NULL;
      float *d_shape_ns=NULL;

      //Who likes hipMalloc cudaMallocs? We like hipMalloc cudaMallocs
      hipMalloc( (void**)&(d_shape_ras), num_shapes*sizeof(float) );
      hipMemcpy( d_shape_ras, catsource.shape_ras, num_shapes*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_shape_decs), num_shapes*sizeof(float) );
      hipMemcpy( d_shape_decs, catsource.shape_decs, num_shapes*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_shape_fluxes), num_shapes*sizeof(float) );
      hipMemcpy( d_shape_fluxes, catsource.shape_fluxes, num_shapes*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_shape_pas), num_shapes*sizeof(float) );
      hipMemcpy( d_shape_pas, catsource.shape_pas, num_shapes*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_shape_majors), num_shapes*sizeof(float) );
      hipMemcpy( d_shape_majors, catsource.shape_majors, num_shapes*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_shape_minors), num_shapes*sizeof(float) );
      hipMemcpy( d_shape_minors, catsource.shape_minors, num_shapes*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_shape_freqs), num_shapes*sizeof(float) );
      hipMemcpy( d_shape_freqs, catsource.shape_freqs, num_shapes*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_shape_coeffs), catsource.n_shape_coeffs*sizeof(float) );
      hipMemcpy( d_shape_coeffs, catsource.shape_coeffs, catsource.n_shape_coeffs*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_shape_n1s), catsource.n_shape_coeffs*sizeof(float) );
      hipMemcpy( d_shape_n1s, catsource.shape_n1s, catsource.n_shape_coeffs*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_shape_n2s), catsource.n_shape_coeffs*sizeof(float) );
      hipMemcpy( d_shape_n2s, catsource.shape_n2s, catsource.n_shape_coeffs*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_shape_param_indexes), catsource.n_shape_coeffs*sizeof(float) );
      hipMemcpy( d_shape_param_indexes, catsource.shape_param_indexes, catsource.n_shape_coeffs*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_sbf), sbf_N*sbf_L*sizeof(float) );
      hipMemcpy( d_sbf, sbf, sbf_N*sbf_L*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&(d_shape_ls), num_shapes*sizeof(float) );
      hipMalloc( (void**)&(d_shape_ms), num_shapes*sizeof(float) );
      hipMalloc( (void**)&(d_shape_ns), num_shapes*sizeof(float) );

      hipMalloc( (void**)&(d_lsts), num_visis*sizeof(float) );
      hipMemcpy( d_lsts, visibility_set->lsts, num_visis*sizeof(float), hipMemcpyHostToDevice );

      hipMalloc( (void**)&d_u_s_metres, num_shapes*num_visis*sizeof(float) );
      hipMalloc( (void**)&d_v_s_metres, num_shapes*num_visis*sizeof(float) );
      hipMalloc( (void**)&d_w_s_metres, num_shapes*num_visis*sizeof(float) );

      //Only the FEE beam currently yields cross pol values, so only malloc what
      //we need here
      if (beam_settings.beamtype == FEE_BEAM) {
        hipMalloc( (void**)&d_primay_beam_J01, beam_settings.num_shape_beam_values*sizeof(hipFloatComplex) );
        hipMalloc( (void**)&d_primay_beam_J10, beam_settings.num_shape_beam_values*sizeof(hipFloatComplex) );
      }

      hipMalloc( (void**)&d_primay_beam_J00, beam_settings.num_shape_beam_values*sizeof(hipFloatComplex) );
      hipMalloc( (void**)&d_primay_beam_J11, beam_settings.num_shape_beam_values*sizeof(hipFloatComplex) );


      threads.x = 128;
      threads.y = 1;
      grid.x = (int)ceil( ((float)num_shapes / (float)threads.x) );
      grid.y = 1;

      kern_calc_lmn<<< grid , threads, 0 >>>(d_angles_array, d_shape_ras, d_shape_decs,
                               d_shape_ls, d_shape_ms, d_shape_ns, num_shapes);


      if (num_shapes == 1) {
        threads.x = 128;
        threads.y = 1;
        grid.x = (int)ceil( (float)num_visis / (float)threads.x );
        grid.y = 1;
      }
      else {
        threads.x = 64;
        threads.y = 2;
        grid.x = (int)ceil( (float)num_visis / (float)threads.x );
        grid.y = (int)ceil( ((float)num_shapes) / ((float)threads.y) );
      }

      kern_calc_uvw_shapelet<<< grid, threads >>>(d_X_diff, d_Y_diff, d_Z_diff,
            d_u_s_metres, d_v_s_metres, d_w_s_metres,
            d_lsts, d_shape_ras, d_shape_decs,
            num_baselines, num_visis, num_shapes);

      if (beam_settings.beamtype == GAUSS_BEAM) {
        calculate_gaussian_beam(num_shapes, num_time_steps, num_freqs,
             fwhm_lm, cos_theta, sin_theta, sin_2theta,
             beam_settings.beam_ref_freq, d_freqs, d_beam_angles_array,
             beam_settings.beam_shape_has, beam_settings.beam_shape_decs,
             d_primay_beam_J00, d_primay_beam_J11);

      }// end if beam == GAUSS

      if (beam_settings.beamtype == FEE_BEAM) {
        calc_CUDA_FEE_beam(catsource.shape_azs, catsource.shape_zas,
                           catsource.sin_shape_para_angs, catsource.cos_shape_para_angs,
                           num_shapes, num_time_steps, beam_settings.FEE_beam);

        threads.x = 64;
        threads.y = 4;
        grid.x = (int)ceil( (float)num_visis / (float)threads.x );
        grid.y = (int)ceil( ((float)num_shapes) / ((float)threads.y) );

        kern_map_FEE_beam_gains<<< grid, threads >>>(
            (hipFloatComplex *)beam_settings.FEE_beam->d_FEE_beam_gain_matrices,
            d_primay_beam_J00, d_primay_beam_J01,
            d_primay_beam_J10, d_primay_beam_J11,
            num_freqs, num_shapes, num_visis,
            num_baselines, num_time_steps);
      }

      if (beam_settings.beamtype == ANALY_DIPOLE) {
        printf("\tTrying to do analytic_dipole\n");

        calculate_analytic_dipole_beam(num_shapes, num_time_steps, num_freqs,
             catsource.shape_azs, catsource.shape_zas, d_freqs,
             d_primay_beam_J00, d_primay_beam_J11);

      }

      if (catsource.n_shape_coeffs == 1) {
        threads.x = 64;
        threads.y = 1;
        grid.x = (int)ceil( (float)num_visis / (float)threads.x );
        grid.y = 1;
      }
      else {
        threads.x = 64;
        threads.y = 2;
        grid.x = (int)ceil( (float)num_visis / (float)threads.x );
        grid.y = (int)ceil( ((float)catsource.n_shape_coeffs) / ((float)threads.y) );
      }

      kern_calc_visi_shapelets<<< grid, threads >>>(d_shape_ras,
              d_shape_decs, d_shape_fluxes, d_shape_freqs,
              d_us, d_vs, d_ws, d_wavelengths,
              d_u_s_metres, d_v_s_metres, d_w_s_metres,
              d_sum_visi_XX_real, d_sum_visi_XX_imag,
              d_sum_visi_XY_real, d_sum_visi_XY_imag,
              d_sum_visi_YX_real, d_sum_visi_YX_imag,
              d_sum_visi_YY_real, d_sum_visi_YY_imag,
              d_angles_array, d_shape_pas, d_shape_majors, d_shape_minors,
              d_shape_n1s, d_shape_n2s, d_shape_coeffs, d_shape_param_indexes,
              d_shape_ls, d_shape_ms, d_shape_ns,
              d_sbf,
              num_shapes, num_baselines, num_freqs, num_visis,
              catsource.n_shape_coeffs, num_time_steps, beam_settings.beamtype,
              d_primay_beam_J00, d_primay_beam_J01,
              d_primay_beam_J10, d_primay_beam_J11);

      hipFree( beam_settings.FEE_beam->d_FEE_beam_gain_matrices);
      hipFree( d_primay_beam_J00 );
      hipFree( d_primay_beam_J11 );

      if (beam_settings.beamtype == FEE_BEAM){
        hipFree( d_primay_beam_J01 );
        hipFree( d_primay_beam_J10 );
      }

      hipFree( d_shape_ns );
      hipFree( d_shape_ms );
      hipFree( d_shape_ls );
      hipFree(d_w_s_metres);
      hipFree(d_v_s_metres);
      hipFree(d_u_s_metres);
      hipFree(d_lsts);
      hipFree(d_sbf);
      hipFree(d_shape_param_indexes);
      hipFree(d_shape_n2s);
      hipFree(d_shape_n1s);
      hipFree(d_shape_coeffs);
      hipFree( d_shape_minors );
      hipFree( d_shape_majors);
      hipFree( d_shape_pas);
      hipFree( d_shape_freqs );
      hipFree( d_shape_fluxes );
      hipFree( d_shape_decs);
      hipFree( d_shape_ras);

    }//if shapelet

    //Get the results into host memory
    // hipMemcpy(visibility_set->sum_visi_real,d_sum_visi_real,num_visis*sizeof(float),hipMemcpyDeviceToHost);
    // hipMemcpy(visibility_set->sum_visi_imag,d_sum_visi_imag,num_visis*sizeof(float),hipMemcpyDeviceToHost);

    hipMemcpy(chunk_visibility_set->sum_visi_XX_real,d_sum_visi_XX_real,num_visis*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(chunk_visibility_set->sum_visi_XX_imag,d_sum_visi_XX_imag,num_visis*sizeof(float),hipMemcpyDeviceToHost);

    hipMemcpy(chunk_visibility_set->sum_visi_XY_real,d_sum_visi_XY_real,num_visis*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(chunk_visibility_set->sum_visi_XY_imag,d_sum_visi_XY_imag,num_visis*sizeof(float),hipMemcpyDeviceToHost);

    hipMemcpy(chunk_visibility_set->sum_visi_YX_real,d_sum_visi_YX_real,num_visis*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(chunk_visibility_set->sum_visi_YX_imag,d_sum_visi_YX_imag,num_visis*sizeof(float),hipMemcpyDeviceToHost);

    hipMemcpy(chunk_visibility_set->sum_visi_YY_real,d_sum_visi_YY_real,num_visis*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(chunk_visibility_set->sum_visi_YY_imag,d_sum_visi_YY_imag,num_visis*sizeof(float),hipMemcpyDeviceToHost);

    hipMemcpy(chunk_visibility_set->us_metres,d_u_metres,num_visis*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(chunk_visibility_set->vs_metres,d_v_metres,num_visis*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(chunk_visibility_set->ws_metres,d_w_metres,num_visis*sizeof(float),hipMemcpyDeviceToHost);


    //add to visiblity_set
    for (int visi = 0; visi < num_visis; visi++) {
      //if the first chunk then initialise our values, and copy across
      //the u,v,w coords
      if (chunk == 0) {
        //ensure temp visi's are 0.0
        visibility_set->sum_visi_XX_real[visi] = 0;
        visibility_set->sum_visi_XX_imag[visi] = 0;
        visibility_set->sum_visi_XY_real[visi] = 0;
        visibility_set->sum_visi_XY_imag[visi] = 0;
        visibility_set->sum_visi_YX_real[visi] = 0;
        visibility_set->sum_visi_YX_imag[visi] = 0;
        visibility_set->sum_visi_YY_real[visi] = 0;
        visibility_set->sum_visi_YY_imag[visi] = 0;

        visibility_set->us_metres[visi] = chunk_visibility_set->us_metres[visi];
        visibility_set->vs_metres[visi] = chunk_visibility_set->vs_metres[visi];
        visibility_set->ws_metres[visi] = chunk_visibility_set->ws_metres[visi];
      }

      //add each chunk of components to visibility set
      visibility_set->sum_visi_XX_real[visi] += chunk_visibility_set->sum_visi_XX_real[visi];
      visibility_set->sum_visi_XX_imag[visi] += chunk_visibility_set->sum_visi_XX_imag[visi];
      visibility_set->sum_visi_XY_real[visi] += chunk_visibility_set->sum_visi_XY_real[visi];
      visibility_set->sum_visi_XY_imag[visi] += chunk_visibility_set->sum_visi_XY_imag[visi];
      visibility_set->sum_visi_YX_real[visi] += chunk_visibility_set->sum_visi_YX_real[visi];
      visibility_set->sum_visi_YX_imag[visi] += chunk_visibility_set->sum_visi_YX_imag[visi];
      visibility_set->sum_visi_YY_real[visi] += chunk_visibility_set->sum_visi_YY_real[visi];
      visibility_set->sum_visi_YY_imag[visi] += chunk_visibility_set->sum_visi_YY_imag[visi];

    }//visi loop

    if (beam_settings.beamtype == GAUSS_BEAM) {
      hipFree( d_beam_angles_array );
      hipFree( d_beam_ref_freq );
    }
  } //chunk loop

  //Free up the GPU memory

  hipFree( d_freqs );

  hipFree( d_ws );
  hipFree( d_vs );
  hipFree( d_us );
  hipFree( d_w_metres );
  hipFree( d_v_metres );
  hipFree( d_u_metres );
  // hipFree( d_sum_visi_imag );
  // hipFree( d_sum_visi_real );
  hipFree( d_wavelengths );
  hipFree( d_cha0s );
  hipFree( d_sha0s );
  hipFree( d_angles_array );
  hipFree( d_Z_diff );
  hipFree( d_Y_diff );
  hipFree( d_X_diff );


  hipFree( d_sum_visi_XX_imag );
  hipFree( d_sum_visi_XX_real );
  hipFree( d_sum_visi_XY_imag );
  hipFree( d_sum_visi_XY_real );
  hipFree( d_sum_visi_YX_imag );
  hipFree( d_sum_visi_YX_real );
  hipFree( d_sum_visi_YY_imag );
  hipFree( d_sum_visi_YY_real );

}
